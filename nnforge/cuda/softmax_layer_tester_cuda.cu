#include "hip/hip_runtime.h"
/*
 *  Copyright 2011-2014 Maxim Milakov
 *
 *  Licensed under the Apache License, Version 2.0 (the "License");
 *  you may not use this file except in compliance with the License.
 *  You may obtain a copy of the License at
 *
 *      http://www.apache.org/licenses/LICENSE-2.0
 *
 *  Unless required by applicable law or agreed to in writing, software
 *  distributed under the License is distributed on an "AS IS" BASIS,
 *  WITHOUT WARRANTIES OR CONDITIONS OF ANY KIND, either express or implied.
 *  See the License for the specific language governing permissions and
 *  limitations under the License.
 */

#include "softmax_layer_tester_cuda.h"

#include <hip/hip_runtime.h>

#include "util_cuda.h"

#include "../softmax_layer.h"

namespace nnforge
{
	namespace cuda
	{
		extern __shared__ float arr_sh[];

		__global__ void softmax_kernel(
			float * __restrict input,
			int feature_map_count,
			int neuron_count_per_feature_map,
			int entry_count)
		{
			int start_feature_map_id = threadIdx.x;
			int neuron_id = blockIdx.x;
			int entry_id = blockIdx.y;
			int threadblock_size = blockDim.x;

			int thread_id = threadIdx.x;
			int lane_id = thread_id & 31;

		#if __CUDA_ARCH__ < 300
			volatile float * arr2 = arr_sh;
		#endif

			float * start_input = input + (int)((entry_id * feature_map_count + start_feature_map_id) * neuron_count_per_feature_map + neuron_id);
			unsigned int input_step = threadblock_size * neuron_count_per_feature_map;

			// calculate max value
			float max_value;
			{
				max_value = -1.0e+37F;
				float * current_input = start_input;
				for(int feature_map_id = start_feature_map_id; feature_map_id < feature_map_count; feature_map_id += threadblock_size, current_input += input_step)
					max_value = max(max_value, __load_nc(current_input));
			#if __CUDA_ARCH__ < 300
				arr2[thread_id] = max_value;
			#endif
				#pragma unroll
				for(int tx = 16; tx > 0; tx >>= 1)
				{
				#if __CUDA_ARCH__ < 300
					if (lane_id < tx)
						arr2[thread_id] = max(arr2[thread_id], arr2[thread_id + tx]);
				#else
					max_value = max(max_value, __shfl_down(max_value, tx));
				#endif
				}
			#if __CUDA_ARCH__ < 300
				max_value = arr2[thread_id];
				__syncthreads();
			#endif
				if (lane_id == 0)
					arr_sh[thread_id >> 5] = max_value;
				__syncthreads();

				if (thread_id == 0)
				{
					for(int i = 1; i < (blockDim.x >> 5); ++i)
						max_value = max(max_value, arr_sh[i]);
					arr_sh[0] = max_value;
				}
				__syncthreads();

				max_value = arr_sh[0];
			}

			// calculate multiplier
			float mult;
			{
				float predicted_sum = 0.0F;
				float * current_input = start_input;
				for(int feature_map_id = start_feature_map_id; feature_map_id < feature_map_count; feature_map_id += threadblock_size, current_input += input_step)
					predicted_sum += __expf(__load_nc(current_input) - max_value);

			#if __CUDA_ARCH__ < 300
				arr2[thread_id] = predicted_sum;
			#endif
				#pragma unroll
				for(int tx = 16; tx > 0; tx >>= 1)
				{
				#if __CUDA_ARCH__ < 300
					if (lane_id < tx)
						arr2[thread_id] += arr2[thread_id + tx];
				#else
					predicted_sum += __shfl_down(predicted_sum, tx);
				#endif
				}
			#if __CUDA_ARCH__ < 300
				predicted_sum = arr2[thread_id];
				__syncthreads();
			#endif

				if (lane_id == 0)
					arr_sh[thread_id >> 5] = predicted_sum;
				__syncthreads();

				if (thread_id == 0)
				{
					for(int i = 1; i < (blockDim.x >> 5); ++i)
						predicted_sum += arr_sh[i];
					arr_sh[0] = __fdividef(1.0F, predicted_sum);
				}
				__syncthreads();

				mult = arr_sh[0];
			}

			// calculate error and gradient
			float * current_input = start_input;
			for(int feature_map_id = start_feature_map_id; feature_map_id < feature_map_count; feature_map_id += threadblock_size, current_input += input_step)
			{
				float val = __expf(__load_nc(current_input) - max_value);
				*current_input = val * mult;
			}
		}

		softmax_layer_tester_cuda::softmax_layer_tester_cuda()
		{
		}

		softmax_layer_tester_cuda::~softmax_layer_tester_cuda()
		{
		}

		void softmax_layer_tester_cuda::enqueue_test(
			hipStream_t stream_id,
			const std::vector<const_cuda_linear_buffer_device_smart_ptr>& schema_data,
			const std::vector<const_cuda_linear_buffer_device_smart_ptr>& data,
			const std::vector<const_cuda_linear_buffer_device_smart_ptr>& data_custom,
			cuda_linear_buffer_device_smart_ptr input_buffer,
			const std::vector<cuda_linear_buffer_device_smart_ptr>& additional_buffers,
			unsigned int entry_count)
		{
			int threadblock_size = get_threadblock_size(input_configuration_specific.feature_map_count);
			dim3 grid_size(input_elem_count_per_feature_map, entry_count, 1);
			dim3 block_size(threadblock_size, 1, 1);

			int smem_size = threadblock_size * sizeof(float);
			softmax_kernel<<<grid_size, block_size, smem_size, stream_id>>>(
				*input_buffer,
				input_configuration_specific.feature_map_count,
				input_elem_count_per_feature_map,
				entry_count);
		}

		int softmax_layer_tester_cuda::get_threadblock_size(int output_neuron_count)
		{
			int threadblock_size;

			if (output_neuron_count < 256)
			{
				threadblock_size = (output_neuron_count + 32 - 1) / 32 * 32;
			}
			else
			{
				int threadblock_count = (output_neuron_count + 256 - 1) / 256;
				threadblock_size = (output_neuron_count + threadblock_count - 1) / threadblock_count;
				threadblock_size = (threadblock_size + 32 - 1) / 32 * 32;
			}

			return threadblock_size;
		}
	}
}
