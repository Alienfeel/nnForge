#include "hip/hip_runtime.h"
/*
 *  Copyright 2011-2014 Maxim Milakov
 *
 *  Licensed under the Apache License, Version 2.0 (the "License");
 *  you may not use this file except in compliance with the License.
 *  You may obtain a copy of the License at
 *
 *      http://www.apache.org/licenses/LICENSE-2.0
 *
 *  Unless required by applicable law or agreed to in writing, software
 *  distributed under the License is distributed on an "AS IS" BASIS,
 *  WITHOUT WARRANTIES OR CONDITIONS OF ANY KIND, either express or implied.
 *  See the License for the specific language governing permissions and
 *  limitations under the License.
 */

#include "sparse_fully_connected_layer_updater_cuda.h"

#include <hip/hip_runtime.h>

#include "util_cuda.h"
#include "neural_network_cuda_exception.h"
#include "../sparse_convolution_layer.h"

namespace nnforge
{
	namespace cuda
	{
		extern __shared__ float arr_sh[];

		__global__ void sparse_fully_connected_update_biases_upd_kernel(
			float * __restrict gradient_biases,
			const float * __restrict output_errors,
			int block_size,
			int output_elem_count_per_entry,
			int entry_count,
			int block_count)
		{
			int output_neuron_id = blockIdx.x * blockDim.x + threadIdx.x;
			int block_id = blockIdx.y * blockDim.y + threadIdx.y;
			if ((output_neuron_id < output_elem_count_per_entry) && (block_id < block_count))
			{
				int base_entry_id = block_size * block_id;
				int iteration_count = min(entry_count - base_entry_id, block_size);
				const float * current_error = output_errors + (base_entry_id * output_elem_count_per_entry + output_neuron_id);
				float sum = 0.0F;
				for(int i = 0; i < iteration_count; ++i)
				{
					sum += *current_error;
					current_error += output_elem_count_per_entry;
				}
				atomicAdd(gradient_biases + output_neuron_id, sum);
			}
		}

		#define OUTPUT_ELEM_COUNT_BLOCK_SIZE 4
		__global__ void sparse_fully_connected_upd_kernel(
			float * __restrict output_neurons,
			const float * __restrict input_neurons,
			const float * __restrict weights,
			const int * __restrict column_indices,
			const int * __restrict row_ptrs,
			int output_elem_count_per_entry,
			int input_elem_count_per_entry,
			int entry_count,
			int input_feature_map_block_size,
			int window_size)
		{
			int row_id = blockIdx.y * blockDim.y + threadIdx.y;
			if (row_id >= output_elem_count_per_entry)
				return;
			int start_column_index = __load_nc(row_ptrs + row_id);
			int end_column_index = __load_nc(row_ptrs + row_id + 1);
			int thread_id_x = blockIdx.x * blockDim.x + threadIdx.x;
			int base_column_index_offset = (thread_id_x >> 5) * input_feature_map_block_size;
			int base_nnz_index = start_column_index + base_column_index_offset;
			if (base_nnz_index >= end_column_index)
				return;

			int base_entry_id = (blockIdx.z * blockDim.z + threadIdx.z) * OUTPUT_ELEM_COUNT_BLOCK_SIZE;
			if (base_entry_id >= entry_count)
				return;

			int lane_id = thread_id_x & 31;
			int it_count = min(input_feature_map_block_size, end_column_index - base_nnz_index);

			int thread_id = blockDim.x * (threadIdx.z * blockDim.y + threadIdx.y) + threadIdx.x;
			int warp_id = thread_id >> 5;

			volatile int * column_indices_sh = (int *)arr_sh;
			if (lane_id < it_count)
				column_indices_sh[thread_id] = column_indices[base_nnz_index + lane_id];

			int window_it_count = (window_size + 31) >> 5;

			bool valid[OUTPUT_ELEM_COUNT_BLOCK_SIZE];
			int entry_ids[OUTPUT_ELEM_COUNT_BLOCK_SIZE];
			#pragma unroll
			for(int i = 0; i < OUTPUT_ELEM_COUNT_BLOCK_SIZE; ++i)
			{
				valid[i] = (i < (entry_count - base_entry_id));
				entry_ids[i] = valid[i] ? (base_entry_id + i) : (entry_count - 1);
			}

			float sums[OUTPUT_ELEM_COUNT_BLOCK_SIZE];
			#pragma unroll
			for(int i = 0; i < OUTPUT_ELEM_COUNT_BLOCK_SIZE; ++i)
				sums[i] = 0.0F;

			for(int i = 0; i < it_count; ++i)
			{
				int index = base_nnz_index + i;
				int column_id = column_indices_sh[warp_id * 32 + i];
				int local_weight_id = lane_id;
				for(int j = 0; j < window_it_count; ++j)
				{
					if (local_weight_id < window_size)
					{
						float w = __load_nc(weights + (int)(index * window_size + local_weight_id));
						#pragma unroll
						for(int k = 0; k < OUTPUT_ELEM_COUNT_BLOCK_SIZE; ++k)
						{
							float inp = __load_nc(input_neurons + entry_ids[k] * input_elem_count_per_entry + column_id * window_size + local_weight_id);
							sums[k] += w * inp;
						}
					}
					local_weight_id += 32;
				}
			}

		#if __CUDA_ARCH__ < 300
			int threadblock_size = blockDim.x * blockDim.y * blockDim.z;
			volatile float * arr = arr_sh;
			for(int i = 0; i < OUTPUT_ELEM_COUNT_BLOCK_SIZE; ++i)
				arr[i * threadblock_size + thread_id] = sums[i];
		#endif
			#pragma unroll
			for(int tx = 16; tx > 0; tx >>= 1)
			{
			#if __CUDA_ARCH__ < 300
				if (lane_id < tx)
				{
					#pragma unroll
					for(int i = 0; i < OUTPUT_ELEM_COUNT_BLOCK_SIZE; ++i)
						arr[i * threadblock_size + thread_id] += arr[i * threadblock_size + thread_id + tx];
				}
			#else
				#pragma unroll
				for(int i = 0; i < OUTPUT_ELEM_COUNT_BLOCK_SIZE; ++i)
					sums[i] += __shfl_xor(sums[i], tx);
			#endif
			}
		#if __CUDA_ARCH__ < 300
			for(int i = 0; i < OUTPUT_ELEM_COUNT_BLOCK_SIZE; ++i)
				sums[i] = arr[i * threadblock_size + thread_id];
		#endif

			if (lane_id == 0)
			{
				for(int i = 0; i < OUTPUT_ELEM_COUNT_BLOCK_SIZE; ++i)
					if (valid[i])
						atomicAdd(output_neurons + (base_entry_id + i) * output_elem_count_per_entry + row_id, sums[i]);
			}
		}

		#define OUTPUT_ELEM_COUNT_BACKPROP_BLOCK_SIZE 4
		__global__ void sparse_fully_connected_backprop_upd_kernel(
			const float * __restrict output_errors,
			float * __restrict input_errors,
			const float * __restrict weights,
			const int * __restrict column_indices,
			const int * __restrict row_ptrs,
			int output_elem_count_per_entry,
			int input_elem_count_per_entry,
			int entry_count,
			int window_size)
		{
			int row_id = blockIdx.y * blockDim.y + threadIdx.y;
			if (row_id >= output_elem_count_per_entry)
				return;
			int start_column_index = __load_nc(row_ptrs + row_id);
			int end_column_index = __load_nc(row_ptrs + row_id + 1);
			int thread_id_x = blockIdx.x * blockDim.x + threadIdx.x;
			int base_column_index_offset = thread_id_x >> 5;
			int base_nnz_index = start_column_index + base_column_index_offset;
			if (base_nnz_index >= end_column_index)
				return;
			int base_entry_id = (blockIdx.z * blockDim.z + threadIdx.z) * OUTPUT_ELEM_COUNT_BACKPROP_BLOCK_SIZE;
			if (base_entry_id >= entry_count)
				return;

			bool valid[OUTPUT_ELEM_COUNT_BACKPROP_BLOCK_SIZE];
			int entry_ids[OUTPUT_ELEM_COUNT_BACKPROP_BLOCK_SIZE];
			int max_local_entry_count = min(OUTPUT_ELEM_COUNT_BACKPROP_BLOCK_SIZE, entry_count - base_entry_id);
			#pragma unroll
			for(int i = 0; i < OUTPUT_ELEM_COUNT_BACKPROP_BLOCK_SIZE; ++i)
			{
				valid[i] = (i < max_local_entry_count);
				entry_ids[i] = valid[i] ? (base_entry_id + i) : (entry_count - 1);
			}

			int column_id = __load_nc(column_indices + base_nnz_index);

			int window_it_count = (window_size + 31) >> 5;

			int lane_id = thread_id_x & 31;
			int thread_id = blockDim.x * (threadIdx.z * blockDim.y + threadIdx.y) + threadIdx.x;
			int warp_id = thread_id >> 5;
			volatile float * output_errors_sh = arr_sh + warp_id * OUTPUT_ELEM_COUNT_BACKPROP_BLOCK_SIZE;
			if (lane_id < max_local_entry_count)
					output_errors_sh[lane_id] = __load_nc(output_errors + (int)((base_entry_id + lane_id) * output_elem_count_per_entry + row_id));

			int local_weight_id = lane_id;
			for(int j = 0; j < window_it_count; ++j)
			{
				if (local_weight_id < window_size)
				{
					float w = __load_nc(weights + (int)(base_nnz_index * window_size + local_weight_id));
					#pragma unroll
					for(int k = 0; k < OUTPUT_ELEM_COUNT_BACKPROP_BLOCK_SIZE; ++k)
					{
						if (valid[k])
						{
							float input_error = output_errors_sh[k] * w;
							atomicAdd(input_errors + (int)(entry_ids[k] * input_elem_count_per_entry + column_id * window_size + local_weight_id), input_error);
						}
					}
				}
				local_weight_id += 32;
			}
		}
		
		__global__ void sparse_fully_connected_update_weights_kernel(
			const float * __restrict output_errors,
			const float * __restrict input_neurons,
			float * __restrict gradient_weights,
			const int * __restrict column_indices,
			const int * __restrict row_ptrs,
			int output_elem_count_per_entry,
			int input_elem_count_per_entry,
			int entry_block_size,
			int entry_count,
			int window_size)
		{
			int row_id = blockIdx.y * blockDim.y + threadIdx.y;
			if (row_id >= output_elem_count_per_entry)
				return;
			int start_column_index = __load_nc(row_ptrs + row_id);
			int end_column_index = __load_nc(row_ptrs + row_id + 1);
			int thread_id_x = blockIdx.x * blockDim.x + threadIdx.x;
			int base_column_index_offset = thread_id_x >> 5;
			int base_nnz_index = start_column_index + base_column_index_offset;
			if (base_nnz_index >= end_column_index)
				return;
			int base_entry_id = (blockIdx.z * blockDim.z + threadIdx.z) * entry_block_size;
			if (base_entry_id >= entry_count)
				return;

			int local_entry_count = min(entry_block_size, entry_count - base_entry_id);

			int column_id = __load_nc(column_indices + base_nnz_index);

			int window_it_count = (window_size + 31) >> 5;

			int lane_id = thread_id_x & 31;
			int thread_id = blockDim.x * (threadIdx.z * blockDim.y + threadIdx.y) + threadIdx.x;
			int warp_id = thread_id >> 5;
			volatile float * output_errors_sh = arr_sh + warp_id * entry_block_size;
			if (lane_id < local_entry_count)
					output_errors_sh[lane_id] = __load_nc(output_errors + (int)((base_entry_id + lane_id) * output_elem_count_per_entry + row_id));

			int local_weight_id = lane_id;
			for(int j = 0; j < window_it_count; ++j)
			{
				if (local_weight_id < window_size)
				{
					float sum = 0.0F;
					const float * current_input_neurons = input_neurons + base_entry_id * input_elem_count_per_entry + column_id * window_size + local_weight_id;
					for(int k = 0; k < local_entry_count; ++k)
					{
						sum += output_errors_sh[k] * __load_nc(current_input_neurons);
						current_input_neurons += input_elem_count_per_entry;
					}
					atomicAdd(gradient_weights + (int)(base_nnz_index * window_size + local_weight_id), sum);
				}
				local_weight_id += 32;
			}
		}
		
		const int sparse_fully_connected_layer_updater_cuda::max_input_feature_map_block_size = 32;
		const int sparse_fully_connected_layer_updater_cuda::absolute_min_update_entry_count_block_size = 4;
		const int sparse_fully_connected_layer_updater_cuda::absolute_max_update_entry_count_block_size = 32;

		sparse_fully_connected_layer_updater_cuda::sparse_fully_connected_layer_updater_cuda()
		{
		}

		sparse_fully_connected_layer_updater_cuda::~sparse_fully_connected_layer_updater_cuda()
		{
		}

		void sparse_fully_connected_layer_updater_cuda::enqueue_test(
			unsigned int offset_input_entry_id,
			hipStream_t stream_id,
			const std::vector<const_cuda_linear_buffer_device_smart_ptr>& schema_data,
			const std::vector<cuda_linear_buffer_device_smart_ptr>& data,
			const std::vector<cuda_linear_buffer_device_smart_ptr>& data_custom,
			const_cuda_linear_buffer_device_smart_ptr input_neurons_buffer,
			cuda_linear_buffer_device_smart_ptr output_neurons_buffer,
			const std::vector<cuda_linear_buffer_device_smart_ptr>& additional_buffers,
			std::vector<cuda_memobject_smart_ptr>& dynamic_memobjects,
			unsigned int entry_count)
		{
			// Copy bias
			cuda_util::duplicate_vector(
				*cuda_config,
				*data[1],
				*output_neurons_buffer,
				output_elem_count_per_entry,
				entry_count,
				stream_id);

			std::pair<int, int> input_feature_map_block_size_and_count = get_input_feature_map_block_size_and_count();
			std::pair<dim3, dim3> kernel_dims = cuda_util::get_grid_and_threadblock_sizes_sequential_access(
				*cuda_config,
				32 * input_feature_map_block_size_and_count.second,
				output_elem_count_per_entry,
				(entry_count + OUTPUT_ELEM_COUNT_BLOCK_SIZE - 1) / OUTPUT_ELEM_COUNT_BLOCK_SIZE,
				32);
			int threadblock_size = kernel_dims.second.x * kernel_dims.second.y * kernel_dims.second.z;
			int smem_size = (cuda_config->get_compute_capability() < 300) ? OUTPUT_ELEM_COUNT_BLOCK_SIZE * threadblock_size * sizeof(float) : threadblock_size * sizeof(float);
			sparse_fully_connected_upd_kernel<<<kernel_dims.first, kernel_dims.second, smem_size, stream_id>>>(
				*output_neurons_buffer,
				*input_neurons_buffer,
				*data[0],
				*data_custom[0],
				*data_custom[1],
				output_elem_count_per_entry,
				input_elem_count_per_entry,
				entry_count,
				input_feature_map_block_size_and_count.first,
				window_size);
		}

		void sparse_fully_connected_layer_updater_cuda::enqueue_backprop(
			hipStream_t stream_id,
			const std::vector<const_cuda_linear_buffer_device_smart_ptr>& schema_data,
			const std::vector<cuda_linear_buffer_device_smart_ptr>& data,
			const std::vector<cuda_linear_buffer_device_smart_ptr>& data_custom,
			const_cuda_linear_buffer_device_smart_ptr output_neurons_buffer,
			const_cuda_linear_buffer_device_smart_ptr input_neurons_buffer,
			cuda_linear_buffer_device_smart_ptr output_errors_buffer,
			cuda_linear_buffer_device_smart_ptr input_errors_buffer,
			const std::vector<cuda_linear_buffer_device_smart_ptr>& additional_buffers,
			std::vector<cuda_memobject_smart_ptr>& dynamic_memobjects,
			unsigned int entry_count)
		{
			cuda_util::set_with_value(
				*cuda_config,
				*input_errors_buffer,
				0.0F,
				input_elem_count_per_entry * entry_count,
				stream_id);

			std::pair<dim3, dim3> kernel_dims = cuda_util::get_grid_and_threadblock_sizes_sequential_access(
				*cuda_config,
				32 * max_column_index_count_per_row,
				output_elem_count_per_entry,
				(entry_count + OUTPUT_ELEM_COUNT_BACKPROP_BLOCK_SIZE - 1) / OUTPUT_ELEM_COUNT_BACKPROP_BLOCK_SIZE,
				32);
			int threadblock_size = kernel_dims.second.x * kernel_dims.second.y * kernel_dims.second.z;
			int smem_size = (threadblock_size / 32) * OUTPUT_ELEM_COUNT_BACKPROP_BLOCK_SIZE * sizeof(float);
			sparse_fully_connected_backprop_upd_kernel<<<kernel_dims.first, kernel_dims.second, smem_size, stream_id>>>(
				*output_errors_buffer,
				*input_errors_buffer,
				*data[0],
				*data_custom[0],
				*data_custom[1],
				output_elem_count_per_entry,
				input_elem_count_per_entry,
				entry_count,
				window_size);
		}

		void sparse_fully_connected_layer_updater_cuda::enqueue_update_weights(
			unsigned int offset_input_entry_id,
			hipStream_t stream_id,
			const std::vector<cuda_linear_buffer_device_smart_ptr>& gradient,
			const std::vector<cuda_linear_buffer_device_smart_ptr>& data_custom,
			const std::vector<const_cuda_linear_buffer_device_smart_ptr>& schema_data,
			cuda_linear_buffer_device_smart_ptr output_errors_buffer,
			const_cuda_linear_buffer_device_smart_ptr input_neurons_buffer,
			const std::vector<cuda_linear_buffer_device_smart_ptr>& additional_buffers,
			std::vector<cuda_memobject_smart_ptr>& dynamic_memobjects,
			unsigned int entry_count)
		{
			// Update weights
			{
				std::pair<int, int> entry_block_size_and_count = get_update_entry_block_size_and_count(entry_count);
				std::pair<dim3, dim3> kernel_dims = cuda_util::get_grid_and_threadblock_sizes_sequential_access(
					*cuda_config,
					32 * max_column_index_count_per_row,
					output_elem_count_per_entry,
					entry_block_size_and_count.second,
					32);
				int threadblock_size = kernel_dims.second.x * kernel_dims.second.y * kernel_dims.second.z;
				int smem_size = (threadblock_size / 32) * entry_block_size_and_count.first * sizeof(float);
				sparse_fully_connected_update_weights_kernel<<<kernel_dims.first, kernel_dims.second, smem_size, stream_id>>>(
					*output_errors_buffer,
					*input_neurons_buffer,
					*gradient[0],
					*data_custom[0],
					*data_custom[1],
					output_elem_count_per_entry,
					input_elem_count_per_entry,
					entry_block_size_and_count.first,
					entry_count,
					window_size);
			}

			// Update biases
			{
				int block_size = get_block_size(entry_count);
				int block_count = (entry_count + block_size - 1) / block_size;
				std::pair<dim3, dim3> kernel_dims = cuda_util::get_grid_and_threadblock_sizes_sequential_access(
					*cuda_config,
					output_elem_count_per_entry,
					block_count,
					1);
				sparse_fully_connected_update_biases_upd_kernel<<<kernel_dims.first, kernel_dims.second, 0, stream_id>>>(
					*gradient[1],
					*output_errors_buffer,
					block_size,
					output_elem_count_per_entry,
					entry_count,
					block_count);
			}
		}

		bool sparse_fully_connected_layer_updater_cuda::is_in_place_backprop() const
		{
			return false;
		}

		int sparse_fully_connected_layer_updater_cuda::get_block_size(int entry_count)
		{
			int block_size = std::min<int>(std::max<int>(static_cast<int>(sqrtf(static_cast<float>(entry_count))), 1), entry_count);
			return block_size;
		}

		void sparse_fully_connected_layer_updater_cuda::updater_configured()
		{
			nnforge_shared_ptr<const sparse_convolution_layer> layer_derived = nnforge_dynamic_pointer_cast<const sparse_convolution_layer>(layer_schema);

			feature_map_connection_count = layer_derived->feature_map_connection_count;

			window_size = 1;
			for(std::vector<unsigned int>::const_iterator it = layer_derived->window_sizes.begin(); it != layer_derived->window_sizes.end(); ++it)
				window_size *= *it;

			int input_data_single_backprop_entry_size = input_elem_count_per_entry * sizeof(float);
			max_update_entry_count_block_size = std::min(std::max(absolute_min_update_entry_count_block_size, cuda_config->l2_cache_size / 2 / input_data_single_backprop_entry_size), absolute_max_update_entry_count_block_size);
		}

		std::vector<size_t> sparse_fully_connected_layer_updater_cuda::get_sizes_of_additional_buffers_per_entry() const
		{
			std::vector<size_t> res;

			return res;
		}

		void sparse_fully_connected_layer_updater_cuda::notify_data_custom(const_layer_data_custom_smart_ptr host_data_custom)
		{
			max_column_index_count_per_row = 0;
			const std::vector<int>& row_indices = host_data_custom->at(1);
			for(int i = 0; i < row_indices.size() - 1; ++i)
				max_column_index_count_per_row = std::max(max_column_index_count_per_row, row_indices[i + 1] - row_indices[i]);
		}

		std::pair<int, int> sparse_fully_connected_layer_updater_cuda::get_input_feature_map_block_size_and_count() const
		{
			int candidate_block_size = max_column_index_count_per_row;

			if (candidate_block_size <= max_input_feature_map_block_size)
				return std::make_pair(candidate_block_size, 1);

			int candidate_block_count2 = (candidate_block_size + max_input_feature_map_block_size - 1) / max_input_feature_map_block_size;
			int candidate_block_size2 = (candidate_block_size + candidate_block_count2 - 1) / candidate_block_count2;

			return std::make_pair(candidate_block_size2, candidate_block_count2);
		}

		std::pair<int, int> sparse_fully_connected_layer_updater_cuda::get_update_entry_block_size_and_count(unsigned int entry_count) const
		{
			int candidate_block_size = entry_count;

			if (candidate_block_size <= max_update_entry_count_block_size)
				return std::make_pair(candidate_block_size, 1);

			int candidate_block_count2 = (candidate_block_size + max_update_entry_count_block_size - 1) / max_update_entry_count_block_size;
			int candidate_block_size2 = (candidate_block_size + candidate_block_count2 - 1) / candidate_block_count2;

			return std::make_pair(candidate_block_size2, candidate_block_count2);
		}
	}
}
