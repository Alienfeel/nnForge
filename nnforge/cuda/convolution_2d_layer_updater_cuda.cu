#include "hip/hip_runtime.h"
/*
 *  Copyright 2011-2013 Maxim Milakov
 *
 *  Licensed under the Apache License, Version 2.0 (the "License");
 *  you may not use this file except in compliance with the License.
 *  You may obtain a copy of the License at
 *
 *      http://www.apache.org/licenses/LICENSE-2.0
 *
 *  Unless required by applicable law or agreed to in writing, software
 *  distributed under the License is distributed on an "AS IS" BASIS,
 *  WITHOUT WARRANTIES OR CONDITIONS OF ANY KIND, either express or implied.
 *  See the License for the specific language governing permissions and
 *  limitations under the License.
 */

#include "convolution_2d_layer_updater_cuda.h"

#include <hip/hip_runtime.h>

#include <boost/format.hpp>

#include "cuda_util.h"
#include "neural_network_cuda_exception.h"
#include "../convolution_layer.h"

texture<float, hipTextureType1D, hipReadModeElementType> input_tex_ref;

template<bool different_input>
__global__ void convolution_2d_tex_upd_kernel(
	float * __restrict output,
	const float * __restrict weights,
	const float * __restrict biases,
	int output_width,
	int output_height,
	int input_width,
	int input_height,
	int window_width,
	int window_height,
	int input_feature_map_count,
	int output_feature_map_count,
	int texture_offset,
	int entry_count)
{
	int x = blockIdx.x * blockDim.x + threadIdx.x;
	int _y = blockIdx.y * blockDim.y + threadIdx.y;
	int output_feature_map_id = _y / output_height;
	int entry_id = blockIdx.z * blockDim.z + threadIdx.z;

	bool in_bounds = (entry_id < entry_count) && (x < output_width) && (output_feature_map_id < output_feature_map_count);
	if (in_bounds)
	{
		int y = _y - (output_feature_map_id * output_height);
		int input_elem_id = ((different_input ? entry_id * input_feature_map_count * input_height : 0) + y) * input_width + x + texture_offset;
		const float * current_weights = weights + (int)((entry_id * output_feature_map_count + output_feature_map_id) * window_width * window_height * input_feature_map_count);

		float sum = biases[output_feature_map_id + entry_id * output_feature_map_count];

		for(int input_layer_id = 0; input_layer_id < input_feature_map_count; ++input_layer_id)
		{
			for(int input_y = 0; input_y < window_height; ++input_y)
			{
				#pragma unroll 4
				for(int input_x = 0; input_x < window_width; ++input_x)
				{
					sum += tex1Dfetch(input_tex_ref, input_elem_id) * *current_weights;
					current_weights++;
					input_elem_id++;
				}
				input_elem_id += input_width - window_width;
			}
			input_elem_id += input_width * (input_height - window_height);
		}

		output[((entry_id * output_feature_map_count + output_feature_map_id) * output_height + y) * output_width + x] = sum;
	}
}

template<int WINDOW_WIDTH, int BLOCK_SIZE, bool different_input>
__global__ void convolution_2d_tex_exact_blocked_upd_kernel(
	float * __restrict output,
	const float * __restrict weights,
	const float * __restrict biases,
	int output_width,
	int output_height,
	int input_width,
	int input_height,
	int window_height,
	int input_feature_map_count,
	int output_feature_map_count,
	int block_count,
	int input_feature_map_group_count,
	int input_feature_map_group_size,
	int texture_offset,
	int entry_count)
{
	int xy = blockIdx.x * blockDim.x + threadIdx.x;
	int y = xy / block_count;
	int dd = blockIdx.y * blockDim.y + threadIdx.y;
	int input_feature_map_group_id = dd / output_feature_map_count;
	int entry_id = blockIdx.z * blockDim.z + threadIdx.z;

	bool in_bounds = (entry_id < entry_count) && (y < output_height) && (input_feature_map_group_id < input_feature_map_group_count);
	if (in_bounds)
	{
		int output_feature_map_id = dd - (input_feature_map_group_id * output_feature_map_count);
		int base_input_feature_map_id = input_feature_map_group_id * input_feature_map_group_size;
		int x = (xy - (y * block_count)) * BLOCK_SIZE;
		int input_elem_id = ((((different_input ? entry_id * input_feature_map_count : 0) + base_input_feature_map_id) * input_height) + y) * input_width + x + texture_offset;
		const float * current_weights = weights + (int)(((entry_id * output_feature_map_count + output_feature_map_id) * input_feature_map_count + base_input_feature_map_id) * WINDOW_WIDTH * window_height);
		int iteration_count = min(input_feature_map_group_size, input_feature_map_count - base_input_feature_map_id);

		float initial_value = 0.0F;
		if (input_feature_map_group_id == 0)
			initial_value = biases[output_feature_map_id + entry_id * output_feature_map_count];
		float sums[BLOCK_SIZE];
		#pragma unroll
		for(int i = 0; i < BLOCK_SIZE; ++i)
			sums[i] = initial_value;

		for(int i = 0; i < iteration_count; ++i)
		{
			for(int input_y = 0; input_y < window_height; ++input_y)
			{
				#pragma unroll
				for(int input_x = 0; input_x < WINDOW_WIDTH; ++input_x)
				{
					float weight = *current_weights;
					#pragma unroll
					for(int i = 0; i < BLOCK_SIZE; ++i)
						sums[i] += tex1Dfetch(input_tex_ref, input_elem_id + i) * weight;
					current_weights++;
					input_elem_id++;
				}
				input_elem_id += input_width - WINDOW_WIDTH;
			}
			input_elem_id += input_width * (input_height - window_height);
		}

		float * base_output = output + ((entry_id * output_feature_map_count + output_feature_map_id) * output_height + y) * output_width + x;
		if (input_feature_map_group_count == 1)
		{
			#pragma unroll
			for(int i = 0; i < BLOCK_SIZE; ++i)
			{
				if (i < output_width - x)
					base_output[i] = sums[i];
			}
		}
		else
		{
			#pragma unroll
			for(int i = 0; i < BLOCK_SIZE; ++i)
			{
				if (i < output_width - x)
					atomicAdd(base_output + i, sums[i]);
			}
		}
	}
}

extern __shared__ float arr[];
__global__ void convolution_2d_update_biases_upd_kernel(
	float * __restrict biases,
	const float * __restrict output_errors,
	const float * __restrict training_speed,
	int output_feature_map_count,
	int output_elem_count_per_feature_map,
	int min_iteration_count)
{
	int thread_id = threadIdx.x;
	int output_feature_map_id = blockIdx.y;
	int entry_id = blockIdx.z;
	int threadblock_size = blockDim.x;
	float sum = 0.0F;
	const float * current_error = output_errors + (entry_id * output_feature_map_count + output_feature_map_id) * output_elem_count_per_feature_map;
	int current_output_neuron_id = thread_id;
	for(int i = 0; i < min_iteration_count; ++i)
	{
		sum += current_error[current_output_neuron_id];
		current_output_neuron_id += threadblock_size;
	}
	if (current_output_neuron_id < output_elem_count_per_feature_map)
		sum += current_error[current_output_neuron_id];
	arr[thread_id] = sum;
	__syncthreads();

	int offset = entry_id * output_feature_map_count + output_feature_map_id;
	float current_bias_val;
	float current_training_speed_val;
	if (thread_id == 0)
	{
		current_bias_val = biases[offset];
		current_training_speed_val = training_speed[offset];
	}

	int t_add_elems = threadblock_size >> 1;
	int t_working_elems = (threadblock_size + 1) >> 1;
	while (t_add_elems > 0)
	{
		if (thread_id < t_add_elems)
			arr[thread_id] += arr[thread_id + t_working_elems];
		t_add_elems = t_working_elems >> 1;
		t_working_elems = (t_working_elems + 1) >> 1;
		__syncthreads();
	}

	if (thread_id == 0)
		biases[offset] = arr[0] * current_training_speed_val + current_bias_val;
}

texture<float, hipTextureType1D, hipReadModeElementType> output_tex_ref;

__global__ void convolution_2d_deriviative_tex_upd_kernel(
	float * __restrict input_errors,
	const float * __restrict weights,
	int output_width,
	int output_height,
	int input_width,
	int input_height,
	int window_width,
	int window_height,
	int input_feature_map_count,
	int output_feature_map_count,
	int entry_count)
{
	int x = blockIdx.x * blockDim.x + threadIdx.x;
	int _y = blockIdx.y * blockDim.y + threadIdx.y;
	int input_feature_map_id = _y / input_height;
	int entry_id = blockIdx.z * blockDim.z + threadIdx.z;

	bool in_bounds = (entry_id < entry_count) && (x < input_width) && (input_feature_map_id < input_feature_map_count);
	if (in_bounds)
	{
		int y = _y - (input_feature_map_id * input_height);
		int output_elem_id = (entry_id * output_feature_map_count * output_height + y) * output_width + x;
		const float * current_weights = weights + (int)((entry_id * input_feature_map_count * output_feature_map_count + input_feature_map_id) * window_width * window_height);

		float sum = 0.0F;

		int min_y_exclusive = y - output_height;
		int max_y_inclusive = y;
		int min_x_exclusive = x - output_width;
		int max_x_inclusive = x;
		for(int output_layer_id = 0; output_layer_id < output_feature_map_count; ++output_layer_id)
		{
			for(int input_y = 0; input_y < window_height; ++input_y)
			{
				bool b_fit1 = (input_y > min_y_exclusive) && (input_y <= max_y_inclusive);
				for(int input_x = 0; input_x < window_width; ++input_x)
				{
					bool b_fit2 = b_fit1 && (input_x > min_x_exclusive) && (input_x <= max_x_inclusive);
					if (b_fit2)
						sum += tex1Dfetch(output_tex_ref, output_elem_id) * *current_weights;
					current_weights++;
					output_elem_id--;
				}
				output_elem_id -= output_width - window_width;
			}
			current_weights += window_width * window_height * (input_feature_map_count - 1);
			output_elem_id += output_width * (output_height + window_height);
		}

		input_errors[((entry_id * input_feature_map_count + input_feature_map_id) * input_height + y) * input_width + x] = sum;
	}
}

template<int WINDOW_WIDTH, int BLOCK_SIZE>
__global__ void convolution_2d_deriviative_tex_exact_blocked_upd_kernel(
	float * __restrict input_errors,
	const float * __restrict weights,
	int output_width,
	int output_height,
	int input_width,
	int input_height,
	int window_height,
	int input_feature_map_count,
	int output_feature_map_count,
	int block_count,
	int output_feature_map_group_count,
	int output_feature_map_group_size,
	int entry_count)
{
	int xy = blockIdx.x * blockDim.x + threadIdx.x;
	int y = xy / block_count;
	int dd = blockIdx.y * blockDim.y + threadIdx.y;
	int output_feature_map_group_id = dd / input_feature_map_count;
	int entry_id = blockIdx.z * blockDim.z + threadIdx.z;

	bool in_bounds = (entry_id < entry_count) && (y < input_height) && (output_feature_map_group_id < output_feature_map_group_count);
	if (in_bounds)
	{
		int input_feature_map_id = dd - (output_feature_map_group_id * input_feature_map_count);
		int base_output_feature_map_id = output_feature_map_group_id * output_feature_map_group_size;
		int x = (xy - (y * block_count)) * BLOCK_SIZE + (BLOCK_SIZE - 1);
		int output_elem_id = ((entry_id * output_feature_map_count + base_output_feature_map_id) * output_height + y) * output_width + x;
		const float * current_weights = weights + (int)(((entry_id * output_feature_map_count + base_output_feature_map_id) * input_feature_map_count + input_feature_map_id) * WINDOW_WIDTH * window_height);
		int iteration_count = min(output_feature_map_group_size, output_feature_map_count - base_output_feature_map_id);

		float sums[BLOCK_SIZE];
		#pragma unroll
		for(int i = 0; i < BLOCK_SIZE; ++i)
			sums[i] = 0.0F;

		int min_y_exclusive = y - output_height;
		int max_y_inclusive = y;
		int min_x_exclusive = x - output_width;
		int max_x_inclusive = x;

		unsigned int mask = 0;
		for(int i = BLOCK_SIZE + WINDOW_WIDTH - 2; i >= 0; --i)
			mask = mask << 1 | (((i > min_x_exclusive) && (i <= max_x_inclusive)) ? 1 : 0);

		for(int i = 0; i < iteration_count; ++i)
		{
			for(int input_y = 0; input_y < window_height; ++input_y)
			{
				bool b_fit1 = (input_y > min_y_exclusive) && (input_y <= max_y_inclusive);

				float output_vals[BLOCK_SIZE + WINDOW_WIDTH - 1];
				#pragma unroll
				for(int i = 0; i < BLOCK_SIZE + WINDOW_WIDTH - 1; ++i)
				{
					bool b_fit2 = b_fit1 && (((1 << i) & mask) != 0);
					if (b_fit2)
						output_vals[i] = tex1Dfetch(output_tex_ref, output_elem_id - i);
					else
						output_vals[i] = 0.0F;
				}

				#pragma unroll
				for(int input_x = 0; input_x < WINDOW_WIDTH; ++input_x)
				{
					float weight = *current_weights;
					#pragma unroll
					for(int i = 0; i < BLOCK_SIZE; ++i)
						sums[i] += output_vals[input_x + i] * weight;
					current_weights++;
				}
				output_elem_id -= output_width;
			}
			current_weights += WINDOW_WIDTH * window_height * (input_feature_map_count - 1);
			output_elem_id += output_width * (output_height + window_height);
		}

		float * base_input = input_errors + ((entry_id * input_feature_map_count + input_feature_map_id) * input_height + y) * input_width + x;
		if (output_feature_map_group_count == 1)
		{
			#pragma unroll
			for(int i = 0; i < BLOCK_SIZE; ++i)
			{
				if (i > x - input_width)
					*(base_input - i) = sums[i];
			}
		}
		else
		{
			#pragma unroll
			for(int i = 0; i < BLOCK_SIZE; ++i)
			{
				if (i > x - input_width)
					atomicAdd(base_input - i, sums[i]);
			}
		}
	}
}

template<bool different_input>
__global__ void convolution_2d_update_weights_upd_kernel(
	float * __restrict weights,
	const float * __restrict output_errors,
	const float * __restrict training_speed,
	int output_width,
	int output_height,
	int input_width,
	int input_height,
	int window_width,
	int window_height,
	int input_feature_map_count,
	int output_feature_map_count,
	int texture_offset,
	int entry_count)
{
	int internal_weights_id = blockIdx.x * blockDim.x + threadIdx.x;
	int feature_map_pair_id = blockIdx.y * blockDim.y + threadIdx.y;
	int entry_id = blockIdx.z * blockDim.z + threadIdx.z;
	int weight_y = internal_weights_id / window_width;
	int output_feature_map_id = feature_map_pair_id / input_feature_map_count;

	if ((weight_y < window_height) && (output_feature_map_id < output_feature_map_count) && (entry_id < entry_count))
	{
		int weight_x = internal_weights_id - (weight_y * window_width);
		int input_feature_map_id = feature_map_pair_id - (output_feature_map_id * input_feature_map_count);

		const float * current_output_errors = output_errors + ((entry_id * output_feature_map_count + output_feature_map_id) * output_height) * output_width;
		int input_elem_id = (((different_input ? entry_id * input_feature_map_count : 0) + input_feature_map_id) * input_height + weight_y) * input_width + weight_x + texture_offset;

		float sum = 0.0F;
		for(int y = 0; y < output_height; ++y)
		{
			for(int x = 0; x < output_width; ++x)
			{
				float inp = tex1Dfetch(input_tex_ref, input_elem_id);
				sum += *current_output_errors * inp;
				current_output_errors++;
				input_elem_id++;
			}
			input_elem_id += (window_width - 1);
		}

		int offset = (((entry_id * output_feature_map_count + output_feature_map_id) * input_feature_map_count + input_feature_map_id) * window_height + weight_y) * window_width + weight_x;
		weights[offset] += sum * training_speed[offset];
	}
}

template<int WINDOW_WIDTH, bool different_input>
__global__ void convolution_2d_update_weights_exact_blocked_upd_kernel(
	float * __restrict weights,
	const float * __restrict output_errors,
	const float * __restrict training_speed,
	int output_width,
	int output_height,
	int input_width,
	int input_height,
	int window_height,
	int input_feature_map_count,
	int output_feature_map_count,
	int output_y_group_count,
	int texture_offset,
	int entry_count)
{
	int dd = blockIdx.x * blockDim.x + threadIdx.x;
	int output_y_group_id = dd / window_height;
	int feature_map_pair_id = blockIdx.y * blockDim.y + threadIdx.y;
	int output_feature_map_id = feature_map_pair_id / input_feature_map_count;
	int entry_id = blockIdx.z * blockDim.z + threadIdx.z;

	if ((output_y_group_id < output_y_group_count) && (output_feature_map_id < output_feature_map_count) && (entry_id < entry_count))
	{
		int weight_y = dd - (output_y_group_id * window_height);
		int input_feature_map_id = feature_map_pair_id - (output_feature_map_id * input_feature_map_count);

		const float * current_output_errors = output_errors + ((entry_id * output_feature_map_count + output_feature_map_id) * output_height + output_y_group_id) * output_width;
		int input_elem_id = (((different_input ? entry_id * input_feature_map_count : 0) + input_feature_map_id) * input_height + weight_y + output_y_group_id) * input_width + texture_offset;

		float sums[WINDOW_WIDTH];
		#pragma unroll
		for(int i = 0; i < WINDOW_WIDTH; ++i)
			sums[i] = 0.0F;

		for(int output_y = output_y_group_id; output_y < output_height; output_y += output_y_group_count)
		{
			float input_buf[WINDOW_WIDTH];
			#pragma unroll
			for(int i = 1; i < WINDOW_WIDTH; ++i)
			{
				input_buf[i] = tex1Dfetch(input_tex_ref, input_elem_id);
				++input_elem_id;
			}

			for(int x = 0; x < output_width; ++x)
			{
				float current_output_error = *current_output_errors;

				#pragma unroll
				for(int i = 0; i < WINDOW_WIDTH - 1; ++i)
					input_buf[i] = input_buf[i + 1];
				input_buf[WINDOW_WIDTH - 1] = tex1Dfetch(input_tex_ref, input_elem_id);

				#pragma unroll
				for(int i = 0; i < WINDOW_WIDTH; ++i)
					sums[i] += current_output_error * input_buf[i];

				current_output_errors++;
				input_elem_id++;
			}

			current_output_errors += output_width * (output_y_group_count - 1);
			input_elem_id += input_width * (output_y_group_count - 1);
		}

		int offset = (((entry_id * output_feature_map_count + output_feature_map_id) * input_feature_map_count + input_feature_map_id) * window_height + weight_y) * WINDOW_WIDTH;
		float * cur_weights = weights + offset;
		const float * cur_training_speed = training_speed + offset;
		if (output_y_group_count == 1)
		{
			#pragma unroll
			for(int i = 0; i < WINDOW_WIDTH; ++i)
				cur_weights[i] += sums[i] * cur_training_speed[i];
		}
		else
		{
			#pragma unroll
			for(int i = 0; i < WINDOW_WIDTH; ++i)
				atomicAdd(cur_weights + i, sums[i] * cur_training_speed[i]);
		}
	}
}

namespace nnforge
{
	namespace cuda
	{
		convolution_2d_layer_updater_cuda::convolution_2d_layer_updater_cuda()
		{
			input_tex_ref.addressMode[0] = hipAddressModeBorder;
			input_tex_ref.normalized = false;
			output_tex_ref.addressMode[0] = hipAddressModeBorder;
			output_tex_ref.normalized = false;
			input_tex_ref.addressMode[0] = hipAddressModeBorder;
			input_tex_ref.normalized = false;
		}

		convolution_2d_layer_updater_cuda::~convolution_2d_layer_updater_cuda()
		{
		}

#define MAX_BLOCK_SIZE 5
#define MAX_WINDOW_WIDTH 10

#define launch_exact_block_kernel_const_const(window_width_const, block_size_const, different_input_const) \
	convolution_2d_tex_exact_blocked_upd_kernel<window_width_const,block_size_const,different_input_const><<<kernel_dims.first, kernel_dims.second, 0, stream_id>>>(*output_neurons_buffer, *data[0], *data[1], output_configuration_specific.dimension_sizes[0], output_configuration_specific.dimension_sizes[1], input_configuration_specific.dimension_sizes[0], input_configuration_specific.dimension_sizes[1], window_sizes[1], input_configuration_specific.feature_map_count, output_configuration_specific.feature_map_count, block_count, input_feature_map_group_count, input_feature_map_group_size, texture_offset, entry_count);

#define launch_exact_block_kernel_const(window_width, block_size_const, different_input_const) \
	switch (window_width) \
		{ \
		case 1: \
			launch_exact_block_kernel_const_const(1, block_size_const, different_input_const); \
			break; \
		case 2: \
			launch_exact_block_kernel_const_const(2, block_size_const, different_input_const); \
			break; \
		case 3: \
			launch_exact_block_kernel_const_const(3, block_size_const, different_input_const); \
			break; \
		case 4: \
			launch_exact_block_kernel_const_const(4, block_size_const, different_input_const); \
			break; \
		case 5: \
			launch_exact_block_kernel_const_const(5, block_size_const, different_input_const); \
			break; \
		case 6: \
			launch_exact_block_kernel_const_const(6, block_size_const, different_input_const); \
			break; \
		case 7: \
			launch_exact_block_kernel_const_const(7, block_size_const, different_input_const); \
			break; \
		case 8: \
			launch_exact_block_kernel_const_const(8, block_size_const, different_input_const); \
			break; \
		case 9: \
			launch_exact_block_kernel_const_const(9, block_size_const, different_input_const); \
			break; \
		case 10: \
			launch_exact_block_kernel_const_const(10, block_size_const, different_input_const); \
			break; \
		};

#define launch_exact_block_kernel(window_width, block_size, different_input_const) \
	switch (block_size) \
		{ \
		case 1: \
			launch_exact_block_kernel_const(window_width, 1, different_input_const); \
			break; \
		case 2: \
			launch_exact_block_kernel_const(window_width, 2, different_input_const); \
			break; \
		case 3: \
			launch_exact_block_kernel_const(window_width, 3, different_input_const); \
			break; \
		case 4: \
			launch_exact_block_kernel_const(window_width, 4, different_input_const); \
			break; \
		case 5: \
			launch_exact_block_kernel_const(window_width, 5, different_input_const); \
			break; \
		};

#define launch_backprop_exact_block_kernel_const_const(window_width_const, block_size_const) \
	convolution_2d_deriviative_tex_exact_blocked_upd_kernel<window_width_const,block_size_const><<<kernel_dims.first, kernel_dims.second, 0, stream_id>>>(*input_errors_buffer, *data[0], output_configuration_specific.dimension_sizes[0], output_configuration_specific.dimension_sizes[1], input_configuration_specific.dimension_sizes[0], input_configuration_specific.dimension_sizes[1], window_sizes[1], input_configuration_specific.feature_map_count, output_configuration_specific.feature_map_count, block_count, output_feature_map_group_count, output_feature_map_group_size, entry_count);

#define launch_backprop_exact_block_kernel_const(window_width, block_size_const) \
	switch (window_width) \
		{ \
		case 1: \
			launch_backprop_exact_block_kernel_const_const(1, block_size_const); \
			break; \
		case 2: \
			launch_backprop_exact_block_kernel_const_const(2, block_size_const); \
			break; \
		case 3: \
			launch_backprop_exact_block_kernel_const_const(3, block_size_const); \
			break; \
		case 4: \
			launch_backprop_exact_block_kernel_const_const(4, block_size_const); \
			break; \
		case 5: \
			launch_backprop_exact_block_kernel_const_const(5, block_size_const); \
			break; \
		case 6: \
			launch_backprop_exact_block_kernel_const_const(6, block_size_const); \
			break; \
		case 7: \
			launch_backprop_exact_block_kernel_const_const(7, block_size_const); \
			break; \
		case 8: \
			launch_backprop_exact_block_kernel_const_const(8, block_size_const); \
			break; \
		case 9: \
			launch_backprop_exact_block_kernel_const_const(9, block_size_const); \
			break; \
		case 10: \
			launch_backprop_exact_block_kernel_const_const(10, block_size_const); \
			break; \
		};

#define launch_backprop_exact_block_kernel(window_width, block_size) \
	switch (block_size) \
		{ \
		case 1: \
			launch_backprop_exact_block_kernel_const(window_width, 1); \
			break; \
		case 2: \
			launch_backprop_exact_block_kernel_const(window_width, 2); \
			break; \
		case 3: \
			launch_backprop_exact_block_kernel_const(window_width, 3); \
			break; \
		case 4: \
			launch_backprop_exact_block_kernel_const(window_width, 4); \
			break; \
		case 5: \
			launch_backprop_exact_block_kernel_const(window_width, 5); \
			break; \
		};

#define launch_update_weights_exact_block_kernel_const(window_width_const, different_input_const) \
	convolution_2d_update_weights_exact_blocked_upd_kernel<window_width_const, different_input_const><<<kernel_dims.first, kernel_dims.second, 0, stream_id>>>(*data[0], *output_errors_buffer, *training_speed[0], output_configuration_specific.dimension_sizes[0], output_configuration_specific.dimension_sizes[1], input_configuration_specific.dimension_sizes[0], input_configuration_specific.dimension_sizes[1], window_sizes[1], input_configuration_specific.feature_map_count, output_configuration_specific.feature_map_count, output_y_group_count, texture_offset, entry_count);

#define launch_update_weights_exact_block_kernel(window_width, different_input_const) \
	switch (window_width) \
		{ \
		case 1: \
			launch_update_weights_exact_block_kernel_const(1, different_input_const); \
			break; \
		case 2: \
			launch_update_weights_exact_block_kernel_const(2, different_input_const); \
			break; \
		case 3: \
			launch_update_weights_exact_block_kernel_const(3, different_input_const); \
			break; \
		case 4: \
			launch_update_weights_exact_block_kernel_const(4, different_input_const); \
			break; \
		case 5: \
			launch_update_weights_exact_block_kernel_const(5, different_input_const); \
			break; \
		case 6: \
			launch_update_weights_exact_block_kernel_const(6, different_input_const); \
			break; \
		case 7: \
			launch_update_weights_exact_block_kernel_const(7, different_input_const); \
			break; \
		case 8: \
			launch_update_weights_exact_block_kernel_const(8, different_input_const); \
			break; \
		case 9: \
			launch_update_weights_exact_block_kernel_const(9, different_input_const); \
			break; \
		case 10: \
			launch_update_weights_exact_block_kernel_const(10, different_input_const); \
			break; \
		};

		void convolution_2d_layer_updater_cuda::enqueue_test(
			unsigned int offset_input_entry_id,
			hipStream_t stream_id,
			const std::vector<const_cuda_linear_buffer_device_smart_ptr>& schema_data,
			const std::vector<cuda_linear_buffer_device_smart_ptr>& data,
			const_cuda_linear_buffer_device_smart_ptr input_neurons_buffer,
			cuda_linear_buffer_device_smart_ptr output_neurons_buffer,
			const std::vector<cuda_linear_buffer_device_smart_ptr>& additional_buffers,
			unsigned int entry_count)
		{
			hipChannelFormatDesc desc = hipCreateChannelDesc<float>();
			size_t texture_offset;
			cuda_safe_call(hipBindTexture(&texture_offset, input_tex_ref, (const float *)(*input_neurons_buffer) + (offset_input_entry_id * input_elem_count_per_entry), desc, input_elem_count_per_entry * sizeof(float) * (different_input ? entry_count : 1)));
			texture_offset /= sizeof(float);

			if (window_sizes[0] <= MAX_WINDOW_WIDTH)
			{
				int block_size = get_block_size(output_configuration_specific.dimension_sizes[0]);
				int block_count = (output_configuration_specific.dimension_sizes[0] + block_size - 1) / block_size;
				int input_feature_map_group_count = cuda_util::get_group_count(
					*cuda_config,
					block_count * output_configuration_specific.dimension_sizes[1] * output_configuration_specific.feature_map_count * entry_count,
					input_configuration_specific.feature_map_count);
				int input_feature_map_group_size = (input_configuration_specific.feature_map_count + input_feature_map_group_count - 1) / input_feature_map_group_count;

				if (input_feature_map_group_count > 1)
					cuda_util::set_with_value(
						*cuda_config,
						*output_neurons_buffer,
						0.0F,
						output_elem_count_per_entry * entry_count,
						stream_id);

				std::pair<dim3, dim3> kernel_dims = cuda_util::get_grid_and_threadblock_sizes_sequential_access(
					*cuda_config,
					block_count * output_configuration_specific.dimension_sizes[1],
					output_configuration_specific.feature_map_count * input_feature_map_group_count,
					entry_count);

				if (different_input)
				{
					launch_exact_block_kernel(window_sizes[0], block_size, true);
				}
				else
				{
					launch_exact_block_kernel(window_sizes[0], block_size, false);
				}
			}
			else
			{
				std::pair<dim3, dim3> kernel_dims = cuda_util::get_grid_and_threadblock_sizes_2d_access(
					*cuda_config,
					output_configuration_specific.dimension_sizes[0],
					output_configuration_specific.dimension_sizes[1] * output_configuration_specific.feature_map_count,
					entry_count);

				if (different_input)
					convolution_2d_tex_upd_kernel<true><<<kernel_dims.first, kernel_dims.second, 0, stream_id>>>(
						*output_neurons_buffer,
						*data[0],
						*data[1],
						output_configuration_specific.dimension_sizes[0],
						output_configuration_specific.dimension_sizes[1],
						input_configuration_specific.dimension_sizes[0],
						input_configuration_specific.dimension_sizes[1],
						window_sizes[0],
						window_sizes[1],
						input_configuration_specific.feature_map_count,
						output_configuration_specific.feature_map_count,
						texture_offset,
						entry_count);
				else
					convolution_2d_tex_upd_kernel<false><<<kernel_dims.first, kernel_dims.second, 0, stream_id>>>(
						*output_neurons_buffer,
						*data[0],
						*data[1],
						output_configuration_specific.dimension_sizes[0],
						output_configuration_specific.dimension_sizes[1],
						input_configuration_specific.dimension_sizes[0],
						input_configuration_specific.dimension_sizes[1],
						window_sizes[0],
						window_sizes[1],
						input_configuration_specific.feature_map_count,
						output_configuration_specific.feature_map_count,
						texture_offset,
						entry_count);
			}
		}

		void convolution_2d_layer_updater_cuda::enqueue_backprop(
			hipStream_t stream_id,
			const std::vector<const_cuda_linear_buffer_device_smart_ptr>& schema_data,
			const std::vector<cuda_linear_buffer_device_smart_ptr>& data,
			const_cuda_linear_buffer_device_smart_ptr output_neurons_buffer,
			const_cuda_linear_buffer_device_smart_ptr input_neurons_buffer,
			cuda_linear_buffer_device_smart_ptr output_errors_buffer,
			cuda_linear_buffer_device_smart_ptr input_errors_buffer,
			const std::vector<cuda_linear_buffer_device_smart_ptr>& additional_buffers,
			unsigned int entry_count)
		{
			if (!different_input)
				throw neural_network_exception("convolution_2d_layer_updater_cuda is not able to backprop to the same input");

			hipChannelFormatDesc desc = hipCreateChannelDesc<float>();
			cuda_safe_call(hipBindTexture(0, output_tex_ref, *output_errors_buffer, desc, output_elem_count_per_entry * entry_count * sizeof(float)));

			if (window_sizes[0] <= MAX_WINDOW_WIDTH)
			{
				int block_size = get_block_size(input_configuration_specific.dimension_sizes[0]);
				int block_count = (input_configuration_specific.dimension_sizes[0] + block_size - 1) / block_size;
				int output_feature_map_group_count = cuda_util::get_group_count(
					*cuda_config,
					block_count * input_configuration_specific.dimension_sizes[1] * input_configuration_specific.feature_map_count * entry_count,
					output_configuration_specific.feature_map_count);
				int output_feature_map_group_size = (output_configuration_specific.feature_map_count + output_feature_map_group_count - 1) / output_feature_map_group_count;

				if (output_feature_map_group_count > 1)
					cuda_util::set_with_value(
						*cuda_config,
						*input_errors_buffer,
						0.0F,
						input_elem_count_per_entry * entry_count,
						stream_id);

				std::pair<dim3, dim3> kernel_dims = cuda_util::get_grid_and_threadblock_sizes_sequential_access(
					*cuda_config,
					block_count * input_configuration_specific.dimension_sizes[1],
					input_configuration_specific.feature_map_count * output_feature_map_group_count,
					entry_count);
				launch_backprop_exact_block_kernel(window_sizes[0], block_size);
			}
			else
			{
				std::pair<dim3, dim3> kernel_dims = cuda_util::get_grid_and_threadblock_sizes_2d_access(
					*cuda_config,
					input_configuration_specific.dimension_sizes[0],
					input_configuration_specific.dimension_sizes[1] * input_configuration_specific.feature_map_count,
					entry_count);

				convolution_2d_deriviative_tex_upd_kernel<<<kernel_dims.first, kernel_dims.second, 0, stream_id>>>(
					*input_errors_buffer,
					*data[0],
					output_configuration_specific.dimension_sizes[0],
					output_configuration_specific.dimension_sizes[1],
					input_configuration_specific.dimension_sizes[0],
					input_configuration_specific.dimension_sizes[1],
					window_sizes[0],
					window_sizes[1],
					input_configuration_specific.feature_map_count,
					output_configuration_specific.feature_map_count,
					entry_count);
			}
		}

		void convolution_2d_layer_updater_cuda::enqueue_update_weights(
			unsigned int offset_input_entry_id,
			hipStream_t stream_id,
			const std::vector<cuda_linear_buffer_device_smart_ptr>& data,
			const std::vector<const_cuda_linear_buffer_device_smart_ptr>& schema_data,
			const std::vector<const_cuda_linear_buffer_device_smart_ptr>& training_speed,
			cuda_linear_buffer_device_smart_ptr output_errors_buffer,
			const_cuda_linear_buffer_device_smart_ptr input_neurons_buffer,
			const std::vector<cuda_linear_buffer_device_smart_ptr>& additional_buffers,
			unsigned int entry_count)
		{
			// Update biases
			{
				int threadblock_size = get_threadblock_size_biases(output_elem_count_per_feature_map);
				dim3 grid_size(1, output_configuration_specific.feature_map_count, entry_count);
				dim3 block_size(threadblock_size, 1, 1);
				int smem_size = threadblock_size * sizeof(float);
				int min_iteration_count = output_elem_count_per_feature_map / threadblock_size;

				convolution_2d_update_biases_upd_kernel<<<grid_size, block_size, smem_size, stream_id>>>(
					*data[1],
					*output_errors_buffer,
					*training_speed[1],
					output_configuration_specific.feature_map_count,
					output_elem_count_per_feature_map,
					min_iteration_count);
			}

			hipChannelFormatDesc desc = hipCreateChannelDesc<float>();
			size_t texture_offset;
			cuda_safe_call(hipBindTexture(&texture_offset, input_tex_ref, (const float *)(*input_neurons_buffer) + (offset_input_entry_id * input_elem_count_per_entry), desc, input_elem_count_per_entry * sizeof(float) * (different_input ? entry_count : 1)));
			texture_offset /= sizeof(float);

			// Update weights
			{
				if (window_sizes[0] <= MAX_WINDOW_WIDTH)
				{
					int output_y_group_count = cuda_util::get_group_count(
						*cuda_config,
						output_configuration_specific.feature_map_count * input_configuration_specific.feature_map_count * window_sizes[1] * entry_count,
						output_configuration_specific.dimension_sizes[1]);
					std::pair<dim3, dim3> kernel_dims = cuda_util::get_grid_and_threadblock_sizes_sequential_access(
						*cuda_config,
						window_sizes[1] * output_y_group_count,
						output_configuration_specific.feature_map_count * input_configuration_specific.feature_map_count,
						entry_count);

					if (different_input)
					{
						launch_update_weights_exact_block_kernel(window_sizes[0], true);
					}
					else
					{
						launch_update_weights_exact_block_kernel(window_sizes[0], false);
					}
				}
				else
				{
					std::pair<dim3, dim3> kernel_dims = cuda_util::get_grid_and_threadblock_sizes_sequential_access(
						*cuda_config,
						window_sizes[0] * window_sizes[1],
						output_configuration_specific.feature_map_count * input_configuration_specific.feature_map_count,
						entry_count);

					if (different_input)
						convolution_2d_update_weights_upd_kernel<true><<<kernel_dims.first, kernel_dims.second, 0, stream_id>>>(
							*data[0],
							*output_errors_buffer,
							*training_speed[0],
							output_configuration_specific.dimension_sizes[0],
							output_configuration_specific.dimension_sizes[1],
							input_configuration_specific.dimension_sizes[0],
							input_configuration_specific.dimension_sizes[1],
							window_sizes[0],
							window_sizes[1],
							input_configuration_specific.feature_map_count,
							output_configuration_specific.feature_map_count,
							texture_offset,
							entry_count);
					else
						convolution_2d_update_weights_upd_kernel<false><<<kernel_dims.first, kernel_dims.second, 0, stream_id>>>(
							*data[0],
							*output_errors_buffer,
							*training_speed[0],
							output_configuration_specific.dimension_sizes[0],
							output_configuration_specific.dimension_sizes[1],
							input_configuration_specific.dimension_sizes[0],
							input_configuration_specific.dimension_sizes[1],
							window_sizes[0],
							window_sizes[1],
							input_configuration_specific.feature_map_count,
							output_configuration_specific.feature_map_count,
							texture_offset,
							entry_count);
				}
			}
		}

		int convolution_2d_layer_updater_cuda::get_block_size(int width)
		{
			int block_count = (width + MAX_BLOCK_SIZE - 1) / MAX_BLOCK_SIZE;
			int block_size = (width + block_count - 1) / block_count;
			return block_size;
		}

		void convolution_2d_layer_updater_cuda::updater_configured()
		{
			std::tr1::shared_ptr<const convolution_layer> layer_derived = std::tr1::dynamic_pointer_cast<const convolution_layer>(layer_schema);

			for(std::vector<unsigned int>::const_iterator it = layer_derived->window_sizes.begin(); it != layer_derived->window_sizes.end(); ++it)
				window_sizes.push_back(static_cast<int>(*it));
		}

		bool convolution_2d_layer_updater_cuda::is_in_place_backprop() const
		{
			return false;
		}

		std::vector<unsigned int> convolution_2d_layer_updater_cuda::get_linear_addressing_through_texture_per_entry() const
		{
			std::vector<unsigned int> res;

			res.push_back(input_elem_count_per_entry);
			res.push_back(output_elem_count_per_entry);

			return res;
		}

		int convolution_2d_layer_updater_cuda::get_threadblock_size_biases(int output_neuron_count)
		{
			if (output_neuron_count < 256)
				return output_neuron_count;

			int threadblock_count = (output_neuron_count + 256 - 1) / 256;
			int threadblock_size = (output_neuron_count + threadblock_count - 1) / threadblock_count;
			threadblock_size = (threadblock_size + 32 - 1) / 32 * 32;

			return threadblock_size;
		}
	}
}
