#include "hip/hip_runtime.h"
/*
 *  Copyright 2011-2013 Maxim Milakov
 *
 *  Licensed under the Apache License, Version 2.0 (the "License");
 *  you may not use this file except in compliance with the License.
 *  You may obtain a copy of the License at
 *
 *      http://www.apache.org/licenses/LICENSE-2.0
 *
 *  Unless required by applicable law or agreed to in writing, software
 *  distributed under the License is distributed on an "AS IS" BASIS,
 *  WITHOUT WARRANTIES OR CONDITIONS OF ANY KIND, either express or implied.
 *  See the License for the specific language governing permissions and
 *  limitations under the License.
 */

#include "rgb_to_yuv_convert_layer_tester_cuda.h"

#include <hip/hip_runtime.h>

#include "util_cuda.h"

#include "../rgb_to_yuv_convert_layer.h"

#define w_r 0.299F
#define w_b 0.114F
#define w_g (1.0F - w_r - w_b)
#define u_max 0.436
#define v_max 0.615F
#define u_mult (u_max / (1.0F - w_b))
#define v_mult (v_max / (1.0F - w_r))

__global__ void rgb_to_yuv_convert_kernel(
	float * __restrict input,
	const int * __restrict color_feature_map_config_list,
	int feature_map_count,
	int elem_count_per_feature_map,
	int color_feature_map_config_count,
	int entry_count)
{
	int elem_id = blockDim.x * blockIdx.x + threadIdx.x;
	int color_feature_map_config_config_id = blockDim.y * blockIdx.y + threadIdx.y;
	int entry_id = blockDim.z * blockIdx.z + threadIdx.z;
	if ((elem_id < elem_count_per_feature_map) && (color_feature_map_config_config_id < color_feature_map_config_count) && (entry_id < entry_count))
	{
		int color_feature_map_config_id_offset = color_feature_map_config_config_id * 3;
		int red_and_y_feature_map_id = color_feature_map_config_list[color_feature_map_config_id_offset];
		int green_and_u_feature_map_id = color_feature_map_config_list[color_feature_map_config_id_offset + 1];
		int blue_and_v_feature_map_id = color_feature_map_config_list[color_feature_map_config_id_offset + 2];

		int base_offset = (entry_id * elem_count_per_feature_map * feature_map_count) + elem_id;
		int red_and_y_offset = red_and_y_feature_map_id * elem_count_per_feature_map + base_offset;
		int green_and_u_offset = green_and_u_feature_map_id * elem_count_per_feature_map + base_offset;
		int blue_and_v_offset = blue_and_v_feature_map_id * elem_count_per_feature_map + base_offset;

		float red = input[red_and_y_offset];
		float green = input[green_and_u_offset];
		float blue = input[blue_and_v_offset];

		float y = w_r * red + w_g * green + w_b * blue;
		float u = u_mult * (blue - y);
		float v = v_mult * (red - y);

		input[red_and_y_offset] = y;
		input[green_and_u_offset] = u;
		input[blue_and_v_offset] = v;
	}
}

namespace nnforge
{
	namespace cuda
	{
		rgb_to_yuv_convert_layer_tester_cuda::rgb_to_yuv_convert_layer_tester_cuda()
		{
		}

		rgb_to_yuv_convert_layer_tester_cuda::~rgb_to_yuv_convert_layer_tester_cuda()
		{
		}

		void rgb_to_yuv_convert_layer_tester_cuda::enqueue_test(
			hipStream_t stream_id,
			const std::vector<const_cuda_linear_buffer_device_smart_ptr>& schema_data,
			const std::vector<const_cuda_linear_buffer_device_smart_ptr>& data,
			cuda_linear_buffer_device_smart_ptr input_buffer,
			const std::vector<cuda_linear_buffer_device_smart_ptr>& additional_buffers,
			unsigned int entry_count)
		{
			std::pair<dim3, dim3> kernel_dims = cuda_util::get_grid_and_threadblock_sizes_sequential_access(
				*cuda_config,
				input_elem_count_per_feature_map,
				color_feature_map_config_count,
				entry_count);
			rgb_to_yuv_convert_kernel<<<kernel_dims.first, kernel_dims.second, 0, stream_id>>>(
				*input_buffer,
				*schema_data[0],
				input_configuration_specific.feature_map_count,
				input_elem_count_per_feature_map,
				color_feature_map_config_count,
				entry_count);
		}

		void rgb_to_yuv_convert_layer_tester_cuda::tester_configured()
		{
			std::tr1::shared_ptr<const rgb_to_yuv_convert_layer> layer_derived = std::tr1::dynamic_pointer_cast<const rgb_to_yuv_convert_layer>(layer_schema);

			color_feature_map_config_count = layer_derived->color_feature_map_config_list.size();
		}
	}
}
