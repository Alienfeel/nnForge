#include "hip/hip_runtime.h"
/*
 *  Copyright 2011-2014 Maxim Milakov
 *
 *  Licensed under the Apache License, Version 2.0 (the "License");
 *  you may not use this file except in compliance with the License.
 *  You may obtain a copy of the License at
 *
 *      http://www.apache.org/licenses/LICENSE-2.0
 *
 *  Unless required by applicable law or agreed to in writing, software
 *  distributed under the License is distributed on an "AS IS" BASIS,
 *  WITHOUT WARRANTIES OR CONDITIONS OF ANY KIND, either express or implied.
 *  See the License for the specific language governing permissions and
 *  limitations under the License.
 */

#include "convolution_1d_layer_hessian_cuda_fermi.h"

#include <hip/hip_runtime.h>

#include <boost/format.hpp>

#include "util_cuda.h"
#include "neural_network_cuda_exception.h"

#include "../convolution_layer.h"
#include "../nn_types.h"

texture<float, hipTextureType1D, hipReadModeElementType> input_tex_ref;
texture<float, hipTextureType1D, hipReadModeElementType> output_tex_ref;
texture<float, hipTextureType1D, hipReadModeElementType> input_squared_tex_ref;

#define FEATURE_MAP_BLOCK_SIZE 4
#define WINDOW_WIDTH_LOCAL 4

namespace nnforge
{
	namespace cuda
	{
		template<int BLOCK_SIZE>
		__global__ void convolution_1d_tex_blocked_hess_kernel_fermi(
			float * __restrict output,
			const float * __restrict weights,
			const float * __restrict biases,
			int output_width,
			int input_width,
			int window_width,
			int input_feature_map_count,
			int output_feature_map_count,
			int entry_count)
		{
			int x = (blockIdx.x * blockDim.x + threadIdx.x) * BLOCK_SIZE;
			int output_feature_map_id = (blockIdx.y * blockDim.y + threadIdx.y) * FEATURE_MAP_BLOCK_SIZE;
			int entry_id = blockIdx.z * blockDim.z + threadIdx.z;

			bool in_bounds = (entry_id < entry_count) && (x < output_width) && (output_feature_map_id < output_feature_map_count);
			if (in_bounds)
			{
				int weight_count_per_output_feature_map = window_width * input_feature_map_count;
				int input_elem_id = entry_id * input_feature_map_count * input_width + x;
				const float * current_weights = weights + (int)(weight_count_per_output_feature_map * output_feature_map_id);

				float bias_list[FEATURE_MAP_BLOCK_SIZE];
				#pragma unroll
				for(int i = 0; i < FEATURE_MAP_BLOCK_SIZE; ++i)
					if (i < output_feature_map_count - output_feature_map_id)
						bias_list[i] = biases[output_feature_map_id + i];
				float sums[BLOCK_SIZE * FEATURE_MAP_BLOCK_SIZE];
				#pragma unroll
				for(int i = 0; i < FEATURE_MAP_BLOCK_SIZE; ++i)
					#pragma unroll
					for(int j = 0; j < BLOCK_SIZE; ++j)
						sums[i * BLOCK_SIZE + j] = bias_list[i];
				int weight_offsets[FEATURE_MAP_BLOCK_SIZE];
				#pragma unroll
				for(int i = 0; i < FEATURE_MAP_BLOCK_SIZE; ++i)
					weight_offsets[i] = (i < output_feature_map_count - output_feature_map_id) ? weight_count_per_output_feature_map * i : 0;

				for(int input_layer_id = 0; input_layer_id < input_feature_map_count; ++input_layer_id)
				{
					#pragma unroll 4
					for(int input_x = 0; input_x < window_width; ++input_x)
					{
						float weight_list[FEATURE_MAP_BLOCK_SIZE];
						#pragma unroll
						for(int i = 0; i < FEATURE_MAP_BLOCK_SIZE; ++i)
							weight_list[i] = current_weights[weight_offsets[i]];
						#pragma unroll
						for(int j = 0; j < BLOCK_SIZE; ++j)
						{
							float inp = tex1Dfetch(input_tex_ref, input_elem_id + j); 
							#pragma unroll
							for(int i = 0; i < FEATURE_MAP_BLOCK_SIZE; ++i)
								sums[i * BLOCK_SIZE + j] += inp * weight_list[i];
						}
						current_weights++;
						input_elem_id++;
					}
					input_elem_id += input_width - window_width;
				}

				float * base_output = output + (entry_id * output_feature_map_count + output_feature_map_id) * output_width + x;
				#pragma unroll
				for(int i = 0; i < FEATURE_MAP_BLOCK_SIZE; ++i)
				{
					if (i < output_feature_map_count - output_feature_map_id)
					{
						#pragma unroll
						for(int j = 0; j < BLOCK_SIZE; ++j)
						{
							if (j < output_width - x)
								base_output[j + output_width * i] = sums[i * BLOCK_SIZE + j];
						}
					}
				}
			}
		}

		template<int WINDOW_WIDTH, int BLOCK_SIZE>
		__global__ void convolution_1d_tex_exact_blocked_hess_kernel_fermi(
			float * __restrict output,
			const float * __restrict weights,
			const float * __restrict biases,
			int output_width,
			int input_width,
			int input_feature_map_count,
			int output_feature_map_count,
			int entry_count)
		{
			int x = (blockIdx.x * blockDim.x + threadIdx.x) * BLOCK_SIZE;
			int output_feature_map_id = (blockIdx.y * blockDim.y + threadIdx.y) * FEATURE_MAP_BLOCK_SIZE;
			int entry_id = blockIdx.z * blockDim.z + threadIdx.z;

			bool in_bounds = (entry_id < entry_count) && (x < output_width) && (output_feature_map_id < output_feature_map_count);
			if (in_bounds)
			{
				int weight_count_per_output_feature_map = WINDOW_WIDTH * input_feature_map_count;
				int input_elem_id = entry_id * input_feature_map_count * input_width + x;
				const float * current_weights = weights + (int)(weight_count_per_output_feature_map * output_feature_map_id);

				float bias_list[FEATURE_MAP_BLOCK_SIZE];
				#pragma unroll
				for(int i = 0; i < FEATURE_MAP_BLOCK_SIZE; ++i)
					if (i < output_feature_map_count - output_feature_map_id)
						bias_list[i] = biases[output_feature_map_id + i];
				float sums[BLOCK_SIZE * FEATURE_MAP_BLOCK_SIZE];
				#pragma unroll
				for(int i = 0; i < FEATURE_MAP_BLOCK_SIZE; ++i)
					#pragma unroll
					for(int j = 0; j < BLOCK_SIZE; ++j)
						sums[i * BLOCK_SIZE + j] = bias_list[i];
				int weight_offsets[FEATURE_MAP_BLOCK_SIZE];
				#pragma unroll
				for(int i = 0; i < FEATURE_MAP_BLOCK_SIZE; ++i)
					weight_offsets[i] = (i < output_feature_map_count - output_feature_map_id) ? weight_count_per_output_feature_map * i : 0;

				for(int input_layer_id = 0; input_layer_id < input_feature_map_count; ++input_layer_id)
				{
					#pragma unroll
					for(int input_x = 0; input_x < WINDOW_WIDTH; ++input_x)
					{
						float weight_list[FEATURE_MAP_BLOCK_SIZE];
						#pragma unroll
						for(int i = 0; i < FEATURE_MAP_BLOCK_SIZE; ++i)
							weight_list[i] = current_weights[weight_offsets[i]];
						#pragma unroll
						for(int j = 0; j < BLOCK_SIZE; ++j)
						{
							float inp = tex1Dfetch(input_tex_ref, input_elem_id + j); 
							#pragma unroll
							for(int i = 0; i < FEATURE_MAP_BLOCK_SIZE; ++i)
								sums[i * BLOCK_SIZE + j] += inp * weight_list[i];
						}
						current_weights++;
						input_elem_id++;
					}
					input_elem_id += input_width - WINDOW_WIDTH;
				}

				float * base_output = output + (entry_id * output_feature_map_count + output_feature_map_id) * output_width + x;
				#pragma unroll
				for(int i = 0; i < FEATURE_MAP_BLOCK_SIZE; ++i)
				{
					if (i < output_feature_map_count - output_feature_map_id)
					{
						#pragma unroll
						for(int j = 0; j < BLOCK_SIZE; ++j)
						{
							if (j < output_width - x)
								base_output[j + output_width * i] = sums[i * BLOCK_SIZE + j];
						}
					}
				}
			}
		}

		extern __shared__ float arr[];
		__global__ void convolution_1d_update_biases_hess_kernel_fermi(
			float * __restrict hessian_biases,
			const float * __restrict output_errors,
			int block_size,
			int output_elem_count_per_feature_map,
			int output_feature_map_count,
			int entry_count)
		{
			int output_neuron_id = blockIdx.x * blockDim.x + threadIdx.x;
			int output_feature_map_id = blockIdx.y;
			int block_id = blockIdx.z * blockDim.z + threadIdx.z;
			int base_entry_id = block_size * block_id;
			int thread_id = blockDim.x * threadIdx.z + threadIdx.x;
			int threadblock_size = blockDim.x * blockDim.z;
			float sum = 0.0F;
			int iteration_count = min(entry_count - base_entry_id, block_size);
			if (output_neuron_id < output_elem_count_per_feature_map)
			{
				const float * current_error = output_errors + (base_entry_id * output_feature_map_count + output_feature_map_id) * output_elem_count_per_feature_map + output_neuron_id;
				int output_elem_count_per_entry = output_elem_count_per_feature_map * output_feature_map_count;
				for(int i = 0; i < iteration_count; ++i)
				{
					sum += *current_error;
					current_error += output_elem_count_per_entry;
				}
			}
			arr[thread_id] = sum;
			__syncthreads();

			int t_add_elems = threadblock_size >> 1;
			int t_working_elems = (threadblock_size + 1) >> 1;
			while (t_add_elems > 0)
			{
				if (thread_id < t_add_elems)
					arr[thread_id] += arr[thread_id + t_working_elems];
				t_add_elems = t_working_elems >> 1;
				t_working_elems = (t_working_elems + 1) >> 1;
				__syncthreads();
			}

			if (thread_id == 0)
				atomicAdd(hessian_biases + output_feature_map_id, arr[0]);
		}

		template<int BLOCK_SIZE>
		__global__ void convolution_1d_square_deriviative_tex_hess_kernel_fermi(
			float * __restrict input_errors,
			const float * __restrict weights_squared,
			int output_width,
			int input_width,
			int window_width,
			int input_feature_map_count,
			int output_feature_map_count,
			int entry_count)
		{
			int x = (blockIdx.x * blockDim.x + threadIdx.x) * BLOCK_SIZE + (BLOCK_SIZE - 1);
			int input_feature_map_id = (blockIdx.y * blockDim.y + threadIdx.y) * FEATURE_MAP_BLOCK_SIZE;
			int entry_id = blockIdx.z * blockDim.z + threadIdx.z;

			bool in_bounds = (entry_id < entry_count) && (x < input_width + (BLOCK_SIZE - 1)) && (input_feature_map_id < input_feature_map_count);
			if (in_bounds)
			{
				int weight_count_per_input_feature_map = window_width;
				int output_elem_id = entry_id * output_feature_map_count * output_width + x;
				const float * current_weights = weights_squared + (int)(window_width * input_feature_map_id);

				float sums[FEATURE_MAP_BLOCK_SIZE * BLOCK_SIZE];
				#pragma unroll
				for(int i = 0; i < FEATURE_MAP_BLOCK_SIZE * BLOCK_SIZE; ++i)
					sums[i] = 0.0F;

				int weight_offsets[FEATURE_MAP_BLOCK_SIZE];
				#pragma unroll
				for(int i = 0; i < FEATURE_MAP_BLOCK_SIZE; ++i)
					weight_offsets[i] = (i < input_feature_map_count - input_feature_map_id) ? weight_count_per_input_feature_map * i : 0;

				int min_x_exclusive = x - output_width;
				int max_x_inclusive = x;

				for(int output_layer_id = 0; output_layer_id < output_feature_map_count; ++output_layer_id)
				{
					int input_x = 0;
					#pragma unroll 1
					for(; input_x < (window_width - (WINDOW_WIDTH_LOCAL - 1)); input_x += WINDOW_WIDTH_LOCAL)
					{
						float output_vals[BLOCK_SIZE + WINDOW_WIDTH_LOCAL - 1];
						#pragma unroll
						for(int i = 0; i < BLOCK_SIZE + WINDOW_WIDTH_LOCAL - 1; ++i)
						{
							bool b_fit2 = (i > min_x_exclusive) && (i <= max_x_inclusive);;
							if (b_fit2)
								output_vals[i] = tex1Dfetch(output_tex_ref, output_elem_id - i);
							else
								output_vals[i] = 0.0F;
						}
						output_elem_id -= WINDOW_WIDTH_LOCAL;

						#pragma unroll
						for(int input_x_local = 0; input_x_local < WINDOW_WIDTH_LOCAL; ++input_x_local)
						{
							float weight_list[FEATURE_MAP_BLOCK_SIZE];
							#pragma unroll
							for(int i = 0; i < FEATURE_MAP_BLOCK_SIZE; ++i)
								weight_list[i] = current_weights[weight_offsets[i]];

							#pragma unroll
							for(int j = 0; j < BLOCK_SIZE; ++j)
							{
								#pragma unroll
								for(int i = 0; i < FEATURE_MAP_BLOCK_SIZE; ++i)
									sums[i * BLOCK_SIZE + j] += output_vals[input_x_local + j] * weight_list[i];
							}
							current_weights++;
						}
					}
					#pragma unroll 1
					for(; input_x < window_width; ++input_x)
					{
						#pragma unroll
						for(int j = 0; j < BLOCK_SIZE; ++j)
						{
							bool b_fit2 = (input_x + j > min_x_exclusive) && (input_x + j <= max_x_inclusive);
							if (b_fit2)
							{
								float inp = tex1Dfetch(output_tex_ref, output_elem_id - j);
								#pragma unroll
								for(int i = 0; i < FEATURE_MAP_BLOCK_SIZE; ++i)
									sums[i * BLOCK_SIZE + j] += inp * current_weights[weight_offsets[i]];
							}
						}
						current_weights++;
						output_elem_id--;
					}

					current_weights += window_width * (input_feature_map_count - 1);
					output_elem_id += window_width + output_width;
				}

				float * base_input = input_errors + (entry_id * input_feature_map_count + input_feature_map_id) * input_width + x;
				#pragma unroll
				for(int i = 0; i < FEATURE_MAP_BLOCK_SIZE; ++i)
				{
					if (i < input_feature_map_count - input_feature_map_id)
					{
						#pragma unroll
						for(int j = 0; j < BLOCK_SIZE; ++j)
						{
							if (j > x - input_width)
								*(base_input + input_width * i - j) = sums[i * BLOCK_SIZE + j];
						}
					}
				}
			}
		}

		template<int WINDOW_WIDTH, int BLOCK_SIZE>
		__global__ void convolution_1d_square_deriviative_tex_exact_hess_kernel_fermi(
			float * __restrict input_errors,
			const float * __restrict weights_squared,
			int output_width,
			int input_width,
			int input_feature_map_count,
			int output_feature_map_count,
			int entry_count)
		{
			int x = (blockIdx.x * blockDim.x + threadIdx.x) * BLOCK_SIZE + (BLOCK_SIZE - 1);
			int input_feature_map_id = (blockIdx.y * blockDim.y + threadIdx.y) * FEATURE_MAP_BLOCK_SIZE;
			int entry_id = blockIdx.z * blockDim.z + threadIdx.z;

			bool in_bounds = (entry_id < entry_count) && (x < input_width + (BLOCK_SIZE - 1)) && (input_feature_map_id < input_feature_map_count);
			if (in_bounds)
			{
				int weight_count_per_input_feature_map = WINDOW_WIDTH;
				int output_elem_id = entry_id * output_feature_map_count * output_width + x;
				const float * current_weights = weights_squared + (int)(WINDOW_WIDTH * input_feature_map_id);

				float sums[FEATURE_MAP_BLOCK_SIZE * BLOCK_SIZE];
				#pragma unroll
				for(int i = 0; i < FEATURE_MAP_BLOCK_SIZE * BLOCK_SIZE; ++i)
					sums[i] = 0.0F;

				int weight_offsets[FEATURE_MAP_BLOCK_SIZE];
				#pragma unroll
				for(int i = 0; i < FEATURE_MAP_BLOCK_SIZE; ++i)
					weight_offsets[i] = (i < input_feature_map_count - input_feature_map_id) ? weight_count_per_input_feature_map * i : 0;

				int min_x_exclusive = x - output_width;
				int max_x_inclusive = x;

				unsigned int mask = 0;
				for(int i = BLOCK_SIZE + WINDOW_WIDTH - 2; i >= 0; --i)
					mask = mask << 1 | (((i > min_x_exclusive) && (i <= max_x_inclusive)) ? 1 : 0);

				for(int output_layer_id = 0; output_layer_id < output_feature_map_count; ++output_layer_id)
				{
					float output_vals[BLOCK_SIZE + WINDOW_WIDTH - 1];
					#pragma unroll
					for(int i = 0; i < BLOCK_SIZE + WINDOW_WIDTH - 1; ++i)
					{
						bool b_fit2 = (((1 << i) & mask) != 0);
						if (b_fit2)
							output_vals[i] = tex1Dfetch(output_tex_ref, output_elem_id - i);
						else
							output_vals[i] = 0.0F;
					}

					#pragma unroll
					for(int input_x = 0; input_x < WINDOW_WIDTH; ++input_x)
					{
						float weight_list[FEATURE_MAP_BLOCK_SIZE];
						#pragma unroll
						for(int i = 0; i < FEATURE_MAP_BLOCK_SIZE; ++i)
							weight_list[i] = current_weights[weight_offsets[i]];

						#pragma unroll
						for(int j = 0; j < BLOCK_SIZE; ++j)
						{
							#pragma unroll
							for(int i = 0; i < FEATURE_MAP_BLOCK_SIZE; ++i)
								sums[i * BLOCK_SIZE + j] += output_vals[input_x + j] * weight_list[i];
						}
						current_weights++;
					}
					current_weights += WINDOW_WIDTH * (input_feature_map_count - 1);
					output_elem_id += output_width;
				}

				float * base_input = input_errors + (entry_id * input_feature_map_count + input_feature_map_id) * input_width + x;
				#pragma unroll
				for(int i = 0; i < FEATURE_MAP_BLOCK_SIZE; ++i)
				{
					if (i < input_feature_map_count - input_feature_map_id)
					{
						#pragma unroll
						for(int j = 0; j < BLOCK_SIZE; ++j)
						{
							if (j > x - input_width)
								*(base_input + input_width * i - j) = sums[i * BLOCK_SIZE + j];
						}
					}
				}
			}
		}

		__global__ void convolution_1d_update_weights_hess_kernel_fermi(
			float * __restrict hessian_weights,
			const float * __restrict output_errors,
			int output_width,
			int input_width,
			int window_width,
			int input_feature_map_count,
			int output_feature_map_count,
			int entry_count,
			int window_x_block_count,
			int block_size)
		{
			int weight_x = (blockIdx.x * blockDim.x + threadIdx.x) * WINDOW_WIDTH_LOCAL;
			int feature_map_pair_id = blockIdx.y * blockDim.y + threadIdx.y;
			int output_feature_map_group_id = feature_map_pair_id / input_feature_map_count;
			int output_feature_map_id = FEATURE_MAP_BLOCK_SIZE * output_feature_map_group_id;
			int base_entry_id = (blockIdx.z * blockDim.z + threadIdx.z) * block_size;

			if ((weight_x < window_width) && (output_feature_map_id < output_feature_map_count) && (base_entry_id < entry_count))
			{
				int output_neuron_count_per_feature_map = output_width;
				int input_feature_map_id = feature_map_pair_id - (output_feature_map_group_id * input_feature_map_count);
				int iteration_count = min(block_size, entry_count - base_entry_id);

				const float * current_output_errors = output_errors + (base_entry_id * output_feature_map_count + output_feature_map_id) * output_width;
				int input_elem_id = (base_entry_id * input_feature_map_count + input_feature_map_id) * input_width + weight_x;

				float sums[FEATURE_MAP_BLOCK_SIZE * WINDOW_WIDTH_LOCAL];
				#pragma unroll
				for(int i = 0; i < FEATURE_MAP_BLOCK_SIZE * WINDOW_WIDTH_LOCAL; ++i)
					sums[i] = 0.0F;

				int output_offsets[FEATURE_MAP_BLOCK_SIZE];
				#pragma unroll
				for(int i = 0; i < FEATURE_MAP_BLOCK_SIZE; ++i)
					output_offsets[i] = (i < output_feature_map_count - output_feature_map_id) ? output_neuron_count_per_feature_map * i : 0;

				for(int t = 0; t < iteration_count; ++t)
				{
					float input_squared_buf[WINDOW_WIDTH_LOCAL];
					#pragma unroll
					for(int i = 1; i < WINDOW_WIDTH_LOCAL; ++i)
					{
						input_squared_buf[i] = tex1Dfetch(input_squared_tex_ref, input_elem_id);
						++input_elem_id;
					}

					for(int x = 0; x < output_width; ++x)
					{
						float output_error_list[FEATURE_MAP_BLOCK_SIZE];
						#pragma unroll
						for(int i = 0; i < FEATURE_MAP_BLOCK_SIZE; ++i)
							output_error_list[i] = current_output_errors[output_offsets[i]];

						#pragma unroll
						for(int i = 0; i < WINDOW_WIDTH_LOCAL - 1; ++i)
							input_squared_buf[i] = input_squared_buf[i + 1];
						input_squared_buf[WINDOW_WIDTH_LOCAL - 1] = tex1Dfetch(input_squared_tex_ref, input_elem_id);

						#pragma unroll
						for(int i = 0; i < FEATURE_MAP_BLOCK_SIZE; ++i)
							#pragma unroll
							for(int j = 0; j < WINDOW_WIDTH_LOCAL; ++j)
								sums[i * WINDOW_WIDTH_LOCAL + j] += output_error_list[i] * input_squared_buf[j];

						current_output_errors++;
						input_elem_id++;
					}
					current_output_errors += (output_feature_map_count - 1) * output_width;
					input_elem_id += (input_feature_map_count - 1) * input_width + (window_width - WINDOW_WIDTH_LOCAL);
				}

				float * base_weights = hessian_weights + (output_feature_map_id * input_feature_map_count + input_feature_map_id) * window_width + weight_x;
				int weight_count_per_output_feature_map = input_feature_map_count * window_width;
				#pragma unroll
				for(int i = 0; i < FEATURE_MAP_BLOCK_SIZE; ++i)
				{
					if (i < output_feature_map_count - output_feature_map_id)
					{
						#pragma unroll
						for(int j = 0; j < WINDOW_WIDTH_LOCAL; ++j)
							if (j < window_width - weight_x)
								atomicAdd(base_weights + i * weight_count_per_output_feature_map + j, sums[i * WINDOW_WIDTH_LOCAL + j]);
					}
				}
			}
		}

		template<int WINDOW_WIDTH>
		__global__ void convolution_1d_update_weights_exact_hess_kernel_fermi(
			float * __restrict hessian_weights,
			const float * __restrict output_errors,
			int output_width,
			int input_width,
			int input_feature_map_count,
			int output_feature_map_count,
			int entry_count,
			int block_size)
		{
			int input_feature_map_id = blockIdx.x * blockDim.x + threadIdx.x;
			int output_feature_map_id = (blockIdx.y * blockDim.y + threadIdx.y) * FEATURE_MAP_BLOCK_SIZE;
			int base_entry_id = (blockIdx.z * blockDim.z + threadIdx.z) * block_size;

			if ((input_feature_map_id < input_feature_map_count) && (output_feature_map_id < output_feature_map_count) && (base_entry_id < entry_count))
			{
				int output_neuron_count_per_feature_map = output_width;
				int iteration_count = min(block_size, entry_count - base_entry_id);

				const float * current_output_errors = output_errors + (base_entry_id * output_feature_map_count + output_feature_map_id) * output_width;
				int input_elem_id = (base_entry_id * input_feature_map_count + input_feature_map_id) * input_width;

				float sums[FEATURE_MAP_BLOCK_SIZE * WINDOW_WIDTH];
				#pragma unroll
				for(int i = 0; i < FEATURE_MAP_BLOCK_SIZE * WINDOW_WIDTH; ++i)
					sums[i] = 0.0F;

				int output_offsets[FEATURE_MAP_BLOCK_SIZE];
				#pragma unroll
				for(int i = 0; i < FEATURE_MAP_BLOCK_SIZE; ++i)
					output_offsets[i] = (i < output_feature_map_count - output_feature_map_id) ? output_neuron_count_per_feature_map * i : 0;

				for(int t = 0; t < iteration_count; ++t)
				{
					float input_squared_buf[WINDOW_WIDTH];
					#pragma unroll
					for(int i = 1; i < WINDOW_WIDTH; ++i)
					{
						input_squared_buf[i] = tex1Dfetch(input_squared_tex_ref, input_elem_id);
						++input_elem_id;
					}

					for(int x = 0; x < output_width; ++x)
					{
						float output_error_list[FEATURE_MAP_BLOCK_SIZE];
						#pragma unroll
						for(int i = 0; i < FEATURE_MAP_BLOCK_SIZE; ++i)
							output_error_list[i] = current_output_errors[output_offsets[i]];

						#pragma unroll
						for(int i = 0; i < WINDOW_WIDTH - 1; ++i)
							input_squared_buf[i] = input_squared_buf[i + 1];
						input_squared_buf[WINDOW_WIDTH - 1] = tex1Dfetch(input_squared_tex_ref, input_elem_id);

						#pragma unroll
						for(int i = 0; i < FEATURE_MAP_BLOCK_SIZE; ++i)
							#pragma unroll
							for(int j = 0; j < WINDOW_WIDTH; ++j)
								sums[i * WINDOW_WIDTH + j] += output_error_list[i] * input_squared_buf[j];

						current_output_errors++;
						input_elem_id++;
					}
					current_output_errors += (output_feature_map_count - 1) * output_width;
					input_elem_id += (input_feature_map_count - 1) * input_width;
				}

				float * base_weights = hessian_weights + (output_feature_map_id * input_feature_map_count + input_feature_map_id) * WINDOW_WIDTH;
				int weight_count_per_output_feature_map = input_feature_map_count * WINDOW_WIDTH;
				#pragma unroll
				for(int i = 0; i < FEATURE_MAP_BLOCK_SIZE; ++i)
				{
					if (i < output_feature_map_count - output_feature_map_id)
					{
						#pragma unroll
						for(int j = 0; j < WINDOW_WIDTH; ++j)
							atomicAdd(base_weights + i * weight_count_per_output_feature_map + j, sums[i * WINDOW_WIDTH + j]);
					}
				}
			}
		}

		convolution_1d_layer_hessian_cuda_fermi::convolution_1d_layer_hessian_cuda_fermi()
		{
			input_tex_ref.addressMode[0] = hipAddressModeBorder;
			input_tex_ref.normalized = false;
			output_tex_ref.addressMode[0] = hipAddressModeBorder;
			output_tex_ref.normalized = false;
			input_squared_tex_ref.addressMode[0] = hipAddressModeBorder;
			input_squared_tex_ref.normalized = false;
		}

		convolution_1d_layer_hessian_cuda_fermi::~convolution_1d_layer_hessian_cuda_fermi()
		{
		}

#define MAX_BLOCK_SIZE 5
#define MAX_WINDOW_WIDTH 10

#define launch_exact_kernel_const_const(window_width_const, block_size_const) \
	convolution_1d_tex_exact_blocked_hess_kernel_fermi<window_width_const,block_size_const><<<kernel_dims.first, kernel_dims.second, 0, stream_id>>>(*output_neurons_buffer, *data[0], *data[1], output_configuration_specific.dimension_sizes[0], input_configuration_specific.dimension_sizes[0], input_configuration_specific.feature_map_count, output_configuration_specific.feature_map_count, entry_count);

#define launch_exact_kernel_const(window_width, block_size_const) \
	switch (window_width) \
		{ \
		case 1: \
			launch_exact_kernel_const_const(1, block_size_const); \
			break; \
		case 2: \
			launch_exact_kernel_const_const(2, block_size_const); \
			break; \
		case 3: \
			launch_exact_kernel_const_const(3, block_size_const); \
			break; \
		case 4: \
			launch_exact_kernel_const_const(4, block_size_const); \
			break; \
		case 5: \
			launch_exact_kernel_const_const(5, block_size_const); \
			break; \
		case 6: \
			launch_exact_kernel_const_const(6, block_size_const); \
			break; \
		case 7: \
			launch_exact_kernel_const_const(7, block_size_const); \
			break; \
		case 8: \
			launch_exact_kernel_const_const(8, block_size_const); \
			break; \
		case 9: \
			launch_exact_kernel_const_const(9, block_size_const); \
			break; \
		case 10: \
			launch_exact_kernel_const_const(10, block_size_const); \
			break; \
		};

#define launch_exact_kernel(window_width, block_size) \
	switch (block_size) \
		{ \
		case 1: \
			launch_exact_kernel_const(window_width, 1); \
			break; \
		case 2: \
			launch_exact_kernel_const(window_width, 2); \
			break; \
		case 3: \
			launch_exact_kernel_const(window_width, 3); \
			break; \
		case 4: \
			launch_exact_kernel_const(window_width, 4); \
			break; \
		case 5: \
			launch_exact_kernel_const(window_width, 5); \
			break; \
		};

#define launch_kernel_const(block_size_const) \
	convolution_1d_tex_blocked_hess_kernel_fermi<block_size_const><<<kernel_dims.first, kernel_dims.second, 0, stream_id>>>(*output_neurons_buffer, *data[0], *data[1], output_configuration_specific.dimension_sizes[0], input_configuration_specific.dimension_sizes[0], window_sizes[0], input_configuration_specific.feature_map_count, output_configuration_specific.feature_map_count, entry_count);

#define launch_kernel(block_size) \
	switch (block_size) \
		{ \
		case 1: \
			launch_kernel_const(1); \
			break; \
		case 2: \
			launch_kernel_const(2); \
			break; \
		case 3: \
			launch_kernel_const(3); \
			break; \
		case 4: \
			launch_kernel_const(4); \
			break; \
		case 5: \
			launch_kernel_const(5); \
			break; \
		};

#define launch_backprop_exact_kernel_const_const(window_width_const, block_size_const) \
	convolution_1d_square_deriviative_tex_exact_hess_kernel_fermi<window_width_const,block_size_const><<<kernel_dims.first, kernel_dims.second, 0, stream_id>>>(*input_errors_buffer, *data_squared[0], output_configuration_specific.dimension_sizes[0], input_configuration_specific.dimension_sizes[0], input_configuration_specific.feature_map_count, output_configuration_specific.feature_map_count, entry_count);

#define launch_backprop_exact_kernel_const(window_width, block_size_const) \
	switch (window_width) \
		{ \
		case 1: \
			launch_backprop_exact_kernel_const_const(1, block_size_const); \
			break; \
		case 2: \
			launch_backprop_exact_kernel_const_const(2, block_size_const); \
			break; \
		case 3: \
			launch_backprop_exact_kernel_const_const(3, block_size_const); \
			break; \
		case 4: \
			launch_backprop_exact_kernel_const_const(4, block_size_const); \
			break; \
		case 5: \
			launch_backprop_exact_kernel_const_const(5, block_size_const); \
			break; \
		case 6: \
			launch_backprop_exact_kernel_const_const(6, block_size_const); \
			break; \
		case 7: \
			launch_backprop_exact_kernel_const_const(7, block_size_const); \
			break; \
		case 8: \
			launch_backprop_exact_kernel_const_const(8, block_size_const); \
			break; \
		case 9: \
			launch_backprop_exact_kernel_const_const(9, block_size_const); \
			break; \
		case 10: \
			launch_backprop_exact_kernel_const_const(10, block_size_const); \
			break; \
		};

#define launch_backprop_exact_kernel(window_width, block_size) \
	switch (block_size) \
		{ \
		case 1: \
			launch_backprop_exact_kernel_const(window_width, 1); \
			break; \
		case 2: \
			launch_backprop_exact_kernel_const(window_width, 2); \
			break; \
		case 3: \
			launch_backprop_exact_kernel_const(window_width, 3); \
			break; \
		case 4: \
			launch_backprop_exact_kernel_const(window_width, 4); \
			break; \
		case 5: \
			launch_backprop_exact_kernel_const(window_width, 5); \
			break; \
		};

#define launch_backprop_kernel_const(block_size_const) \
	convolution_1d_square_deriviative_tex_hess_kernel_fermi<block_size_const><<<kernel_dims.first, kernel_dims.second, 0, stream_id>>>(*input_errors_buffer, *data_squared[0], output_configuration_specific.dimension_sizes[0], input_configuration_specific.dimension_sizes[0], window_sizes[0], input_configuration_specific.feature_map_count, output_configuration_specific.feature_map_count, entry_count);

#define launch_backprop_kernel(block_size) \
	switch (block_size) \
		{ \
		case 1: \
			launch_backprop_kernel_const(1); \
			break; \
		case 2: \
			launch_backprop_kernel_const(2); \
			break; \
		case 3: \
			launch_backprop_kernel_const(3); \
			break; \
		case 4: \
			launch_backprop_kernel_const(4); \
			break; \
		case 5: \
			launch_backprop_kernel_const(5); \
			break; \
		};

#define launch_update_weights_exact_kernel_const(window_width_const) \
	convolution_1d_update_weights_exact_hess_kernel_fermi<window_width_const><<<kernel_dims.first, kernel_dims.second, 0, stream_id>>>(*hessian_data[0], *output_errors_buffer, output_configuration_specific.dimension_sizes[0], input_configuration_specific.dimension_sizes[0], input_configuration_specific.feature_map_count, output_configuration_specific.feature_map_count, entry_count, block_size);

#define launch_update_weights_exact_kernel(window_width) \
	switch (window_width) \
		{ \
		case 1: \
			launch_update_weights_exact_kernel_const(1); \
			break; \
		case 2: \
			launch_update_weights_exact_kernel_const(2); \
			break; \
		case 3: \
			launch_update_weights_exact_kernel_const(3); \
			break; \
		case 4: \
			launch_update_weights_exact_kernel_const(4); \
			break; \
		case 5: \
			launch_update_weights_exact_kernel_const(5); \
			break; \
		case 6: \
			launch_update_weights_exact_kernel_const(6); \
			break; \
		case 7: \
			launch_update_weights_exact_kernel_const(7); \
			break; \
		case 8: \
			launch_update_weights_exact_kernel_const(8); \
			break; \
		case 9: \
			launch_update_weights_exact_kernel_const(9); \
			break; \
		case 10: \
			launch_update_weights_exact_kernel_const(10); \
			break; \
		};

		void convolution_1d_layer_hessian_cuda_fermi::enqueue_test(
			hipStream_t stream_id,
			const std::vector<const_cuda_linear_buffer_device_smart_ptr>& schema_data,
			const std::vector<const_cuda_linear_buffer_device_smart_ptr>& data,
			const_cuda_linear_buffer_device_smart_ptr input_neurons_buffer,
			cuda_linear_buffer_device_smart_ptr output_neurons_buffer,
			const std::vector<cuda_linear_buffer_device_smart_ptr>& additional_buffers,
			unsigned int entry_count)
		{
			hipChannelFormatDesc desc = hipCreateChannelDesc<float>();
			cuda_safe_call(hipBindTexture(0, input_tex_ref, *input_neurons_buffer, desc, input_elem_count_per_entry * entry_count * sizeof(float)));

			int block_size = get_block_size(output_configuration_specific.dimension_sizes[0]);
			std::pair<dim3, dim3> kernel_dims = cuda_util::get_grid_and_threadblock_sizes_2d_access(
				*cuda_config,
				(output_configuration_specific.dimension_sizes[0] + block_size - 1) / block_size,
				((output_configuration_specific.feature_map_count + FEATURE_MAP_BLOCK_SIZE - 1) / FEATURE_MAP_BLOCK_SIZE),
				entry_count);

			if (window_sizes[0] <= MAX_WINDOW_WIDTH)
			{
				launch_exact_kernel(window_sizes[0], block_size);
			}
			else
			{
				launch_kernel(block_size);
			}
		}

		void convolution_1d_layer_hessian_cuda_fermi::enqueue_backprop(
			hipStream_t stream_id,
			const std::vector<const_cuda_linear_buffer_device_smart_ptr>& schema_data,
			const std::vector<const_cuda_linear_buffer_device_smart_ptr>& data_squared,
			const_cuda_linear_buffer_device_smart_ptr output_neurons_buffer,
			cuda_linear_buffer_device_smart_ptr output_errors_buffer,
			cuda_linear_buffer_device_smart_ptr input_errors_buffer,
			const std::vector<cuda_linear_buffer_device_smart_ptr>& additional_buffers,
			unsigned int entry_count)
		{
			hipChannelFormatDesc desc = hipCreateChannelDesc<float>();
			cuda_safe_call(hipBindTexture(0, output_tex_ref, *output_errors_buffer, desc, output_elem_count_per_entry * entry_count * sizeof(float)));

			int block_size = get_block_size(input_configuration_specific.dimension_sizes[0]);
			std::pair<dim3, dim3> kernel_dims = cuda_util::get_grid_and_threadblock_sizes_2d_access(
				*cuda_config,
				(input_configuration_specific.dimension_sizes[0] + block_size - 1) / block_size,
				((input_configuration_specific.feature_map_count + FEATURE_MAP_BLOCK_SIZE - 1) / FEATURE_MAP_BLOCK_SIZE),
				entry_count);

			if (window_sizes[0] <= MAX_WINDOW_WIDTH)
			{
				launch_backprop_exact_kernel(window_sizes[0], block_size);
			}
			else
			{
				launch_backprop_kernel(block_size);
			}
		}

		void convolution_1d_layer_hessian_cuda_fermi::enqueue_update_hessian(
			hipStream_t stream_id,
			const std::vector<const_cuda_linear_buffer_device_smart_ptr>& schema_data,
			const std::vector<cuda_linear_buffer_device_smart_ptr>& hessian_data,
			cuda_linear_buffer_device_smart_ptr output_errors_buffer,
			const_cuda_linear_buffer_device_smart_ptr input_neurons_buffer,
			const std::vector<cuda_linear_buffer_device_smart_ptr>& additional_buffers,
			unsigned int entry_count)
		{
			hipChannelFormatDesc desc = hipCreateChannelDesc<float>();
			cuda_safe_call(hipBindTexture(0, input_squared_tex_ref, *additional_buffers[0], desc, input_elem_count_per_entry * entry_count * sizeof(float)));

			// Update weights
			{
				// Store input neurons multiplied element-wise by themselves
				cuda_util::multiply_by_itself(
					*cuda_config,
					*input_neurons_buffer,
					*additional_buffers[0],
					input_elem_count_per_entry * entry_count,
					stream_id);

				if (window_sizes[0] <= MAX_WINDOW_WIDTH)
				{
					int block_size = get_weights_update_block_size(entry_count);
					int block_count = (entry_count + block_size - 1) / block_size;
					std::pair<dim3, dim3> kernel_dims = cuda_util::get_grid_and_threadblock_sizes_sequential_access(
						*cuda_config,
						((output_configuration_specific.feature_map_count + FEATURE_MAP_BLOCK_SIZE - 1) / FEATURE_MAP_BLOCK_SIZE),
						input_configuration_specific.feature_map_count,
						block_count);

					launch_update_weights_exact_kernel(window_sizes[0]);
				}
				else
				{
					int window_x_block_count = (window_sizes[0] + WINDOW_WIDTH_LOCAL - 1) / WINDOW_WIDTH_LOCAL;
					int block_size = get_weights_update_block_size(entry_count);
					int block_count = (entry_count + block_size - 1) / block_size;
					std::pair<dim3, dim3> kernel_dims = cuda_util::get_grid_and_threadblock_sizes_sequential_access(
						*cuda_config,
						window_x_block_count,
						((output_configuration_specific.feature_map_count + FEATURE_MAP_BLOCK_SIZE - 1) / FEATURE_MAP_BLOCK_SIZE) * input_configuration_specific.feature_map_count,
						block_count);

					convolution_1d_update_weights_hess_kernel_fermi<<<kernel_dims.first, kernel_dims.second, 0, stream_id>>>(
						*hessian_data[0],
						*output_errors_buffer,
						output_configuration_specific.dimension_sizes[0],
						input_configuration_specific.dimension_sizes[0],
						window_sizes[0],
						input_configuration_specific.feature_map_count,
						output_configuration_specific.feature_map_count,
						entry_count,
						window_x_block_count,
						block_size);
				}
			}

			// Update biases
			{
				int block_size = get_bias_update_block_size(entry_count);
				int block_count = (entry_count + block_size - 1) / block_size;
				std::pair<dim3, dim3> kernel_dims = cuda_util::get_grid_and_threadblock_sizes_sequential_access(
					*cuda_config,
					output_elem_count_per_feature_map,
					1,
					block_count);
				kernel_dims.first.y = output_configuration_specific.feature_map_count;
				int threadblock_size = kernel_dims.second.x * kernel_dims.second.y * kernel_dims.second.z;
				int smem_size = threadblock_size * sizeof(float);
				convolution_1d_update_biases_hess_kernel_fermi<<<kernel_dims.first, kernel_dims.second, smem_size, stream_id>>>(
					*hessian_data[1],
					*output_errors_buffer,
					block_size,
					output_elem_count_per_feature_map,
					output_configuration_specific.feature_map_count,
					entry_count);
			}
		}

		int convolution_1d_layer_hessian_cuda_fermi::get_block_size(int width)
		{
			int block_count = (width + MAX_BLOCK_SIZE - 1) / MAX_BLOCK_SIZE;
			int block_size = (width + block_count - 1) / block_count;
			return block_size;
		}

		void convolution_1d_layer_hessian_cuda_fermi::hessian_configured()
		{
			nnforge_shared_ptr<const convolution_layer> layer_derived = nnforge_dynamic_pointer_cast<const convolution_layer>(layer_schema);

			for(std::vector<unsigned int>::const_iterator it = layer_derived->window_sizes.begin(); it != layer_derived->window_sizes.end(); ++it)
				window_sizes.push_back(static_cast<int>(*it));
		}

		bool convolution_1d_layer_hessian_cuda_fermi::is_in_place_backprop() const
		{
			return false;
		}

		std::vector<size_t> convolution_1d_layer_hessian_cuda_fermi::get_sizes_of_additional_buffers_per_entry() const
		{
			std::vector<size_t> res;

			res.push_back(input_elem_count_per_entry * sizeof(float));

			return res;
		}

		std::vector<unsigned int> convolution_1d_layer_hessian_cuda_fermi::get_linear_addressing_through_texture_per_entry() const
		{
			std::vector<unsigned int> res;

			res.push_back(input_elem_count_per_entry);
			res.push_back(output_elem_count_per_entry);

			return res;
		}

		int convolution_1d_layer_hessian_cuda_fermi::get_bias_update_block_size(int entry_count)
		{
			int block_size = std::min<int>(std::max<int>(static_cast<int>(sqrtf(static_cast<float>(entry_count))), 1), entry_count);
			return block_size;
		}

		int convolution_1d_layer_hessian_cuda_fermi::get_weights_update_block_size(int entry_count)
		{
			int block_size = std::min<int>(std::max<int>(static_cast<int>(sqrtf(static_cast<float>(entry_count))), 1), entry_count);
			return block_size;
		}
	}
}
