#include "hip/hip_runtime.h"
/*
 *  Copyright 2011-2013 Maxim Milakov
 *
 *  Licensed under the Apache License, Version 2.0 (the "License");
 *  you may not use this file except in compliance with the License.
 *  You may obtain a copy of the License at
 *
 *      http://www.apache.org/licenses/LICENSE-2.0
 *
 *  Unless required by applicable law or agreed to in writing, software
 *  distributed under the License is distributed on an "AS IS" BASIS,
 *  WITHOUT WARRANTIES OR CONDITIONS OF ANY KIND, either express or implied.
 *  See the License for the specific language governing permissions and
 *  limitations under the License.
 */

#include "network_updater_cuda.h"

#include "neural_network_cuda_exception.h"
#include "layer_testing_schema_factory.h"
#include "cuda_linear_buffer_device.h"
#include "cuda_linear_buffer_host.h"
#include "cuda_util.h"
#include "cuda_event.h"
#include "layer_updater_schema_factory.h"

#include <hip/hip_runtime.h>
#include <boost/format.hpp>
#include <stack>

__global__ void convert_compacted_to_raw_upd_kernel(
	const unsigned char * __restrict input,
	float * __restrict output,
	const float * __restrict scale_addition,
	const float * __restrict scale_multiplication,
	int elem_count_per_feature_map,
	int feature_map_count,
	int entry_count)
{
	int elem_id_inside_feature_map = blockIdx.x * blockDim.x + threadIdx.x;
	int feature_map_id = blockIdx.y * blockDim.y + threadIdx.y;
	int entry_id = blockIdx.z * blockDim.z + threadIdx.z;
	bool in_bounds = (entry_id < entry_count) && (elem_id_inside_feature_map < elem_count_per_feature_map) && (feature_map_id < feature_map_count);
	if (in_bounds)
	{
		int offset = elem_count_per_feature_map * (entry_id * feature_map_count + feature_map_id) + elem_id_inside_feature_map;
		unsigned char val = input[offset];
		float converted_val = ((val * (1.0F / 255.0F)) + scale_addition[feature_map_id]) * scale_multiplication[feature_map_id];
		output[offset] = converted_val;
	}
}

__global__ void compute_error_upd_kernel(
	float * __restrict errors,
	float * __restrict mse,
	const float * __restrict desired_output_neurons,
	const float * __restrict actual_output_neurons,
	int output_entry_id,
	int output_elem_count,
	int updater_entry_count)
{
	int elem_id = blockIdx.x * blockDim.x + threadIdx.x;
	int updater_entry_id = blockIdx.y * blockDim.y + threadIdx.y;
	bool in_bounds = (elem_id < output_elem_count) && (updater_entry_id < updater_entry_count);
	if (in_bounds)
	{
		int offset = updater_entry_id * output_elem_count + elem_id;
		float err = desired_output_neurons[output_entry_id * output_elem_count + elem_id] - actual_output_neurons[offset];
		errors[offset] = err;
		mse[offset] += err * err * 0.5F;
	}
}

namespace nnforge
{
	namespace cuda
	{
		unsigned int network_updater_cuda::max_entry_count_in_single_batch = 1024;

		network_updater_cuda::network_updater_cuda(
			network_schema_smart_ptr schema,
			const_data_scale_params_smart_ptr scale_params,
			cuda_running_configuration_const_smart_ptr cuda_config)
			: network_updater(schema, scale_params)
			, cuda_config(cuda_config)
		{
			const const_layer_list& layer_list = *schema;

			testing_layer_count = 0;
			start_layer_nonempty_weights_iterator = layer_list.begin();
			for(const_layer_list::const_iterator it = layer_list.begin(); it != layer_list.end(); ++it)
			{
				start_layer_nonempty_weights_iterator = it;

				if (!(*it)->is_empty_data())
					break;

				testing_layer_count++;
			}

			for(const_layer_list::const_iterator it = layer_list.begin(); it != start_layer_nonempty_weights_iterator; ++it)
				testing_schemas.push_back(single_layer_testing_schema_factory::get_const_instance().create_testing_schema_layer(*it, cuda_config));

			for(const_layer_list::const_iterator it = start_layer_nonempty_weights_iterator; it != layer_list.end(); ++it)
				updater_schemas.push_back(single_layer_updater_schema_factory::get_const_instance().create_updater_schema_layer(*it, cuda_config));

			setup_network_cuda();

			for(const_layer_testing_schema_list::const_iterator it = testing_schemas.begin(); it != testing_schemas.end(); ++it)
				testing_schema_data.push_back((*it)->get_schema_buffers());

			for(const_layer_updater_schema_list::const_iterator it = updater_schemas.begin(); it != updater_schemas.end(); ++it)
				updater_schema_data.push_back((*it)->get_schema_buffers());
		}

		network_updater_cuda::~network_updater_cuda()
		{
		}

		void network_updater_cuda::setup_network_cuda()
		{
			command_stream = cuda_stream_smart_ptr(new hip_stream());
			data_stream = cuda_stream_smart_ptr(new hip_stream());
		}

		std::vector<testing_result_smart_ptr> network_updater_cuda::actual_update(
			supervised_data_reader_byte& reader,
			const std::vector<network_data_smart_ptr>& training_speed_vector_list,
			std::vector<network_data_smart_ptr>& data_list,
			const std::map<unsigned int, float>& layer_to_dropout_rate_map,
			const std::vector<float>& random_uniform_list)
		{
			std::vector<testing_result_smart_ptr> res;

			unsigned int min_dropout_layer_id = testing_layer_count + 1;
			for(std::map<unsigned int, float>::const_iterator it = layer_to_dropout_rate_map.begin(); it != layer_to_dropout_rate_map.end(); ++it)
				if (it->first < min_dropout_layer_id)
					throw neural_network_exception((boost::format("Unable to apply dropout to layer %1%") % it->first).str());

			reader.reset();

			layer_configuration_specific input_configuration = reader.get_input_configuration();
			layer_configuration_specific output_configuration = reader.get_output_configuration();

			unsigned int input_neuron_count = input_configuration.get_neuron_count();
			unsigned int output_neuron_count = output_configuration.get_neuron_count();
			unsigned int input_neuron_count_per_feature_map = input_configuration.get_neuron_count_per_feature_map();
			unsigned int input_feature_map_count = input_configuration.feature_map_count;

			unsigned int updater_entry_count = static_cast<unsigned int>(data_list.size());
			if (updater_entry_count == 0)
				return res;

			for(unsigned int i = 0; i < training_speed_vector_list.size(); ++i)
				res.push_back(testing_result_smart_ptr(new testing_result(output_neuron_count)));

			std::vector<std::vector<cuda_linear_buffer_device_smart_ptr> > net_data = enqueue_get_data(data_list, *command_stream);
			std::vector<std::vector<const_cuda_linear_buffer_device_smart_ptr> > training_speed_data = enqueue_get_training_speed(training_speed_vector_list, *command_stream);

			buffer_cuda_size_configuration buffers_config;
			update_buffers_configuration(buffers_config, updater_entry_count);

			buffers_config.add_per_entry_buffer(input_neuron_count * sizeof(unsigned char)); // input
			buffers_config.add_per_entry_buffer(input_neuron_count * sizeof(unsigned char)); // input
			buffers_config.add_per_entry_buffer(input_neuron_count * sizeof(float)); // converted input
			buffers_config.add_per_entry_buffer(output_neuron_count * sizeof(float)); // output
			buffers_config.add_per_entry_buffer(output_neuron_count * sizeof(float)); // output
			buffers_config.add_constant_buffer(output_neuron_count * sizeof(float) * updater_entry_count); // initial error
			buffers_config.add_constant_buffer(output_neuron_count * sizeof(float) * updater_entry_count); // mse
			if (!random_uniform_list.empty())
				buffers_config.add_constant_buffer(random_uniform_list.size() * sizeof(float)); // random_uniform_list

			for(std::vector<std::vector<cuda_linear_buffer_device_smart_ptr> >::const_iterator it = net_data.begin(); it != net_data.end(); ++it)
				for(std::vector<cuda_linear_buffer_device_smart_ptr>::const_iterator it2 = it->begin(); it2 != it->end(); ++it2)
					buffers_config.add_constant_buffer((*it2)->get_size());

			for(std::vector<std::vector<const_cuda_linear_buffer_device_smart_ptr> >::const_iterator it = training_speed_data.begin(); it != training_speed_data.end(); ++it)
				for(std::vector<const_cuda_linear_buffer_device_smart_ptr>::const_iterator it2 = it->begin(); it2 != it->end(); ++it2)
					buffers_config.add_constant_buffer((*it2)->get_size());

			unsigned int max_entry_count = std::min<unsigned int>(std::min<unsigned int>(cuda_config->get_max_entry_count(buffers_config), reader.get_entry_count()), max_entry_count_in_single_batch);

			cuda_linear_buffer_device_smart_ptr input_buf[2] = 
			{
				cuda_linear_buffer_device_smart_ptr(new cuda_linear_buffer_device(input_neuron_count * max_entry_count * sizeof(unsigned char))),
				cuda_linear_buffer_device_smart_ptr(new cuda_linear_buffer_device(input_neuron_count * max_entry_count * sizeof(unsigned char))),
			};

			cuda_linear_buffer_device_smart_ptr output_buf[2] = 
			{
				cuda_linear_buffer_device_smart_ptr(new cuda_linear_buffer_device(output_neuron_count * max_entry_count * sizeof(float))),
				cuda_linear_buffer_device_smart_ptr(new cuda_linear_buffer_device(output_neuron_count * max_entry_count * sizeof(float))),
			};

			cuda_linear_buffer_device_smart_ptr input_converted_buf(new cuda_linear_buffer_device(input_neuron_count * max_entry_count * sizeof(float)));

			cuda_linear_buffer_device_smart_ptr initial_error_buf(new cuda_linear_buffer_device(output_neuron_count * updater_entry_count * sizeof(float)));

			cuda_linear_buffer_device_smart_ptr mse_buf(new cuda_linear_buffer_device(output_neuron_count * updater_entry_count * sizeof(float)));

			cuda_linear_buffer_device_smart_ptr random_uniform_buf;
			if (!random_uniform_list.empty())
			{
				random_uniform_buf = cuda_linear_buffer_device_smart_ptr(new cuda_linear_buffer_device(random_uniform_list.size() * sizeof(float)));
				cuda_safe_call(hipMemcpyAsync(*random_uniform_buf, &(*random_uniform_list.begin()), random_uniform_list.size() * sizeof(float), hipMemcpyHostToDevice, *command_stream));
			}

			cuda_linear_buffer_device_smart_ptr output_buffer = input_converted_buf;
			std::vector<std::pair<cuda_linear_buffer_device_smart_ptr, std::vector<cuda_linear_buffer_device_smart_ptr> > > testing_input_and_additional_buffers_pack;
			for(std::vector<layer_tester_cuda_smart_ptr>::iterator it = tester_list.begin(); it != tester_list.end(); ++it)
			{
				std::vector<cuda_linear_buffer_device_smart_ptr> additional_buffers = (*it)->allocate_additional_buffers(max_entry_count);
				testing_input_and_additional_buffers_pack.push_back(std::make_pair<cuda_linear_buffer_device_smart_ptr, std::vector<cuda_linear_buffer_device_smart_ptr> >(output_buffer, additional_buffers));
				output_buffer = (*it)->get_output_buffer(output_buffer, additional_buffers);
			}
			std::vector<std::pair<cuda_linear_buffer_device_smart_ptr, layer_updater_cuda::buffer_set> > updater_input_and_all_buffers_pack;
			for(std::vector<layer_updater_cuda_smart_ptr>::iterator it = updater_list.begin(); it != updater_list.end(); ++it)
			{
				layer_updater_cuda::buffer_set all_buffers = (*it)->allocate_all_buffers(updater_entry_count);
				updater_input_and_all_buffers_pack.push_back(std::make_pair<cuda_linear_buffer_device_smart_ptr, layer_updater_cuda::buffer_set>(output_buffer, all_buffers));
				output_buffer = all_buffers.output_neurons_buffer;
			}

			std::vector<cuda_linear_buffer_device_smart_ptr> output_errors_buffers;
			cuda_linear_buffer_device_smart_ptr output_errors = initial_error_buf;
			for(std::vector<std::pair<cuda_linear_buffer_device_smart_ptr, layer_updater_cuda::buffer_set> >::reverse_iterator it = updater_input_and_all_buffers_pack.rbegin(); it != updater_input_and_all_buffers_pack.rend(); ++it)
			{
				output_errors_buffers.push_back(output_errors);
				layer_updater_cuda::buffer_set& all_buffers = it->second;

				if (all_buffers.input_errors_buffer != 0)
					output_errors = all_buffers.input_errors_buffer;
			}

			cuda_linear_buffer_host_smart_ptr input_host_buf(new cuda_linear_buffer_host(input_neuron_count * max_entry_count * sizeof(unsigned char)));
			unsigned char * input = *input_host_buf;
			cuda_linear_buffer_host_smart_ptr output_host_buf(new cuda_linear_buffer_host(output_neuron_count * max_entry_count * sizeof(float)));
			float * output = *output_host_buf;

			// zero mse
			cuda_util::set_with_value(
				*cuda_config,
				*mse_buf,
				0.0F,
				output_neuron_count * updater_entry_count,
				*command_stream);

			unsigned int current_data_slot = 0;
			unsigned int current_command_slot = 1;
			unsigned int entries_available_for_copy_in_count = reader.get_entry_count();
			unsigned int entries_available_for_processing_count = 0;
			cuda_event data_processed_event;
			cuda_event input_copied_event;
			if (cuda_config->is_flush_required())
			{
				cuda_safe_call(hipEventRecord(data_processed_event, *command_stream));
				hipEventQuery(data_processed_event);
			}
			std::tr1::variate_generator<random_generator, std::tr1::uniform_int<unsigned int> > gen_random_offset(
				rnd::get_random_generator(),
				std::tr1::uniform_int<unsigned int>(0, static_cast<unsigned int>(random_uniform_list.size() - 1)));
			unsigned int mask = static_cast<unsigned int>(random_uniform_list.size() - 1);
			while((entries_available_for_copy_in_count > 0) || (entries_available_for_processing_count > 0))
			{
				if (entries_available_for_processing_count > 0)
				{
					// Convert input
					{
						std::pair<dim3, dim3> convert_compacted_to_raw_2d_surf_kernel_dims = cuda_util::get_grid_and_threadblock_sizes_sequential_access(
							*cuda_config,
							input_neuron_count_per_feature_map,
							input_feature_map_count,
							entries_available_for_processing_count);
						convert_compacted_to_raw_upd_kernel<<<convert_compacted_to_raw_2d_surf_kernel_dims.first, convert_compacted_to_raw_2d_surf_kernel_dims.second, 0, *command_stream>>>(
							*input_buf[current_command_slot],
							*input_converted_buf,
							*scale_addition,
							*scale_multiplication,
							input_neuron_count_per_feature_map,
							input_feature_map_count,
							entries_available_for_processing_count);
					}

					// Run ann
					{
						std::vector<std::pair<cuda_linear_buffer_device_smart_ptr, std::vector<cuda_linear_buffer_device_smart_ptr> > >::iterator input_and_additional_buffers_pack_it = testing_input_and_additional_buffers_pack.begin();
						std::vector<std::vector<const_cuda_linear_buffer_device_smart_ptr> >::iterator schema_data_it = testing_schema_data.begin();
						for(std::vector<layer_tester_cuda_smart_ptr>::iterator it = tester_list.begin(); it != tester_list.end(); ++it, ++input_and_additional_buffers_pack_it, ++schema_data_it)
						{
							(*it)->enqueue_test(
								*command_stream,
								*schema_data_it,
								std::vector<const_cuda_linear_buffer_device_smart_ptr>(),
								input_and_additional_buffers_pack_it->first,
								input_and_additional_buffers_pack_it->second,
								entries_available_for_processing_count);
						}
					}

					for(unsigned int input_entry_id = 0; input_entry_id < entries_available_for_processing_count; ++input_entry_id)
					{
						std::stack<unsigned int> offset_list;

						// Forward updater
						{
							std::vector<std::pair<cuda_linear_buffer_device_smart_ptr, layer_updater_cuda::buffer_set> >::iterator input_and_all_buffers_pack_it = updater_input_and_all_buffers_pack.begin();
							std::vector<std::vector<cuda_linear_buffer_device_smart_ptr> >::iterator net_data_it = net_data.begin();
							std::vector<std::vector<const_cuda_linear_buffer_device_smart_ptr> >::iterator schema_data_it = updater_schema_data.begin();
							unsigned int layer_id = testing_layer_count;
							for(std::vector<layer_updater_cuda_smart_ptr>::iterator it = updater_list.begin(); it != updater_list.end(); ++it, ++input_and_all_buffers_pack_it, ++schema_data_it, ++net_data_it, ++layer_id)
							{
								if (it != updater_list.begin())
								{
									std::map<unsigned int, float>::const_iterator dropout_it = layer_to_dropout_rate_map.find(layer_id);
									if (dropout_it != layer_to_dropout_rate_map.end())
									{
										unsigned int offset = gen_random_offset();
										offset_list.push(offset);
										(*it)->enqueue_forward_dropout(
											*command_stream,
											random_uniform_buf,
											input_and_all_buffers_pack_it->first,
											dropout_it->second,
											mask,
											updater_entry_count,
											offset);
									}
								}

								(*it)->enqueue_test(
									it == updater_list.begin() ? input_entry_id : 0,
									*command_stream,
									*schema_data_it,
									*net_data_it,
									input_and_all_buffers_pack_it->first,
									input_and_all_buffers_pack_it->second.output_neurons_buffer,
									input_and_all_buffers_pack_it->second.additional_buffers,
									updater_entry_count);
							}
						}

						// Compute errors
						{
							std::pair<dim3, dim3> kernel_dims = cuda_util::get_grid_and_threadblock_sizes_sequential_access(
								*cuda_config,
								output_neuron_count,
								updater_entry_count,
								1);
							compute_error_upd_kernel<<<kernel_dims.first, kernel_dims.second, 0, *command_stream>>>(
								*initial_error_buf,
								*mse_buf,
								*output_buf[current_command_slot],
								*output_buffer,
								input_entry_id,
								output_neuron_count,
								updater_entry_count);
						}

						// Backward updater
						{
							std::vector<cuda_linear_buffer_device_smart_ptr>::iterator output_errors_it = output_errors_buffers.begin();
							std::vector<std::pair<cuda_linear_buffer_device_smart_ptr, layer_updater_cuda::buffer_set> >::reverse_iterator input_and_all_buffers_pack_it = updater_input_and_all_buffers_pack.rbegin();
							std::vector<std::vector<cuda_linear_buffer_device_smart_ptr> >::reverse_iterator net_data_it = net_data.rbegin();
							std::vector<std::vector<const_cuda_linear_buffer_device_smart_ptr> >::reverse_iterator training_speed_data_it = training_speed_data.rbegin();
							std::vector<std::vector<const_cuda_linear_buffer_device_smart_ptr> >::reverse_iterator schema_data_it = updater_schema_data.rbegin();
							unsigned int reverse_layer_id = static_cast<unsigned int>(updater_list.size() + testing_layer_count) - 1;
							for(std::vector<layer_updater_cuda_smart_ptr>::reverse_iterator it = updater_list.rbegin(); it != updater_list.rend(); ++it, ++input_and_all_buffers_pack_it, ++schema_data_it, ++training_speed_data_it, ++output_errors_it, ++net_data_it, --reverse_layer_id)
							{
								if (it != (updater_list.rend() - 1))
								{
									(*it)->enqueue_backprop(
										*command_stream,
										*schema_data_it,
										*net_data_it,
										input_and_all_buffers_pack_it->second.output_neurons_buffer,
										input_and_all_buffers_pack_it->first,
										*output_errors_it,
										input_and_all_buffers_pack_it->second.input_errors_buffer,
										input_and_all_buffers_pack_it->second.additional_buffers,
										updater_entry_count);

									std::map<unsigned int, float>::const_iterator dropout_it = layer_to_dropout_rate_map.find(reverse_layer_id);
									if (dropout_it != layer_to_dropout_rate_map.end())
									{
										unsigned int offset = offset_list.top();
										offset_list.pop();
										(*it)->enqueue_backward_dropout(
											*command_stream,
											random_uniform_buf,
											input_and_all_buffers_pack_it->second.input_errors_buffer,
											dropout_it->second,
											mask,
											updater_entry_count,
											offset);
									}
								}

								(*it)->enqueue_update_weights(
									(it == (updater_list.rend() - 1)) ? input_entry_id : 0,
									*command_stream,
									*net_data_it,
									*schema_data_it,
									*training_speed_data_it,
									*output_errors_it,
									input_and_all_buffers_pack_it->first,
									input_and_all_buffers_pack_it->second.additional_buffers,
									updater_entry_count);
							}
						}

						if (((input_entry_id % 16) == 1) && cuda_config->is_flush_required())
						{
							cuda_safe_call(hipEventRecord(data_processed_event, *command_stream));
							hipEventQuery(data_processed_event);
						}
					} // for(unsigned int input_entry_id

					if (profile_mode)
						entry_count_updated_in_profile_mode = entries_available_for_processing_count;

					for(std::vector<testing_result_smart_ptr>::iterator it = res.begin(); it != res.end(); ++it)
						(*it)->entry_count += entries_available_for_processing_count;

					if (cuda_config->is_flush_required())
					{
						cuda_safe_call(hipEventRecord(data_processed_event, *command_stream));
						hipEventQuery(data_processed_event);
					}
				} // if (entries_available_for_processing_count > 0)

				unsigned int entries_read_count = 0;
				if (entries_available_for_copy_in_count > 0)
				{
					unsigned int entries_to_read_count = std::min<unsigned int>(max_entry_count, entries_available_for_copy_in_count);
					while(entries_read_count < entries_to_read_count)
					{
						bool entry_read = reader.read(
							input + (input_neuron_count * entries_read_count),
							output + (output_neuron_count * entries_read_count));

						if (!entry_read)
							break;

						entries_read_count++;
					}
					cuda_safe_call(hipMemcpyAsync(
						*(input_buf[current_data_slot]),
						input,
						entries_read_count * input_neuron_count * sizeof(unsigned char),
						hipMemcpyHostToDevice,
						*data_stream));
					cuda_safe_call(hipMemcpyAsync(
						*(output_buf[current_data_slot]),
						output,
						entries_read_count * output_neuron_count * sizeof(float),
						hipMemcpyHostToDevice,
						*data_stream));
				}

				cuda_safe_call(hipStreamSynchronize(*data_stream));
				cuda_safe_call(hipStreamSynchronize(*command_stream));

				entries_available_for_processing_count = entries_read_count;
				entries_available_for_copy_in_count -= entries_read_count;

				current_data_slot = 1 - current_data_slot;
				current_command_slot = 1 - current_command_slot;

				if (profile_mode)
					entries_available_for_copy_in_count = 0;
			}

			read_data(net_data, data_list, *command_stream);

			std::vector<float> mse_list(output_neuron_count * updater_entry_count);
			cuda_safe_call(hipMemcpyAsync(&(*mse_list.begin()), *mse_buf, mse_list.size() * sizeof(float), hipMemcpyDeviceToHost, *command_stream));
			cuda_safe_call(hipStreamSynchronize(*command_stream));

			for(unsigned int i = 0; i < updater_entry_count; ++i)
				std::copy(mse_list.begin() + output_neuron_count * i, mse_list.begin() + output_neuron_count * (i + 1), res[i]->cumulative_mse_list.begin());

			return res;
		}

		void network_updater_cuda::layer_config_list_modified()
		{
			layer_configuration_specific_list::const_iterator it_conf = layer_config_list.begin();

			tester_list.clear();
			for(const_layer_testing_schema_list::const_iterator it = testing_schemas.begin(); it != testing_schemas.end(); ++it, ++it_conf)
			{
				tester_list.push_back(
					(*it)->create_tester(
						*it_conf,
						*(it_conf + 1)));
			}

			updater_list.clear();
			for(const_layer_updater_schema_list::const_iterator it = updater_schemas.begin(); it != updater_schemas.end(); ++it, ++it_conf)
			{
				updater_list.push_back(
					(*it)->create_updater(
						*it_conf,
						*(it_conf + 1),
						(it_conf > layer_config_list.begin() + testing_layer_count),
						(it_conf > layer_config_list.begin() + testing_layer_count)));
			}

			scale_multiplication = cuda_linear_buffer_device_smart_ptr(new cuda_linear_buffer_device(
				&(*current_scale_params->multiplication_list.begin()),
				current_scale_params->multiplication_list.size() * sizeof(float)));

			scale_addition = cuda_linear_buffer_device_smart_ptr(new cuda_linear_buffer_device(
				&(*current_scale_params->addition_list.begin()),
				current_scale_params->addition_list.size() * sizeof(float)));
		}

		std::vector<std::vector<const_cuda_linear_buffer_device_smart_ptr> > network_updater_cuda::enqueue_get_training_speed(
			const std::vector<network_data_smart_ptr>& training_speed_list,
			hipStream_t stream_id) const
		{
			std::vector<std::vector<const_cuda_linear_buffer_device_smart_ptr> > res;

			const network_data_smart_ptr& first_data = training_speed_list[0];

			for(unsigned int layer_id = testing_layer_count; layer_id < updater_schemas.size() + testing_layer_count; ++layer_id)
			{
				std::vector<const_cuda_linear_buffer_device_smart_ptr> buffer_list;
				unsigned int subindex = 0;
				for(std::vector<std::vector<float> >::iterator it = (*first_data)[layer_id]->begin(); it != (*first_data)[layer_id]->end(); ++it, ++subindex)
				{
					size_t single_size = it->size();
					std::vector<float> pack(single_size * training_speed_list.size());

					std::vector<float>::iterator fill_it = pack.begin();
					for(std::vector<network_data_smart_ptr>::const_iterator sample_it = training_speed_list.begin(); sample_it != training_speed_list.end(); sample_it++)
					{
						const std::vector<float>& inp_buf = (*sample_it)->at(layer_id)->at(subindex);
						fill_it = std::copy(inp_buf.begin(), inp_buf.end(), fill_it);
					}

					buffer_list.push_back(cuda_linear_buffer_device_smart_ptr(new cuda_linear_buffer_device(
						&(*pack.begin()),
						pack.size() * sizeof(float),
						stream_id)));
				}
				res.push_back(buffer_list);
			}

			return res;
		}

		std::vector<std::vector<cuda_linear_buffer_device_smart_ptr> > network_updater_cuda::enqueue_get_data(
			const std::vector<network_data_smart_ptr>& data_list,
			hipStream_t stream_id) const
		{
			std::vector<std::vector<cuda_linear_buffer_device_smart_ptr> > res;

			const network_data_smart_ptr& first_data = data_list[0];

			for(unsigned int layer_id = testing_layer_count; layer_id < updater_schemas.size() + testing_layer_count; ++layer_id)
			{
				std::vector<cuda_linear_buffer_device_smart_ptr> buffer_list;
				unsigned int subindex = 0;
				for(std::vector<std::vector<float> >::iterator it = (*first_data)[layer_id]->begin(); it != (*first_data)[layer_id]->end(); ++it, ++subindex)
				{
					size_t single_size = it->size();
					std::vector<float> pack(single_size * data_list.size());

					std::vector<float>::iterator fill_it = pack.begin();
					for(std::vector<network_data_smart_ptr>::const_iterator sample_it = data_list.begin(); sample_it != data_list.end(); sample_it++)
					{
						const std::vector<float>& inp_buf = (*sample_it)->at(layer_id)->at(subindex);
						fill_it = std::copy(inp_buf.begin(), inp_buf.end(), fill_it);
					}

					buffer_list.push_back(cuda_linear_buffer_device_smart_ptr(new cuda_linear_buffer_device(
						&(*pack.begin()),
						pack.size() * sizeof(float),
						stream_id)));
				}
				res.push_back(buffer_list);
			}

			return res;
		}

		void network_updater_cuda::read_data(
			std::vector<std::vector<cuda_linear_buffer_device_smart_ptr> >& data_list,
			std::vector<network_data_smart_ptr>& res,
			hipStream_t stream_id) const
		{
			const network_data_smart_ptr& first_data = res[0];
			unsigned int layer_id = testing_layer_count;
			for(std::vector<std::vector<cuda_linear_buffer_device_smart_ptr> >::iterator src_it = data_list.begin(); src_it != data_list.end(); ++src_it, ++layer_id)
			{
				unsigned int subindex = 0;
				for(std::vector<cuda_linear_buffer_device_smart_ptr>::iterator src_it2 = src_it->begin(); src_it2 != src_it->end(); ++src_it2, ++subindex)
				{
					cuda_linear_buffer_device_smart_ptr src = *src_it2;
					std::vector<float> pack(src->get_size() / sizeof(float));
					cuda_safe_call(hipMemcpyAsync(&(*pack.begin()), *src, pack.size() * sizeof(float), hipMemcpyDeviceToHost, stream_id));
					cuda_safe_call(hipStreamSynchronize(stream_id));

					std::vector<float>::const_iterator src_buf_it = pack.begin();
					for(std::vector<network_data_smart_ptr>::const_iterator sample_it = res.begin(); sample_it != res.end(); sample_it++)
					{
						std::vector<float>& dst_buf = (*sample_it)->at(layer_id)->at(subindex);
						std::copy(src_buf_it, src_buf_it + dst_buf.size(), dst_buf.begin());
						src_buf_it += dst_buf.size();
					}
				}
			}
		}

		void network_updater_cuda::update_buffers_configuration(
			buffer_cuda_size_configuration& buffer_configuration,
			unsigned int updater_entry_count) const
		{
			buffer_configuration.add_constant_buffer(scale_addition->get_size());
			buffer_configuration.add_constant_buffer(scale_multiplication->get_size());

			for(std::vector<std::vector<const_cuda_linear_buffer_device_smart_ptr> >::const_iterator it = testing_schema_data.begin(); it != testing_schema_data.end(); ++it)
				for(std::vector<const_cuda_linear_buffer_device_smart_ptr>::const_iterator it2 = it->begin(); it2 != it->end(); ++it2)
					buffer_configuration.add_constant_buffer((*it2)->get_size());

			for(std::vector<layer_tester_cuda_smart_ptr>::const_iterator it = tester_list.begin(); it != tester_list.end(); ++it)
				(*it)->update_buffer_configuration(buffer_configuration);

			for(std::vector<std::vector<const_cuda_linear_buffer_device_smart_ptr> >::const_iterator it = updater_schema_data.begin(); it != updater_schema_data.end(); ++it)
				for(std::vector<const_cuda_linear_buffer_device_smart_ptr>::const_iterator it2 = it->begin(); it2 != it->end(); ++it2)
					buffer_configuration.add_constant_buffer((*it2)->get_size());

			for(std::vector<layer_updater_cuda_smart_ptr>::const_iterator it = updater_list.begin(); it != updater_list.end(); ++it)
				(*it)->update_buffer_configuration(buffer_configuration, updater_entry_count);
		}

		unsigned int network_updater_cuda::get_max_batch_size() const
		{
			buffer_cuda_size_configuration buffer_configuration;

			for(std::vector<layer_updater_cuda_smart_ptr>::const_iterator it = updater_list.begin(); it != updater_list.end(); ++it)
				(*it)->update_buffer_configuration(buffer_configuration);

			return cuda_config->get_max_entry_count(buffer_configuration, 0.5F);
		}
	}
}
