#include "hip/hip_runtime.h"
/*
 *  Copyright 2011-2014 Maxim Milakov
 *
 *  Licensed under the Apache License, Version 2.0 (the "License");
 *  you may not use this file except in compliance with the License.
 *  You may obtain a copy of the License at
 *
 *      http://www.apache.org/licenses/LICENSE-2.0
 *
 *  Unless required by applicable law or agreed to in writing, software
 *  distributed under the License is distributed on an "AS IS" BASIS,
 *  WITHOUT WARRANTIES OR CONDITIONS OF ANY KIND, either express or implied.
 *  See the License for the specific language governing permissions and
 *  limitations under the License.
 */

#include "network_updater_cuda.h"

#include "neural_network_cuda_exception.h"
#include "layer_testing_schema_factory.h"
#include "cuda_linear_buffer_device.h"
#include "cuda_linear_buffer_host.h"
#include "util_cuda.h"
#include "cuda_event.h"
#include "layer_updater_schema_factory.h"
#include "weight_vector_bound_cuda_factory.h"
#include "supervised_data_reader_async_helper.h"

#include <hip/hip_runtime.h>
#include <boost/format.hpp>
#include <stack>

__global__ void convert_compacted_to_raw_upd_kernel(
	const uchar4 * __restrict input,
	float4 * __restrict output,
	int elem_count)
{
	int elem_id = blockDim.x * (blockIdx.y * gridDim.x + blockIdx.x) + threadIdx.x;
	if (elem_id < elem_count)
	{
		uchar4 inp = input[elem_id];
		float4 val;
		val.x = inp.x * (1.0F / 255.0F);
		val.y = inp.y * (1.0F / 255.0F);
		val.z = inp.z * (1.0F / 255.0F);
		val.w = inp.w * (1.0F / 255.0F);
		output[elem_id] = val;
	}
}

__global__ void compute_error_upd_kernel(
	float * __restrict errors,
	float * __restrict mse,
	const float * __restrict actual_output_neurons,
	const float * __restrict predicted_output_neurons,
	bool is_squared_hinge_loss,
	int output_entry_id,
	int output_elem_count,
	int updater_entry_count)
{
	int elem_id = blockIdx.x * blockDim.x + threadIdx.x;
	int updater_entry_id = blockIdx.y * blockDim.y + threadIdx.y;
	bool in_bounds = (elem_id < output_elem_count) && (updater_entry_id < updater_entry_count);
	if (in_bounds)
	{
		int offset = updater_entry_id * output_elem_count + elem_id;
		float actual_val = actual_output_neurons[output_entry_id * output_elem_count + elem_id];
		float predicted_val = predicted_output_neurons[offset];
		float err = 0.0F;
		{
			if (!is_squared_hinge_loss || ((actual_val > 0.0F) && (predicted_val < actual_val)) || ((actual_val <= 0.0F) && (predicted_val > actual_val)))
				err = actual_val - predicted_val;
		}
		errors[offset] = err;
		mse[offset] += err * err * 0.5F;
	}
}

__global__ void dropout_kernel(
	float * __restrict neurons,
	const float * __restrict random_buf,
	float dropout_rate,
	int offset,
	unsigned int mask,
	int elem_count)
{
	int elem_id = blockDim.x * (blockIdx.y * gridDim.x + blockIdx.x) + threadIdx.x;
	if (elem_id < elem_count)
	{
		unsigned int random_elem_id = (elem_id + offset) & mask;
		if (random_buf[random_elem_id] < dropout_rate)
			neurons[elem_id] = 0.0F;
	}
}

namespace nnforge
{
	namespace cuda
	{
		unsigned int network_updater_cuda::max_entry_count_in_single_batch = 1024;

		network_updater_cuda::network_updater_cuda(
			network_schema_smart_ptr schema,
			bool is_squared_hinge_loss,
			const std::map<unsigned int, float>& layer_to_dropout_rate_map,
			const std::map<unsigned int, weight_vector_bound>& layer_to_weight_vector_bound_map,
			cuda_running_configuration_const_smart_ptr cuda_config)
			: network_updater(schema, is_squared_hinge_loss, layer_to_dropout_rate_map, layer_to_weight_vector_bound_map)
			, cuda_config(cuda_config)
		{
			const const_layer_list& layer_list = *schema;

			testing_layer_count = 0;
			start_layer_nonempty_weights_iterator = layer_list.begin();
			for(const_layer_list::const_iterator it = layer_list.begin(); it != layer_list.end(); ++it)
			{
				start_layer_nonempty_weights_iterator = it;

				if (!(*it)->is_empty_data())
					break;

				testing_layer_count++;
			}

			for(const_layer_list::const_iterator it = layer_list.begin(); it != start_layer_nonempty_weights_iterator; ++it)
				testing_schemas.push_back(single_layer_testing_schema_factory::get_const_instance().create_testing_schema_layer(*it, cuda_config));

			for(const_layer_list::const_iterator it = start_layer_nonempty_weights_iterator; it != layer_list.end(); ++it)
				updater_schemas.push_back(single_layer_updater_schema_factory::get_const_instance().create_updater_schema_layer(*it, cuda_config));

			for(std::map<unsigned int, weight_vector_bound>::const_iterator it = this->layer_to_weight_vector_bound_map.begin(); it != this->layer_to_weight_vector_bound_map.end(); ++it)
			{
				unsigned int layer_id = it->first;
				if (layer_id < testing_layer_count)
					throw neural_network_exception((boost::format("Weight vector bound is specified fo layer %1% while it is in testing part (consisting of %2% layers) of the updater") % layer_id  % testing_layer_count).str());

				weight_vector_bounds.insert(std::make_pair(layer_id, single_weight_vector_bound_factory::get_const_instance().create_weight_vector_bound(layer_list[layer_id], cuda_config)));
			}

			setup_network_cuda();

			for(const_layer_testing_schema_list::const_iterator it = testing_schemas.begin(); it != testing_schemas.end(); ++it)
				testing_schema_data.push_back((*it)->get_schema_buffers());

			for(const_layer_updater_schema_list::const_iterator it = updater_schemas.begin(); it != updater_schemas.end(); ++it)
				updater_schema_data.push_back((*it)->get_schema_buffers());
		}

		network_updater_cuda::~network_updater_cuda()
		{
		}

		void network_updater_cuda::setup_network_cuda()
		{
			command_stream = cuda_stream_smart_ptr(new hip_stream());
			data_stream = cuda_stream_smart_ptr(new hip_stream());
		}

		std::vector<testing_result_smart_ptr> network_updater_cuda::actual_update(
			supervised_data_reader& reader,
			const std::vector<network_data_smart_ptr>& training_speed_vector_list,
			std::vector<network_data_smart_ptr>& data_list)
		{
			std::vector<testing_result_smart_ptr> res;

			reader.reset();

			layer_configuration_specific input_configuration = reader.get_input_configuration();
			layer_configuration_specific output_configuration = reader.get_output_configuration();

			unsigned int input_neuron_count = input_configuration.get_neuron_count();
			unsigned int output_neuron_count = output_configuration.get_neuron_count();
			unsigned int input_neuron_count_per_feature_map = input_configuration.get_neuron_count_per_feature_map();
			neuron_data_type::input_type type_code = reader.get_input_type();
			size_t input_neuron_elem_size = reader.get_input_neuron_elem_size();

			unsigned int updater_entry_count = static_cast<unsigned int>(data_list.size());
			if (updater_entry_count == 0)
				return res;

			for(unsigned int i = 0; i < training_speed_vector_list.size(); ++i)
				res.push_back(testing_result_smart_ptr(new testing_result(is_squared_hinge_loss, output_neuron_count)));

			std::vector<std::vector<cuda_linear_buffer_device_smart_ptr> > net_data = enqueue_get_data(data_list, *command_stream);
			std::vector<std::vector<const_cuda_linear_buffer_device_smart_ptr> > training_speed_data = enqueue_get_training_speed(training_speed_vector_list, *command_stream);

			buffer_cuda_size_configuration buffers_config;
			update_buffers_configuration(buffers_config, updater_entry_count);

			buffers_config.add_per_entry_linear_addressing_through_texture(layer_config_list[testing_layer_count].get_neuron_count()); // This is for the first updater to safely read input data through the texture
			buffers_config.add_per_entry_buffer(input_neuron_count * input_neuron_elem_size); // input
			buffers_config.add_per_entry_buffer(input_neuron_count * input_neuron_elem_size); // input
			buffers_config.add_per_entry_buffer(input_neuron_count * sizeof(float)); // converted input
			buffers_config.add_per_entry_buffer(output_neuron_count * sizeof(float)); // output
			buffers_config.add_per_entry_buffer(output_neuron_count * sizeof(float)); // output
			buffers_config.add_constant_buffer(output_neuron_count * sizeof(float) * updater_entry_count); // initial error
			buffers_config.add_constant_buffer(output_neuron_count * sizeof(float) * updater_entry_count); // mse
			if (!random_uniform_list.empty())
				buffers_config.add_constant_buffer(random_uniform_list.size() * sizeof(float)); // random_uniform_list

			for(std::vector<std::vector<cuda_linear_buffer_device_smart_ptr> >::const_iterator it = net_data.begin(); it != net_data.end(); ++it)
				for(std::vector<cuda_linear_buffer_device_smart_ptr>::const_iterator it2 = it->begin(); it2 != it->end(); ++it2)
					buffers_config.add_constant_buffer((*it2)->get_size());

			for(std::vector<std::vector<const_cuda_linear_buffer_device_smart_ptr> >::const_iterator it = training_speed_data.begin(); it != training_speed_data.end(); ++it)
				for(std::vector<const_cuda_linear_buffer_device_smart_ptr>::const_iterator it2 = it->begin(); it2 != it->end(); ++it2)
					buffers_config.add_constant_buffer((*it2)->get_size());

			unsigned int max_entry_count = std::min<unsigned int>(std::min<unsigned int>(cuda_config->get_max_entry_count(buffers_config), reader.get_entry_count()), max_entry_count_in_single_batch);

			cuda_linear_buffer_device_smart_ptr input_buf[2] = 
			{
				cuda_linear_buffer_device_smart_ptr(new cuda_linear_buffer_device(input_neuron_count * max_entry_count * input_neuron_elem_size)),
				cuda_linear_buffer_device_smart_ptr(new cuda_linear_buffer_device(input_neuron_count * max_entry_count * input_neuron_elem_size)),
			};

			cuda_linear_buffer_device_smart_ptr output_buf[2] = 
			{
				cuda_linear_buffer_device_smart_ptr(new cuda_linear_buffer_device(output_neuron_count * max_entry_count * sizeof(float))),
				cuda_linear_buffer_device_smart_ptr(new cuda_linear_buffer_device(output_neuron_count * max_entry_count * sizeof(float))),
			};

			cuda_linear_buffer_device_smart_ptr input_converted_buf(new cuda_linear_buffer_device(input_neuron_count * max_entry_count * sizeof(float)));

			cuda_linear_buffer_device_smart_ptr initial_error_buf(new cuda_linear_buffer_device(output_neuron_count * updater_entry_count * sizeof(float)));

			cuda_linear_buffer_device_smart_ptr mse_buf(new cuda_linear_buffer_device(output_neuron_count * updater_entry_count * sizeof(float)));

			cuda_linear_buffer_device_smart_ptr random_uniform_buf;
			if (!random_uniform_list.empty())
			{
				random_uniform_buf = cuda_linear_buffer_device_smart_ptr(new cuda_linear_buffer_device(random_uniform_list.size() * sizeof(float)));
				cuda_safe_call(hipMemcpyAsync(*random_uniform_buf, &(*random_uniform_list.begin()), random_uniform_list.size() * sizeof(float), hipMemcpyHostToDevice, *command_stream));
			}

			cuda_linear_buffer_device_smart_ptr output_buffer = input_converted_buf;
			std::vector<std::pair<cuda_linear_buffer_device_smart_ptr, std::vector<cuda_linear_buffer_device_smart_ptr> > > testing_input_and_additional_buffers_pack;
			for(std::vector<layer_tester_cuda_smart_ptr>::iterator it = tester_list.begin(); it != tester_list.end(); ++it)
			{
				std::vector<cuda_linear_buffer_device_smart_ptr> additional_buffers = (*it)->allocate_additional_buffers(max_entry_count);
				testing_input_and_additional_buffers_pack.push_back(std::make_pair(output_buffer, additional_buffers));
				output_buffer = (*it)->get_output_buffer(output_buffer, additional_buffers);
			}
			std::vector<std::pair<cuda_linear_buffer_device_smart_ptr, layer_updater_cuda::buffer_set> > updater_input_and_all_buffers_pack;
			for(std::vector<layer_updater_cuda_smart_ptr>::iterator it = updater_list.begin(); it != updater_list.end(); ++it)
			{
				layer_updater_cuda::buffer_set all_buffers = (*it)->allocate_all_buffers(updater_entry_count);
				updater_input_and_all_buffers_pack.push_back(std::make_pair(output_buffer, all_buffers));
				output_buffer = all_buffers.output_neurons_buffer;
			}

			std::vector<cuda_linear_buffer_device_smart_ptr> output_errors_buffers;
			cuda_linear_buffer_device_smart_ptr output_errors = initial_error_buf;
			for(std::vector<std::pair<cuda_linear_buffer_device_smart_ptr, layer_updater_cuda::buffer_set> >::reverse_iterator it = updater_input_and_all_buffers_pack.rbegin(); it != updater_input_and_all_buffers_pack.rend(); ++it)
			{
				output_errors_buffers.push_back(output_errors);
				layer_updater_cuda::buffer_set& all_buffers = it->second;

				if (all_buffers.input_errors_buffer != 0)
					output_errors = all_buffers.input_errors_buffer;
			}

			std::map<unsigned int, std::vector<cuda_linear_buffer_device_smart_ptr> > weight_vector_bound_buffers;
			for(std::map<unsigned int, weight_vector_bound_cuda_smart_ptr>::const_iterator it = weight_vector_bounds.begin(); it != weight_vector_bounds.end(); ++it)
				weight_vector_bound_buffers.insert(std::make_pair(it->first, it->second->allocate_additional_buffers(max_entry_count)));

			cuda_linear_buffer_host_smart_ptr input_host_buf(new cuda_linear_buffer_host(input_neuron_count * max_entry_count * input_neuron_elem_size));
			unsigned char * input = *input_host_buf;
			cuda_linear_buffer_host_smart_ptr output_host_buf(new cuda_linear_buffer_host(output_neuron_count * max_entry_count * sizeof(float)));
			float * output = *output_host_buf;

			// zero mse
			cuda_util::set_with_value(
				*cuda_config,
				*mse_buf,
				0.0F,
				output_neuron_count * updater_entry_count,
				*command_stream);

			unsigned int current_data_slot = 0;
			unsigned int current_command_slot = 1;
			unsigned int entries_available_for_copy_in_count = reader.get_entry_count();
			unsigned int entries_available_for_processing_count = 0;
			cuda_event data_processed_event;
			cuda_event input_copied_event;
			if (cuda_config->is_flush_required())
			{
				cuda_safe_call(hipEventRecord(data_processed_event, *command_stream));
				cuda_safe_call(hipEventQuery(data_processed_event));
			}

			random_generator gen = rnd::get_random_generator();
			std::tr1::uniform_int<unsigned int> dist(0, static_cast<unsigned int>(random_uniform_list.size() - 1));
			unsigned int mask = static_cast<unsigned int>(random_uniform_list.size() - 1);
			while((entries_available_for_copy_in_count > 0) || (entries_available_for_processing_count > 0))
			{
				supervised_data_reader_async_helper async_reader;
				if (entries_available_for_copy_in_count > 0)
				{
					unsigned int entries_to_read_count = std::min<unsigned int>(max_entry_count, entries_available_for_copy_in_count);
					async_reader.fun = supervised_data_reader_functor(
						entries_to_read_count,
						&reader,
						input,
						output,
						*(input_buf[current_data_slot]),
						*(output_buf[current_data_slot]),
						*data_stream);
					async_reader.start();
				}

				if (entries_available_for_processing_count > 0)
				{
					// Convert input
					if (type_code == neuron_data_type::type_byte)
					{
						int elem_count = (input_neuron_count * entries_available_for_processing_count + 3) / 4;
						std::pair<dim3, dim3> kernel_dims = cuda_util::get_grid_and_threadblock_sizes_sequential_access(
							*cuda_config,
							elem_count);
						convert_compacted_to_raw_upd_kernel<<<kernel_dims.first, kernel_dims.second, 0, *command_stream>>>(
							*input_buf[current_command_slot],
							*input_converted_buf,
							elem_count);
					}
					else if (type_code == neuron_data_type::type_float)
					{
						cuda_safe_call(hipMemcpyAsync(
							*input_converted_buf,
							*input_buf[current_command_slot],
							input_neuron_count * entries_available_for_processing_count * sizeof(float),
							hipMemcpyDeviceToDevice,
							*command_stream));
					}
					else throw neural_network_exception((boost::format("actual_update cannot handle input neurons of type %1%") % type_code).str());

					// Run ann
					{
						std::vector<std::pair<cuda_linear_buffer_device_smart_ptr, std::vector<cuda_linear_buffer_device_smart_ptr> > >::iterator input_and_additional_buffers_pack_it = testing_input_and_additional_buffers_pack.begin();
						std::vector<std::vector<const_cuda_linear_buffer_device_smart_ptr> >::iterator schema_data_it = testing_schema_data.begin();
						unsigned int layer_id = 0;
						layer_configuration_specific_list::const_iterator layer_config_it = layer_config_list.begin();
						for(std::vector<layer_tester_cuda_smart_ptr>::iterator it = tester_list.begin(); it != tester_list.end(); ++it, ++input_and_additional_buffers_pack_it, ++schema_data_it, ++layer_id, ++layer_config_it)
						{
							std::map<unsigned int, float>::const_iterator dropout_it = layer_to_dropout_rate_map.find(layer_id);
							if (dropout_it != layer_to_dropout_rate_map.end())
							{
								unsigned int offset = dist(gen);
								enqueue_dropout(
									*command_stream,
									random_uniform_buf,
									input_and_additional_buffers_pack_it->first,
									dropout_it->second,
									mask,
									entries_available_for_processing_count * layer_config_it->get_neuron_count(),
									offset);
							}

							(*it)->enqueue_test(
								*command_stream,
								*schema_data_it,
								std::vector<const_cuda_linear_buffer_device_smart_ptr>(),
								input_and_additional_buffers_pack_it->first,
								input_and_additional_buffers_pack_it->second,
								entries_available_for_processing_count);
						}
					}

					// Apply dropout to the input of the first updater layer
					{
						std::map<unsigned int, float>::const_iterator dropout_it = layer_to_dropout_rate_map.find(testing_layer_count);
						if (dropout_it != layer_to_dropout_rate_map.end())
						{
							unsigned int offset = dist(gen);
							enqueue_dropout(
								*command_stream,
								random_uniform_buf,
								updater_input_and_all_buffers_pack[0].first,
								dropout_it->second,
								mask,
								entries_available_for_processing_count * layer_config_list[testing_layer_count].get_neuron_count(),
								offset);
						}
					}

					for(unsigned int input_entry_id = 0; input_entry_id < entries_available_for_processing_count; ++input_entry_id)
					{
						std::stack<unsigned int> offset_list;

						// Forward updater
						{
							std::vector<std::pair<cuda_linear_buffer_device_smart_ptr, layer_updater_cuda::buffer_set> >::iterator input_and_all_buffers_pack_it = updater_input_and_all_buffers_pack.begin();
							std::vector<std::vector<cuda_linear_buffer_device_smart_ptr> >::iterator net_data_it = net_data.begin();
							std::vector<std::vector<const_cuda_linear_buffer_device_smart_ptr> >::iterator schema_data_it = updater_schema_data.begin();
							unsigned int layer_id = testing_layer_count;
							layer_configuration_specific_list::const_iterator layer_config_it = layer_config_list.begin() + testing_layer_count;
							for(std::vector<layer_updater_cuda_smart_ptr>::iterator it = updater_list.begin(); it != updater_list.end(); ++it, ++input_and_all_buffers_pack_it, ++schema_data_it, ++net_data_it, ++layer_id, ++layer_config_it)
							{
								if (it != updater_list.begin())
								{
									std::map<unsigned int, float>::const_iterator dropout_it = layer_to_dropout_rate_map.find(layer_id);
									if (dropout_it != layer_to_dropout_rate_map.end())
									{
										unsigned int offset = dist(gen);
										offset_list.push(offset);
										enqueue_dropout(
											*command_stream,
											random_uniform_buf,
											input_and_all_buffers_pack_it->first,
											dropout_it->second,
											mask,
											updater_entry_count * layer_config_it->get_neuron_count(),
											offset);
									}
								}

								(*it)->enqueue_test(
									it == updater_list.begin() ? input_entry_id : 0,
									*command_stream,
									*schema_data_it,
									*net_data_it,
									input_and_all_buffers_pack_it->first,
									input_and_all_buffers_pack_it->second.output_neurons_buffer,
									input_and_all_buffers_pack_it->second.additional_buffers,
									input_and_all_buffers_pack_it->second.dynamic_memobjects,
									updater_entry_count);
							}
						}

						// Compute errors
						{
							std::pair<dim3, dim3> kernel_dims = cuda_util::get_grid_and_threadblock_sizes_sequential_access(
								*cuda_config,
								output_neuron_count,
								updater_entry_count,
								1);
							compute_error_upd_kernel<<<kernel_dims.first, kernel_dims.second, 0, *command_stream>>>(
								*initial_error_buf,
								*mse_buf,
								*output_buf[current_command_slot],
								*output_buffer,
								is_squared_hinge_loss,
								input_entry_id,
								output_neuron_count,
								updater_entry_count);
						}

						// Backward updater
						{
							std::vector<cuda_linear_buffer_device_smart_ptr>::iterator output_errors_it = output_errors_buffers.begin();
							std::vector<std::pair<cuda_linear_buffer_device_smart_ptr, layer_updater_cuda::buffer_set> >::reverse_iterator input_and_all_buffers_pack_it = updater_input_and_all_buffers_pack.rbegin();
							std::vector<std::vector<cuda_linear_buffer_device_smart_ptr> >::reverse_iterator net_data_it = net_data.rbegin();
							std::vector<std::vector<const_cuda_linear_buffer_device_smart_ptr> >::reverse_iterator training_speed_data_it = training_speed_data.rbegin();
							std::vector<std::vector<const_cuda_linear_buffer_device_smart_ptr> >::reverse_iterator schema_data_it = updater_schema_data.rbegin();
							unsigned int reverse_layer_id = static_cast<unsigned int>(updater_list.size() + testing_layer_count) - 1;
							layer_configuration_specific_list::const_reverse_iterator layer_config_it = layer_config_list.rbegin() + 1;
							for(std::vector<layer_updater_cuda_smart_ptr>::reverse_iterator it = updater_list.rbegin(); it != updater_list.rend(); ++it, ++input_and_all_buffers_pack_it, ++schema_data_it, ++training_speed_data_it, ++output_errors_it, ++net_data_it, --reverse_layer_id, ++layer_config_it)
							{
								if (it != (updater_list.rend() - 1))
								{
									(*it)->enqueue_backprop(
										*command_stream,
										*schema_data_it,
										*net_data_it,
										input_and_all_buffers_pack_it->second.output_neurons_buffer,
										input_and_all_buffers_pack_it->first,
										*output_errors_it,
										input_and_all_buffers_pack_it->second.input_errors_buffer,
										input_and_all_buffers_pack_it->second.additional_buffers,
										input_and_all_buffers_pack_it->second.dynamic_memobjects,
										updater_entry_count);

									std::map<unsigned int, float>::const_iterator dropout_it = layer_to_dropout_rate_map.find(reverse_layer_id);
									if (dropout_it != layer_to_dropout_rate_map.end())
									{
										unsigned int offset = offset_list.top();
										offset_list.pop();
										enqueue_dropout(
											*command_stream,
											random_uniform_buf,
											(input_and_all_buffers_pack_it->second.input_errors_buffer == 0) ? *output_errors_it : input_and_all_buffers_pack_it->second.input_errors_buffer,
											dropout_it->second,
											mask,
											updater_entry_count * layer_config_it->get_neuron_count(),
											offset);
									}
								}

								(*it)->enqueue_update_weights(
									(it == (updater_list.rend() - 1)) ? input_entry_id : 0,
									*command_stream,
									*net_data_it,
									*schema_data_it,
									*training_speed_data_it,
									*output_errors_it,
									input_and_all_buffers_pack_it->first,
									input_and_all_buffers_pack_it->second.additional_buffers,
									input_and_all_buffers_pack_it->second.dynamic_memobjects,
									updater_entry_count);

								weight_vector_bound_map::iterator bound_it = weight_vector_bounds.find(reverse_layer_id);
								if (bound_it != weight_vector_bounds.end())
								{
									const weight_vector_bound& bound = layer_to_weight_vector_bound_map.find(reverse_layer_id)->second;
									const std::vector<cuda_linear_buffer_device_smart_ptr>& additional_buffers = weight_vector_bound_buffers.find(reverse_layer_id)->second;
									bound_it->second->enqueue_normalize_weights(
										*command_stream,
										bound,
										*net_data_it,
										additional_buffers,
										updater_entry_count);
								}
							}
						}

						if (((input_entry_id % 16) == 1) && cuda_config->is_flush_required())
						{
							cuda_safe_call(hipEventRecord(data_processed_event, *command_stream));
							cuda_safe_call(hipEventQuery(data_processed_event));
						}
					} // for(unsigned int input_entry_id

					for(std::vector<testing_result_smart_ptr>::iterator it = res.begin(); it != res.end(); ++it)
						(*it)->entry_count += entries_available_for_processing_count;

					if (cuda_config->is_flush_required())
					{
						cuda_safe_call(hipEventRecord(data_processed_event, *command_stream));
						cuda_safe_call(hipEventQuery(data_processed_event));
					}
				} // if (entries_available_for_processing_count > 0)

				unsigned int entries_read_count = 0;
				if (entries_available_for_copy_in_count > 0)
					entries_read_count = async_reader.wait();

				cuda_safe_call(hipStreamSynchronize(*data_stream));
				cuda_safe_call(hipStreamSynchronize(*command_stream));

				entries_available_for_processing_count = entries_read_count;
				entries_available_for_copy_in_count -= entries_read_count;

				current_data_slot = 1 - current_data_slot;
				current_command_slot = 1 - current_command_slot;
			}

			read_data(net_data, data_list, *command_stream);

			std::vector<float> mse_list(output_neuron_count * updater_entry_count);
			cuda_safe_call(hipMemcpyAsync(&(*mse_list.begin()), *mse_buf, mse_list.size() * sizeof(float), hipMemcpyDeviceToHost, *command_stream));
			cuda_safe_call(hipStreamSynchronize(*command_stream));

			for(unsigned int i = 0; i < updater_entry_count; ++i)
				std::copy(mse_list.begin() + output_neuron_count * i, mse_list.begin() + output_neuron_count * (i + 1), res[i]->cumulative_mse_list.begin());

			return res;
		}

		void network_updater_cuda::layer_config_list_modified()
		{
			layer_configuration_specific_list::const_iterator it_conf = layer_config_list.begin();

			tester_list.clear();
			for(const_layer_testing_schema_list::const_iterator it = testing_schemas.begin(); it != testing_schemas.end(); ++it, ++it_conf)
			{
				tester_list.push_back(
					(*it)->create_tester(
						*it_conf,
						*(it_conf + 1)));
			}

			updater_list.clear();
			for(const_layer_updater_schema_list::const_iterator it = updater_schemas.begin(); it != updater_schemas.end(); ++it, ++it_conf)
			{
				updater_list.push_back(
					(*it)->create_updater(
						*it_conf,
						*(it_conf + 1),
						(it_conf > layer_config_list.begin() + testing_layer_count),
						(it_conf > layer_config_list.begin() + testing_layer_count)));
			}
		}

		std::vector<std::vector<const_cuda_linear_buffer_device_smart_ptr> > network_updater_cuda::enqueue_get_training_speed(
			const std::vector<network_data_smart_ptr>& training_speed_list,
			hipStream_t stream_id) const
		{
			std::vector<std::vector<const_cuda_linear_buffer_device_smart_ptr> > res;

			const network_data_smart_ptr& first_data = training_speed_list[0];

			for(unsigned int layer_id = testing_layer_count; layer_id < updater_schemas.size() + testing_layer_count; ++layer_id)
			{
				std::vector<const_cuda_linear_buffer_device_smart_ptr> buffer_list;
				unsigned int subindex = 0;
				for(std::vector<std::vector<float> >::iterator it = (*first_data)[layer_id]->begin(); it != (*first_data)[layer_id]->end(); ++it, ++subindex)
				{
					size_t single_size = it->size();
					std::vector<float> pack(single_size * training_speed_list.size());

					std::vector<float>::iterator fill_it = pack.begin();
					for(std::vector<network_data_smart_ptr>::const_iterator sample_it = training_speed_list.begin(); sample_it != training_speed_list.end(); sample_it++)
					{
						const std::vector<float>& inp_buf = (*sample_it)->at(layer_id)->at(subindex);
						fill_it = std::copy(inp_buf.begin(), inp_buf.end(), fill_it);
					}

					buffer_list.push_back(cuda_linear_buffer_device_smart_ptr(new cuda_linear_buffer_device(
						&(*pack.begin()),
						pack.size() * sizeof(float),
						stream_id)));
				}
				res.push_back(buffer_list);
			}

			return res;
		}

		std::vector<std::vector<cuda_linear_buffer_device_smart_ptr> > network_updater_cuda::enqueue_get_data(
			const std::vector<network_data_smart_ptr>& data_list,
			hipStream_t stream_id) const
		{
			std::vector<std::vector<cuda_linear_buffer_device_smart_ptr> > res;

			const network_data_smart_ptr& first_data = data_list[0];

			for(unsigned int layer_id = testing_layer_count; layer_id < updater_schemas.size() + testing_layer_count; ++layer_id)
			{
				std::vector<cuda_linear_buffer_device_smart_ptr> buffer_list;
				unsigned int subindex = 0;
				for(std::vector<std::vector<float> >::iterator it = (*first_data)[layer_id]->begin(); it != (*first_data)[layer_id]->end(); ++it, ++subindex)
				{
					size_t single_size = it->size();
					std::vector<float> pack(single_size * data_list.size());

					std::vector<float>::iterator fill_it = pack.begin();
					for(std::vector<network_data_smart_ptr>::const_iterator sample_it = data_list.begin(); sample_it != data_list.end(); sample_it++)
					{
						const std::vector<float>& inp_buf = (*sample_it)->at(layer_id)->at(subindex);
						fill_it = std::copy(inp_buf.begin(), inp_buf.end(), fill_it);
					}

					buffer_list.push_back(cuda_linear_buffer_device_smart_ptr(new cuda_linear_buffer_device(
						&(*pack.begin()),
						pack.size() * sizeof(float),
						stream_id)));
				}
				res.push_back(buffer_list);
			}

			return res;
		}

		void network_updater_cuda::read_data(
			std::vector<std::vector<cuda_linear_buffer_device_smart_ptr> >& data_list,
			std::vector<network_data_smart_ptr>& res,
			hipStream_t stream_id) const
		{
			const network_data_smart_ptr& first_data = res[0];
			unsigned int layer_id = testing_layer_count;
			for(std::vector<std::vector<cuda_linear_buffer_device_smart_ptr> >::iterator src_it = data_list.begin(); src_it != data_list.end(); ++src_it, ++layer_id)
			{
				unsigned int subindex = 0;
				for(std::vector<cuda_linear_buffer_device_smart_ptr>::iterator src_it2 = src_it->begin(); src_it2 != src_it->end(); ++src_it2, ++subindex)
				{
					cuda_linear_buffer_device_smart_ptr src = *src_it2;
					std::vector<float> pack(src->get_size() / sizeof(float));
					cuda_safe_call(hipMemcpyAsync(&(*pack.begin()), *src, pack.size() * sizeof(float), hipMemcpyDeviceToHost, stream_id));
					cuda_safe_call(hipStreamSynchronize(stream_id));

					std::vector<float>::const_iterator src_buf_it = pack.begin();
					for(std::vector<network_data_smart_ptr>::const_iterator sample_it = res.begin(); sample_it != res.end(); sample_it++)
					{
						std::vector<float>& dst_buf = (*sample_it)->at(layer_id)->at(subindex);
						std::copy(src_buf_it, src_buf_it + dst_buf.size(), dst_buf.begin());
						src_buf_it += dst_buf.size();
					}
				}
			}
		}

		void network_updater_cuda::update_buffers_configuration(
			buffer_cuda_size_configuration& buffer_configuration,
			unsigned int updater_entry_count) const
		{
			for(std::vector<std::vector<const_cuda_linear_buffer_device_smart_ptr> >::const_iterator it = testing_schema_data.begin(); it != testing_schema_data.end(); ++it)
				for(std::vector<const_cuda_linear_buffer_device_smart_ptr>::const_iterator it2 = it->begin(); it2 != it->end(); ++it2)
					buffer_configuration.add_constant_buffer((*it2)->get_size());

			for(std::vector<layer_tester_cuda_smart_ptr>::const_iterator it = tester_list.begin(); it != tester_list.end(); ++it)
				(*it)->update_buffer_configuration(buffer_configuration);

			for(std::vector<std::vector<const_cuda_linear_buffer_device_smart_ptr> >::const_iterator it = updater_schema_data.begin(); it != updater_schema_data.end(); ++it)
				for(std::vector<const_cuda_linear_buffer_device_smart_ptr>::const_iterator it2 = it->begin(); it2 != it->end(); ++it2)
					buffer_configuration.add_constant_buffer((*it2)->get_size());

			for(std::vector<layer_updater_cuda_smart_ptr>::const_iterator it = updater_list.begin(); it != updater_list.end(); ++it)
				(*it)->update_buffer_configuration(buffer_configuration, updater_entry_count);

			for(std::map<unsigned int, weight_vector_bound_cuda_smart_ptr>::const_iterator it = weight_vector_bounds.begin(); it != weight_vector_bounds.end(); ++it)
				it->second->update_buffer_configuration(buffer_configuration, updater_entry_count);
		}

		unsigned int network_updater_cuda::get_max_batch_size() const
		{
			buffer_cuda_size_configuration buffer_configuration;

			for(std::vector<layer_updater_cuda_smart_ptr>::const_iterator it = updater_list.begin(); it != updater_list.end(); ++it)
				(*it)->update_buffer_configuration(buffer_configuration);

			for(std::map<unsigned int, weight_vector_bound_cuda_smart_ptr>::const_iterator it = weight_vector_bounds.begin(); it != weight_vector_bounds.end(); ++it)
				it->second->update_buffer_configuration(buffer_configuration);

			return cuda_config->get_max_entry_count(buffer_configuration, 0.5F);
		}

		void network_updater_cuda::enqueue_dropout(
			hipStream_t stream_id,
			const_cuda_linear_buffer_device_smart_ptr random_buffer,
			cuda_linear_buffer_device_smart_ptr target_buffer,
			float dropout_rate,
			unsigned int mask,
			unsigned int elem_count,
			unsigned int offset_in_random_list)
		{
			std::pair<dim3, dim3> kernel_dims = cuda_util::get_grid_and_threadblock_sizes_sequential_access(
				*cuda_config,
				elem_count);
			dropout_kernel<<<kernel_dims.first, kernel_dims.second, 0, stream_id>>>(
				*target_buffer,
				*random_buffer,
				dropout_rate,
				offset_in_random_list,
				mask,
				elem_count);
		}
	}
}
