#include "hip/hip_runtime.h"
/*
 *  Copyright 2011-2014 Maxim Milakov
 *
 *  Licensed under the Apache License, Version 2.0 (the "License");
 *  you may not use this file except in compliance with the License.
 *  You may obtain a copy of the License at
 *
 *      http://www.apache.org/licenses/LICENSE-2.0
 *
 *  Unless required by applicable law or agreed to in writing, software
 *  distributed under the License is distributed on an "AS IS" BASIS,
 *  WITHOUT WARRANTIES OR CONDITIONS OF ANY KIND, either express or implied.
 *  See the License for the specific language governing permissions and
 *  limitations under the License.
 */

#include "convolution_3d_layer_updater_cuda_kepler.h"

#include <hip/hip_runtime.h>

#include <boost/format.hpp>

#include "util_cuda.h"
#include "neural_network_cuda_exception.h"
#include "cuda_texture.h"
#include "packed_config.h"
#include "space_filling_curve.h"

#include "../convolution_layer.h"

#define FEATURE_MAP_BLOCK_SIZE 4
#define WINDOW_WIDTH_LOCAL 4

namespace nnforge
{
	namespace cuda
	{
		template<int BLOCK_SIZE, bool single_input_feature_map_group>
		__global__ void convolution_3d_tex_upd_kernel_kepler(
			float * __restrict output,
			hipTextureObject_t input_tex,
			hipTextureObject_t weights_tex,
			const float * __restrict biases,
			const packed_config<5> * __restrict packed_config_list,
			int output_width,
			int output_height,
			int output_depth,
			int input_width,
			int input_height,
			int input_depth,
			int window_width,
			int window_height,
			int window_depth,
			int input_feature_map_count,
			int output_feature_map_count,
			int input_feature_map_group_size,
			int texture_offset,
			int entry_count,
			bool different_input,
			int packed_config_count)
		{
			int packed_config_id = blockIdx.x * blockDim.x + threadIdx.x;
			int entry_id = blockIdx.y * blockDim.y + threadIdx.y;

			bool in_bounds = (entry_id < entry_count) && (packed_config_id < packed_config_count);
			if (in_bounds)
			{
				packed_config<5> conf = packed_config_list[packed_config_id];
				int x = conf.get_val(0);
				int y = conf.get_val(1);
				int z = conf.get_val(2);
				int output_feature_map_id = conf.get_val(3);
				int base_input_feature_map_id = conf.get_val(4);

				int weight_count_per_output_feature_map = window_depth * window_height * window_width * input_feature_map_count;
				int input_elem_id = ((((different_input ? entry_id * input_feature_map_count : 0) + base_input_feature_map_id) * input_depth + z) * input_height + y) * input_width + x + texture_offset;
				int weights_offset = ((entry_id * output_feature_map_count + output_feature_map_id) * input_feature_map_count + base_input_feature_map_id) * window_depth * window_height * window_width;
				int iteration_count = min(input_feature_map_group_size, input_feature_map_count - base_input_feature_map_id);

				float initial_values[FEATURE_MAP_BLOCK_SIZE];
				#pragma unroll
				for(int i = 0; i < FEATURE_MAP_BLOCK_SIZE; ++i)
					initial_values[i] = 0.0F;
				if (base_input_feature_map_id == 0)
				{
					#pragma unroll
					for(int i = 0; i < FEATURE_MAP_BLOCK_SIZE; ++i)
						if (i < output_feature_map_count - output_feature_map_id)
							initial_values[i] = biases[entry_id * output_feature_map_count + output_feature_map_id + i];
				}
				float sums[BLOCK_SIZE * FEATURE_MAP_BLOCK_SIZE];
				#pragma unroll
				for(int i = 0; i < FEATURE_MAP_BLOCK_SIZE; ++i)
					#pragma unroll
					for(int j = 0; j < BLOCK_SIZE; ++j)
						sums[i * BLOCK_SIZE + j] = initial_values[i];

				for(int i = 0; i < iteration_count; ++i)
				{
					for(int input_z = 0; input_z < window_depth; ++input_z)
					{
						for(int input_y = 0; input_y < window_height; ++input_y)
						{
							#pragma unroll 4
							for(int input_x = 0; input_x < window_width; ++input_x)
							{
								float weight_list[FEATURE_MAP_BLOCK_SIZE];
								#pragma unroll
								for(int i = 0; i < FEATURE_MAP_BLOCK_SIZE; ++i)
									weight_list[i] = tex1Dfetch<float>(weights_tex, weights_offset + weight_count_per_output_feature_map * i);
								#pragma unroll
								for(int j = 0; j < BLOCK_SIZE; ++j)
								{
									float inp = tex1Dfetch<float>(input_tex, input_elem_id + j); 
									#pragma unroll
									for(int i = 0; i < FEATURE_MAP_BLOCK_SIZE; ++i)
										sums[i * BLOCK_SIZE + j] += inp * weight_list[i];
								}
								weights_offset++;
								input_elem_id++;
							} // for input_x
							input_elem_id += input_width - window_width;
						} // for input_y
						input_elem_id += input_width * (input_height - window_height);
					} // for input_z
					input_elem_id += input_height * input_width * (input_depth - window_depth);
				}

				float * base_output = output + (((entry_id * output_feature_map_count + output_feature_map_id) * output_depth + z) * output_height + y) * output_width + x;
				int output_neuron_count_per_feature_map = output_depth * output_height * output_width;
				if (single_input_feature_map_group)
				{
					#pragma unroll
					for(int i = 0; i < FEATURE_MAP_BLOCK_SIZE; ++i)
					{
						if (i < output_feature_map_count - output_feature_map_id)
						{
							#pragma unroll
							for(int j = 0; j < BLOCK_SIZE; ++j)
							{
								if (j < output_width - x)
									base_output[output_neuron_count_per_feature_map * i + j] = sums[i * BLOCK_SIZE + j];
							}
						}
					}
				}
				else
				{
					#pragma unroll
					for(int i = 0; i < FEATURE_MAP_BLOCK_SIZE; ++i)
					{
						if (i < output_feature_map_count - output_feature_map_id)
						{
							#pragma unroll
							for(int j = 0; j < BLOCK_SIZE; ++j)
							{
								if (j < output_width - x)
									atomicAdd(base_output + output_neuron_count_per_feature_map * i + j, sums[i * BLOCK_SIZE + j]);
							}
						}
					}
				}
			}
		}

		template<int WINDOW_WIDTH, int BLOCK_SIZE, bool single_input_feature_map_group>
		__global__ void convolution_3d_tex_exact_upd_kernel_kepler(
			float * __restrict output,
			hipTextureObject_t input_tex,
			hipTextureObject_t weights_tex,
			const float * __restrict biases,
			const packed_config<5> * __restrict packed_config_list,
			int output_width,
			int output_height,
			int output_depth,
			int input_width,
			int input_height,
			int input_depth,
			int window_height,
			int window_depth,
			int input_feature_map_count,
			int output_feature_map_count,
			int input_feature_map_group_size,
			int texture_offset,
			int entry_count,
			bool different_input,
			int packed_config_count)
		{
			int packed_config_id = blockIdx.x * blockDim.x + threadIdx.x;
			int entry_id = blockIdx.y * blockDim.y + threadIdx.y;

			bool in_bounds = (entry_id < entry_count) && (packed_config_id < packed_config_count);
			if (in_bounds)
			{
				packed_config<5> conf = packed_config_list[packed_config_id];
				int x = conf.get_val(0);
				int y = conf.get_val(1);
				int z = conf.get_val(2);
				int output_feature_map_id = conf.get_val(3);
				int base_input_feature_map_id = conf.get_val(4);

				int weight_count_per_output_feature_map = window_depth * window_height * WINDOW_WIDTH * input_feature_map_count;
				int input_elem_id = ((((different_input ? entry_id * input_feature_map_count : 0) + base_input_feature_map_id) * input_depth + z) * input_height + y) * input_width + x + texture_offset;
				int weights_offset = ((entry_id * output_feature_map_count + output_feature_map_id) * input_feature_map_count + base_input_feature_map_id) * window_depth * window_height * WINDOW_WIDTH;
				int iteration_count = min(input_feature_map_group_size, input_feature_map_count - base_input_feature_map_id);

				float initial_values[FEATURE_MAP_BLOCK_SIZE];
				#pragma unroll
				for(int i = 0; i < FEATURE_MAP_BLOCK_SIZE; ++i)
					initial_values[i] = 0.0F;
				if (base_input_feature_map_id == 0)
				{
					#pragma unroll
					for(int i = 0; i < FEATURE_MAP_BLOCK_SIZE; ++i)
						if (i < output_feature_map_count - output_feature_map_id)
							initial_values[i] = biases[entry_id * output_feature_map_count + output_feature_map_id + i];
				}
				float sums[BLOCK_SIZE * FEATURE_MAP_BLOCK_SIZE];
				#pragma unroll
				for(int i = 0; i < FEATURE_MAP_BLOCK_SIZE; ++i)
					#pragma unroll
					for(int j = 0; j < BLOCK_SIZE; ++j)
						sums[i * BLOCK_SIZE + j] = initial_values[i];

				for(int i = 0; i < iteration_count; ++i)
				{
					for(int input_z = 0; input_z < window_depth; ++input_z)
					{
						for(int input_y = 0; input_y < window_height; ++input_y)
						{
							#pragma unroll
							for(int input_x = 0; input_x < WINDOW_WIDTH; ++input_x)
							{
								float weight_list[FEATURE_MAP_BLOCK_SIZE];
								#pragma unroll
								for(int i = 0; i < FEATURE_MAP_BLOCK_SIZE; ++i)
									weight_list[i] = tex1Dfetch<float>(weights_tex, weights_offset + weight_count_per_output_feature_map * i);
								#pragma unroll
								for(int j = 0; j < BLOCK_SIZE; ++j)
								{
									float inp = tex1Dfetch<float>(input_tex, input_elem_id + j); 
									#pragma unroll
									for(int i = 0; i < FEATURE_MAP_BLOCK_SIZE; ++i)
										sums[i * BLOCK_SIZE + j] += inp * weight_list[i];
								}
								weights_offset++;
								input_elem_id++;
							} // for input_x
							input_elem_id += input_width - WINDOW_WIDTH;
						} // for input_y
						input_elem_id += input_width * (input_height - window_height);
					} // for input_z
					input_elem_id += input_height * input_width * (input_depth - window_depth);
				}

				float * base_output = output + (((entry_id * output_feature_map_count + output_feature_map_id) * output_depth + z) * output_height + y) * output_width + x;
				int output_neuron_count_per_feature_map = output_depth * output_height * output_width;
				if (single_input_feature_map_group)
				{
					#pragma unroll
					for(int i = 0; i < FEATURE_MAP_BLOCK_SIZE; ++i)
					{
						if (i < output_feature_map_count - output_feature_map_id)
						{
							#pragma unroll
							for(int j = 0; j < BLOCK_SIZE; ++j)
							{
								if (j < output_width - x)
									base_output[output_neuron_count_per_feature_map * i + j] = sums[i * BLOCK_SIZE + j];
							}
						}
					}
				}
				else
				{
					#pragma unroll
					for(int i = 0; i < FEATURE_MAP_BLOCK_SIZE; ++i)
					{
						if (i < output_feature_map_count - output_feature_map_id)
						{
							#pragma unroll
							for(int j = 0; j < BLOCK_SIZE; ++j)
							{
								if (j < output_width - x)
									atomicAdd(base_output + output_neuron_count_per_feature_map * i + j, sums[i * BLOCK_SIZE + j]);
							}
						}
					}
				}
			}
		}

		__global__ void convolution_3d_update_biases_upd_kernel_kepler(
			float * __restrict biases,
			const float * __restrict output_errors,
			const float * __restrict learning_rate,
			int output_feature_map_count,
			int output_elem_count_per_feature_map,
			int min_iteration_count)
		{
			int thread_id = threadIdx.x;
			int output_feature_map_id = blockIdx.y;
			int entry_id = blockIdx.z;
			int threadblock_size = blockDim.x;

			float sum = 0.0F;
			const float * current_error = output_errors + (entry_id * output_feature_map_count + output_feature_map_id) * output_elem_count_per_feature_map;
			int current_output_neuron_id = thread_id;
			for(int i = 0; i < min_iteration_count; ++i)
			{
				sum += current_error[current_output_neuron_id];
				current_output_neuron_id += threadblock_size;
			}
			if (current_output_neuron_id < output_elem_count_per_feature_map)
				sum += current_error[current_output_neuron_id];

			int lane_id = thread_id & 31;
			#pragma unroll
			for(int tx = 16; tx > 0; tx >>= 1)
			{
				sum += __shfl_down(sum, tx);
			}

			if (lane_id == 0)
			{
				int offset = entry_id * output_feature_map_count + output_feature_map_id;
				float current_learning_rate_val = learning_rate[offset];
				atomicAdd(biases + offset, sum * current_learning_rate_val);
			}
		}

		template<int BLOCK_SIZE, bool single_output_feature_map_group>
		__global__ void convolution_3d_deriviative_tex_upd_kernel_kepler(
			float * __restrict input_errors,
			hipTextureObject_t output_tex,
			hipTextureObject_t weights_tex,
			const packed_config<5> * __restrict packed_config_list,
			int output_width,
			int output_height,
			int output_depth,
			int input_width,
			int input_height,
			int input_depth,
			int window_width,
			int window_height,
			int window_depth,
			int input_feature_map_count,
			int output_feature_map_count,
			int output_feature_map_group_size,
			int entry_count,
			int packed_config_count)
		{
			int packed_config_id = blockIdx.x * blockDim.x + threadIdx.x;
			int entry_id = blockIdx.y * blockDim.y + threadIdx.y;

			bool in_bounds = (entry_id < entry_count) && (packed_config_id < packed_config_count);
			if (in_bounds)
			{
				packed_config<5> conf = packed_config_list[packed_config_id];
				int x = conf.get_val(0);
				int y = conf.get_val(1);
				int z = conf.get_val(2);
				int input_feature_map_id = conf.get_val(3);
				int base_output_feature_map_id = conf.get_val(4);

				int weight_count_per_input_feature_map = window_depth * window_height * window_width;
				int output_elem_id = (((entry_id * output_feature_map_count + base_output_feature_map_id) * output_depth + z) * output_height + y) * output_width + x;
				int weights_offset = ((entry_id * output_feature_map_count + base_output_feature_map_id) * input_feature_map_count + input_feature_map_id) * weight_count_per_input_feature_map;
				int iteration_count = min(output_feature_map_group_size, output_feature_map_count - base_output_feature_map_id);

				float sums[FEATURE_MAP_BLOCK_SIZE * BLOCK_SIZE];
				#pragma unroll
				for(int i = 0; i < FEATURE_MAP_BLOCK_SIZE * BLOCK_SIZE; ++i)
					sums[i] = 0.0F;

				int min_z_exclusive = z - output_depth;
				int max_z_inclusive = z;
				int min_y_exclusive = y - output_height;
				int max_y_inclusive = y;
				int min_x_exclusive = x - output_width;
				int max_x_inclusive = x;

				for(int i = 0; i < iteration_count; ++i)
				{
					for(int input_z = 0; input_z < window_depth; ++input_z)
					{
						bool b_fit_z = (input_z > min_z_exclusive) && (input_z <= max_z_inclusive);

						for(int input_y = 0; input_y < window_height; ++input_y)
						{
							bool b_fit_y = b_fit_z && (input_y > min_y_exclusive) && (input_y <= max_y_inclusive);

							int input_x = 0;
							#pragma unroll 1
							for(; input_x < (window_width - (WINDOW_WIDTH_LOCAL - 1)); input_x += WINDOW_WIDTH_LOCAL)
							{
								float output_vals[BLOCK_SIZE + WINDOW_WIDTH_LOCAL - 1];
								#pragma unroll
								for(int i = 0; i < BLOCK_SIZE + WINDOW_WIDTH_LOCAL - 1; ++i)
								{
									bool b_fit_x = b_fit_y && (i > min_x_exclusive) && (i <= max_x_inclusive);;
									if (b_fit_x)
										output_vals[i] = tex1Dfetch<float>(output_tex, output_elem_id - i);
									else
										output_vals[i] = 0.0F;
								}
								output_elem_id -= WINDOW_WIDTH_LOCAL;

								#pragma unroll
								for(int input_x_local = 0; input_x_local < WINDOW_WIDTH_LOCAL; ++input_x_local)
								{
									float weight_list[FEATURE_MAP_BLOCK_SIZE];
									#pragma unroll
									for(int i = 0; i < FEATURE_MAP_BLOCK_SIZE; ++i)
										weight_list[i] = tex1Dfetch<float>(weights_tex, weights_offset + weight_count_per_input_feature_map * i);

									#pragma unroll
									for(int j = 0; j < BLOCK_SIZE; ++j)
									{
										#pragma unroll
										for(int i = 0; i < FEATURE_MAP_BLOCK_SIZE; ++i)
											sums[i * BLOCK_SIZE + j] += output_vals[input_x_local + j] * weight_list[i];
									}
									weights_offset++;
								}
							}
							#pragma unroll 1
							for(; input_x < window_width; ++input_x)
							{
								#pragma unroll
								for(int j = 0; j < BLOCK_SIZE; ++j)
								{
									bool b_fit_x = b_fit_y && (input_x + j > min_x_exclusive) && (input_x + j <= max_x_inclusive);
									if (b_fit_x)
									{
										float inp = tex1Dfetch<float>(output_tex, output_elem_id - j);
										#pragma unroll
										for(int i = 0; i < FEATURE_MAP_BLOCK_SIZE; ++i)
											sums[i * BLOCK_SIZE + j] += inp * tex1Dfetch<float>(weights_tex, weights_offset + weight_count_per_input_feature_map * i);
									}
								}
								weights_offset++;
								output_elem_id--;
							}

							output_elem_id += window_width - output_width;
						} // for input_y
						output_elem_id += output_width * (window_height - output_height);
					} // for input_z
					output_elem_id += output_width * output_height * (output_depth + window_depth);
					weights_offset += weight_count_per_input_feature_map * (input_feature_map_count - 1);
				}

				float * base_input = input_errors + (((entry_id * input_feature_map_count + input_feature_map_id) * input_depth + z) * input_height + y) * input_width + x;
				int input_neuron_count_per_feature_map = input_depth * input_height * input_width;
				if (single_output_feature_map_group == 1)
				{
					#pragma unroll
					for(int i = 0; i < FEATURE_MAP_BLOCK_SIZE; ++i)
					{
						if (i < input_feature_map_count - input_feature_map_id)
						{
							#pragma unroll
							for(int j = 0; j < BLOCK_SIZE; ++j)
							{
								if (j > x - input_width)
									*(base_input + input_neuron_count_per_feature_map * i - j) = sums[i * BLOCK_SIZE + j];
							}
						}
					}
				}
				else
				{
					#pragma unroll
					for(int i = 0; i < FEATURE_MAP_BLOCK_SIZE; ++i)
					{
						if (i < input_feature_map_count - input_feature_map_id)
						{
							#pragma unroll
							for(int j = 0; j < BLOCK_SIZE; ++j)
							{
								if (j > x - input_width)
									atomicAdd(base_input + input_neuron_count_per_feature_map * i - j, sums[i * BLOCK_SIZE + j]);
							}
						}
					}
				}
			}
		}

		template<int WINDOW_WIDTH, int BLOCK_SIZE, bool single_output_feature_map_group>
		__global__ void convolution_3d_deriviative_tex_exact_upd_kernel_kepler(
			float * __restrict input_errors,
			hipTextureObject_t output_tex,
			hipTextureObject_t weights_tex,
			const packed_config<5> * __restrict packed_config_list,
			int output_width,
			int output_height,
			int output_depth,
			int input_width,
			int input_height,
			int input_depth,
			int window_height,
			int window_depth,
			int input_feature_map_count,
			int output_feature_map_count,
			int output_feature_map_group_size,
			int entry_count,
			int packed_config_count)
		{
			int packed_config_id = blockIdx.x * blockDim.x + threadIdx.x;
			int entry_id = blockIdx.y * blockDim.y + threadIdx.y;

			bool in_bounds = (entry_id < entry_count) && (packed_config_id < packed_config_count);
			if (in_bounds)
			{
				packed_config<5> conf = packed_config_list[packed_config_id];
				int x = conf.get_val(0);
				int y = conf.get_val(1);
				int z = conf.get_val(2);
				int input_feature_map_id = conf.get_val(3);
				int base_output_feature_map_id = conf.get_val(4);

				int weight_count_per_input_feature_map = window_depth * window_height * WINDOW_WIDTH;
				int output_elem_id = (((entry_id * output_feature_map_count + base_output_feature_map_id) * output_depth + z) * output_height + y) * output_width + x;
				int weights_offset = ((entry_id * output_feature_map_count + base_output_feature_map_id) * input_feature_map_count + input_feature_map_id) * weight_count_per_input_feature_map;
				int iteration_count = min(output_feature_map_group_size, output_feature_map_count - base_output_feature_map_id);

				float sums[FEATURE_MAP_BLOCK_SIZE * BLOCK_SIZE];
				#pragma unroll
				for(int i = 0; i < FEATURE_MAP_BLOCK_SIZE * BLOCK_SIZE; ++i)
					sums[i] = 0.0F;

				int min_z_exclusive = z - output_depth;
				int max_z_inclusive = z;
				int min_y_exclusive = y - output_height;
				int max_y_inclusive = y;
				int min_x_exclusive = x - output_width;
				int max_x_inclusive = x;

				unsigned int mask = 0;
				for(int i = BLOCK_SIZE + WINDOW_WIDTH - 2; i >= 0; --i)
					mask = mask << 1 | (((i > min_x_exclusive) && (i <= max_x_inclusive)) ? 1 : 0);

				for(int i = 0; i < iteration_count; ++i)
				{
					for(int input_z = 0; input_z < window_depth; ++input_z)
					{
						bool b_fit_z = (input_z > min_z_exclusive) && (input_z <= max_z_inclusive);

						for(int input_y = 0; input_y < window_height; ++input_y)
						{
							bool b_fit_y = b_fit_z && (input_y > min_y_exclusive) && (input_y <= max_y_inclusive);

							float output_vals[BLOCK_SIZE + WINDOW_WIDTH - 1];
							#pragma unroll
							for(int i = 0; i < BLOCK_SIZE + WINDOW_WIDTH - 1; ++i)
							{
								bool b_fit_x = b_fit_y && (((1 << i) & mask) != 0);
								if (b_fit_x)
									output_vals[i] = tex1Dfetch<float>(output_tex, output_elem_id - i);
								else
									output_vals[i] = 0.0F;
							}

							#pragma unroll
							for(int input_x = 0; input_x < WINDOW_WIDTH; ++input_x)
							{
								float weight_list[FEATURE_MAP_BLOCK_SIZE];
								#pragma unroll
								for(int i = 0; i < FEATURE_MAP_BLOCK_SIZE; ++i)
									weight_list[i] = tex1Dfetch<float>(weights_tex, weights_offset + weight_count_per_input_feature_map * i);

								#pragma unroll
								for(int j = 0; j < BLOCK_SIZE; ++j)
								{
									#pragma unroll
									for(int i = 0; i < FEATURE_MAP_BLOCK_SIZE; ++i)
										sums[i * BLOCK_SIZE + j] += output_vals[input_x + j] * weight_list[i];
								}
								weights_offset++;
							}
							output_elem_id -= output_width;
						} // for input_y
						output_elem_id += output_width * (window_height - output_height);
					} // for input_z
					output_elem_id += output_width * output_height * (output_depth + window_depth);
					weights_offset += weight_count_per_input_feature_map * (input_feature_map_count - 1);
				}

				float * base_input = input_errors + (((entry_id * input_feature_map_count + input_feature_map_id) * input_depth + z) * input_height + y) * input_width + x;
				int input_neuron_count_per_feature_map = input_depth * input_height * input_width;
				if (single_output_feature_map_group == 1)
				{
					#pragma unroll
					for(int i = 0; i < FEATURE_MAP_BLOCK_SIZE; ++i)
					{
						if (i < input_feature_map_count - input_feature_map_id)
						{
							#pragma unroll
							for(int j = 0; j < BLOCK_SIZE; ++j)
							{
								if (j > x - input_width)
									*(base_input + input_neuron_count_per_feature_map * i - j) = sums[i * BLOCK_SIZE + j];
							}
						}
					}
				}
				else
				{
					#pragma unroll
					for(int i = 0; i < FEATURE_MAP_BLOCK_SIZE; ++i)
					{
						if (i < input_feature_map_count - input_feature_map_id)
						{
							#pragma unroll
							for(int j = 0; j < BLOCK_SIZE; ++j)
							{
								if (j > x - input_width)
									atomicAdd(base_input + input_neuron_count_per_feature_map * i - j, sums[i * BLOCK_SIZE + j]);
							}
						}
					}
				}
			}
		}

		template<bool single_output_z_group>
		__global__ void convolution_3d_update_weights_upd_kernel_kepler(
			float * __restrict weights,
			hipTextureObject_t input_tex,
			hipTextureObject_t output_tex,
			const float * __restrict learning_rate,
			const packed_config<5> * __restrict packed_config_list,
			int output_width,
			int output_height,
			int output_depth,
			int input_width,
			int input_height,
			int input_depth,
			int window_width,
			int window_height,
			int window_depth,
			int input_feature_map_count,
			int output_feature_map_count,
			int output_z_group_count,
			int texture_offset,
			int entry_count,
			bool different_input,
			int packed_config_count)
		{
			int weight_x = (blockIdx.x * blockDim.x + threadIdx.x) * WINDOW_WIDTH_LOCAL;
			int packed_config_id = blockIdx.y * blockDim.y + threadIdx.y;
			int entry_id = blockIdx.z * blockDim.z + threadIdx.z;

			bool in_bounds = (packed_config_id < packed_config_count) && (entry_id < entry_count) && (weight_x < window_width); 
			if (in_bounds)
			{
				packed_config<5> conf = packed_config_list[packed_config_id];
				int weight_y = conf.get_val(0);
				int weight_z = conf.get_val(1);
				int input_feature_map_id = conf.get_val(2);
				int output_feature_map_id = conf.get_val(3);
				int output_z_start_id = conf.get_val(4);

				int output_neuron_count_per_feature_map = output_depth * output_width * output_height;
				int output_elem_id = (((entry_id * output_feature_map_count + output_feature_map_id) * output_depth + output_z_start_id) * output_height) * output_width;
				int input_elem_id = ((((different_input ? entry_id * input_feature_map_count : 0) + input_feature_map_id) * input_depth + weight_z + output_z_start_id) * input_height + weight_y) * input_width + texture_offset + weight_x;

				float sums[FEATURE_MAP_BLOCK_SIZE * WINDOW_WIDTH_LOCAL];
				#pragma unroll
				for(int i = 0; i < FEATURE_MAP_BLOCK_SIZE * WINDOW_WIDTH_LOCAL; ++i)
					sums[i] = 0.0F;

				for(int output_z = output_z_start_id; output_z < output_depth; output_z += output_z_group_count)
				{
					for(int output_y = 0; output_y < output_height; output_y++)
					{
						float input_buf[WINDOW_WIDTH_LOCAL];
						#pragma unroll
						for(int i = 1; i < WINDOW_WIDTH_LOCAL; ++i)
						{
							input_buf[i] = tex1Dfetch<float>(input_tex, input_elem_id);
							++input_elem_id;
						}

						for(int x = 0; x < output_width; ++x)
						{
							float output_error_list[FEATURE_MAP_BLOCK_SIZE];
							#pragma unroll
							for(int i = 0; i < FEATURE_MAP_BLOCK_SIZE; ++i)
								output_error_list[i] = tex1Dfetch<float>(output_tex, output_elem_id + output_neuron_count_per_feature_map * i);

							#pragma unroll
							for(int i = 0; i < WINDOW_WIDTH_LOCAL - 1; ++i)
								input_buf[i] = input_buf[i + 1];
							input_buf[WINDOW_WIDTH_LOCAL - 1] = tex1Dfetch<float>(input_tex, input_elem_id);

							#pragma unroll
							for(int i = 0; i < FEATURE_MAP_BLOCK_SIZE; ++i)
								#pragma unroll
								for(int j = 0; j < WINDOW_WIDTH_LOCAL; ++j)
									sums[i * WINDOW_WIDTH_LOCAL + j] += output_error_list[i] * input_buf[j];

							output_elem_id++;
							input_elem_id++;
						}

						input_elem_id += window_width - WINDOW_WIDTH_LOCAL;
					}

					output_elem_id += output_height * output_width * (output_z_group_count - 1);
					input_elem_id += input_height * input_width * (output_z_group_count - 1) + (input_width * (window_height - 1));
				}

				int offset = ((((entry_id * output_feature_map_count + output_feature_map_id) * input_feature_map_count + input_feature_map_id) * window_depth + weight_z) * window_height + weight_y) * window_width + weight_x;
				int weight_count_per_output_feature_map = input_feature_map_count * window_depth * window_height * window_width;
				float * cur_weights = weights + offset;
				const float * cur_learning_rate = learning_rate + offset;
				if (single_output_z_group)
				{
					#pragma unroll
					for(int i = 0; i < FEATURE_MAP_BLOCK_SIZE; ++i)
					{
						if (i < output_feature_map_count - output_feature_map_id)
						{
							#pragma unroll
							for(int j = 0; j < WINDOW_WIDTH_LOCAL; ++j)
								if (j < window_width - weight_x)
									cur_weights[i * weight_count_per_output_feature_map + j] += sums[i * WINDOW_WIDTH_LOCAL + j] * cur_learning_rate[i * weight_count_per_output_feature_map + j];
						}
					}
				}
				else
				{
					#pragma unroll
					for(int i = 0; i < FEATURE_MAP_BLOCK_SIZE; ++i)
					{
						if (i < output_feature_map_count - output_feature_map_id)
						{
							#pragma unroll
							for(int j = 0; j < WINDOW_WIDTH_LOCAL; ++j)
								if (j < window_width - weight_x)
									atomicAdd(cur_weights + i * weight_count_per_output_feature_map + j, sums[i * WINDOW_WIDTH_LOCAL + j] * cur_learning_rate[i * weight_count_per_output_feature_map + j]);
						}
					}
				}
			}
		}

		template<int WINDOW_WIDTH, bool single_output_z_group>
		__global__ void convolution_3d_update_weights_exact_upd_kernel_kepler(
			float * __restrict weights,
			hipTextureObject_t input_tex,
			hipTextureObject_t output_tex,
			const float * __restrict learning_rate,
			const packed_config<5> * __restrict packed_config_list,
			int output_width,
			int output_height,
			int output_depth,
			int input_width,
			int input_height,
			int input_depth,
			int window_height,
			int window_depth,
			int input_feature_map_count,
			int output_feature_map_count,
			int output_z_group_count,
			int texture_offset,
			int entry_count,
			bool different_input,
			int packed_config_count)
		{
			int packed_config_id = blockIdx.x * blockDim.x + threadIdx.x;
			int entry_id = blockIdx.y * blockDim.y + threadIdx.y;

			bool in_bounds = (packed_config_id < packed_config_count) && (entry_id < entry_count);
			if (in_bounds)
			{
				packed_config<5> conf = packed_config_list[packed_config_id];
				int weight_y = conf.get_val(0);
				int weight_z = conf.get_val(1);
				int input_feature_map_id = conf.get_val(2);
				int output_feature_map_id = conf.get_val(3);
				int output_z_start_id = conf.get_val(4);

				int output_neuron_count_per_feature_map = output_depth * output_width * output_height;
				int output_elem_id = (((entry_id * output_feature_map_count + output_feature_map_id) * output_depth + output_z_start_id) * output_height) * output_width;
				int input_elem_id = ((((different_input ? entry_id * input_feature_map_count : 0) + input_feature_map_id) * input_depth + weight_z + output_z_start_id) * input_height + weight_y) * input_width + texture_offset;

				float sums[FEATURE_MAP_BLOCK_SIZE * WINDOW_WIDTH];
				#pragma unroll
				for(int i = 0; i < FEATURE_MAP_BLOCK_SIZE * WINDOW_WIDTH; ++i)
					sums[i] = 0.0F;

				for(int output_z = output_z_start_id; output_z < output_depth; output_z += output_z_group_count)
				{
					for(int output_y = 0; output_y < output_height; output_y++)
					{
						float input_buf[WINDOW_WIDTH];
						#pragma unroll
						for(int i = 1; i < WINDOW_WIDTH; ++i)
						{
							input_buf[i] = tex1Dfetch<float>(input_tex, input_elem_id);
							++input_elem_id;
						}

						for(int x = 0; x < output_width; ++x)
						{
							float output_error_list[FEATURE_MAP_BLOCK_SIZE];
							#pragma unroll
							for(int i = 0; i < FEATURE_MAP_BLOCK_SIZE; ++i)
								output_error_list[i] = tex1Dfetch<float>(output_tex, output_elem_id + output_neuron_count_per_feature_map * i);

							#pragma unroll
							for(int i = 0; i < WINDOW_WIDTH - 1; ++i)
								input_buf[i] = input_buf[i + 1];
							input_buf[WINDOW_WIDTH - 1] = tex1Dfetch<float>(input_tex, input_elem_id);

							#pragma unroll
							for(int i = 0; i < FEATURE_MAP_BLOCK_SIZE; ++i)
								#pragma unroll
								for(int j = 0; j < WINDOW_WIDTH; ++j)
									sums[i * WINDOW_WIDTH + j] += output_error_list[i] * input_buf[j];

							output_elem_id++;
							input_elem_id++;
						}
					}
					output_elem_id += output_height * output_width * (output_z_group_count - 1);
					input_elem_id += input_height * input_width * (output_z_group_count - 1) + (input_width * (window_height - 1));
				}

				int offset = ((((entry_id * output_feature_map_count + output_feature_map_id) * input_feature_map_count + input_feature_map_id) * window_depth + weight_z) * window_height + weight_y) * WINDOW_WIDTH;
				int weight_count_per_output_feature_map = input_feature_map_count * window_depth * window_height * WINDOW_WIDTH;
				float * cur_weights = weights + offset;
				const float * cur_learning_rate = learning_rate + offset;
				if (single_output_z_group)
				{
					#pragma unroll
					for(int i = 0; i < FEATURE_MAP_BLOCK_SIZE; ++i)
					{
						if (i < output_feature_map_count - output_feature_map_id)
						{
							#pragma unroll
							for(int j = 0; j < WINDOW_WIDTH; ++j)
								cur_weights[i * weight_count_per_output_feature_map + j] += sums[i * WINDOW_WIDTH + j] * cur_learning_rate[i * weight_count_per_output_feature_map + j];
						}
					}
				}
				else
				{
					#pragma unroll
					for(int i = 0; i < FEATURE_MAP_BLOCK_SIZE; ++i)
					{
						if (i < output_feature_map_count - output_feature_map_id)
						{
							#pragma unroll
							for(int j = 0; j < WINDOW_WIDTH; ++j)
								atomicAdd(cur_weights + i * weight_count_per_output_feature_map + j, sums[i * WINDOW_WIDTH + j] * cur_learning_rate[i * weight_count_per_output_feature_map + j]);
						}
					}
				}
			}
		}

		convolution_3d_layer_updater_cuda_kepler::convolution_3d_layer_updater_cuda_kepler()
		{
		}

		convolution_3d_layer_updater_cuda_kepler::~convolution_3d_layer_updater_cuda_kepler()
		{
		}

#define MAX_BLOCK_SIZE 5
#define MAX_WINDOW_WIDTH 10

#define launch_exact_kernel_const_const(window_width_const, block_size_const, single_input_feature_map_group) \
	convolution_3d_tex_exact_upd_kernel_kepler<window_width_const,block_size_const,single_input_feature_map_group><<<kernel_dims.first, kernel_dims.second, 0, stream_id>>>(*output_neurons_buffer, input_tex, weights_tex, *data[1], packed_config_list, output_configuration_specific.dimension_sizes[0], output_configuration_specific.dimension_sizes[1], output_configuration_specific.dimension_sizes[2], input_configuration_specific.dimension_sizes[0], input_configuration_specific.dimension_sizes[1], input_configuration_specific.dimension_sizes[2], window_sizes[1], window_sizes[2], input_configuration_specific.feature_map_count, output_configuration_specific.feature_map_count, forward_input_feature_map_group_size, texture_offset, entry_count, different_input, packed_config_count);

#define launch_exact_kernel_const(window_width, block_size_const, single_input_feature_map_group) \
	switch (window_width) \
		{ \
		case 1: \
			launch_exact_kernel_const_const(1, block_size_const, single_input_feature_map_group); \
			break; \
		case 2: \
			launch_exact_kernel_const_const(2, block_size_const, single_input_feature_map_group); \
			break; \
		case 3: \
			launch_exact_kernel_const_const(3, block_size_const, single_input_feature_map_group); \
			break; \
		case 4: \
			launch_exact_kernel_const_const(4, block_size_const, single_input_feature_map_group); \
			break; \
		case 5: \
			launch_exact_kernel_const_const(5, block_size_const, single_input_feature_map_group); \
			break; \
		case 6: \
			launch_exact_kernel_const_const(6, block_size_const, single_input_feature_map_group); \
			break; \
		case 7: \
			launch_exact_kernel_const_const(7, block_size_const, single_input_feature_map_group); \
			break; \
		case 8: \
			launch_exact_kernel_const_const(8, block_size_const, single_input_feature_map_group); \
			break; \
		case 9: \
			launch_exact_kernel_const_const(9, block_size_const, single_input_feature_map_group); \
			break; \
		case 10: \
			launch_exact_kernel_const_const(10, block_size_const, single_input_feature_map_group); \
			break; \
		};

#define launch_exact_kernel(window_width, block_size, single_input_feature_map_group) \
	switch (block_size) \
		{ \
		case 1: \
			launch_exact_kernel_const(window_width, 1, single_input_feature_map_group); \
			break; \
		case 2: \
			launch_exact_kernel_const(window_width, 2, single_input_feature_map_group); \
			break; \
		case 3: \
			launch_exact_kernel_const(window_width, 3, single_input_feature_map_group); \
			break; \
		case 4: \
			launch_exact_kernel_const(window_width, 4, single_input_feature_map_group); \
			break; \
		case 5: \
			launch_exact_kernel_const(window_width, 5, single_input_feature_map_group); \
			break; \
		};

#define launch_kernel_const(block_size_const, single_input_feature_map_group) \
	convolution_3d_tex_upd_kernel_kepler<block_size_const,single_input_feature_map_group><<<kernel_dims.first, kernel_dims.second, 0, stream_id>>>(*output_neurons_buffer, input_tex, weights_tex, *data[1], packed_config_list, output_configuration_specific.dimension_sizes[0], output_configuration_specific.dimension_sizes[1], output_configuration_specific.dimension_sizes[2], input_configuration_specific.dimension_sizes[0], input_configuration_specific.dimension_sizes[1], input_configuration_specific.dimension_sizes[2], window_sizes[0], window_sizes[1], window_sizes[2], input_configuration_specific.feature_map_count, output_configuration_specific.feature_map_count, forward_input_feature_map_group_size, texture_offset, entry_count, different_input, packed_config_count);

#define launch_kernel(block_size, single_input_feature_map_group) \
	switch (block_size) \
		{ \
		case 1: \
			launch_kernel_const(1, single_input_feature_map_group); \
			break; \
		case 2: \
			launch_kernel_const(2, single_input_feature_map_group); \
			break; \
		case 3: \
			launch_kernel_const(3, single_input_feature_map_group); \
			break; \
		case 4: \
			launch_kernel_const(4, single_input_feature_map_group); \
			break; \
		case 5: \
			launch_kernel_const(5, single_input_feature_map_group); \
			break; \
		};

#define launch_backprop_exact_kernel_const_const(window_width_const, block_size_const, single_output_feature_map_group) \
	convolution_3d_deriviative_tex_exact_upd_kernel_kepler<window_width_const,block_size_const,single_output_feature_map_group><<<kernel_dims.first, kernel_dims.second, 0, stream_id>>>(*input_errors_buffer, output_tex, weights_tex, packed_config_list, output_configuration_specific.dimension_sizes[0], output_configuration_specific.dimension_sizes[1], output_configuration_specific.dimension_sizes[2], input_configuration_specific.dimension_sizes[0], input_configuration_specific.dimension_sizes[1], input_configuration_specific.dimension_sizes[2], window_sizes[1], window_sizes[2], input_configuration_specific.feature_map_count, output_configuration_specific.feature_map_count, backward_output_feature_map_group_size, entry_count, packed_config_count);

#define launch_backprop_exact_kernel_const(window_width, block_size_const, single_output_feature_map_group) \
	switch (window_width) \
		{ \
		case 1: \
			launch_backprop_exact_kernel_const_const(1, block_size_const, single_output_feature_map_group); \
			break; \
		case 2: \
			launch_backprop_exact_kernel_const_const(2, block_size_const, single_output_feature_map_group); \
			break; \
		case 3: \
			launch_backprop_exact_kernel_const_const(3, block_size_const, single_output_feature_map_group); \
			break; \
		case 4: \
			launch_backprop_exact_kernel_const_const(4, block_size_const, single_output_feature_map_group); \
			break; \
		case 5: \
			launch_backprop_exact_kernel_const_const(5, block_size_const, single_output_feature_map_group); \
			break; \
		case 6: \
			launch_backprop_exact_kernel_const_const(6, block_size_const, single_output_feature_map_group); \
			break; \
		case 7: \
			launch_backprop_exact_kernel_const_const(7, block_size_const, single_output_feature_map_group); \
			break; \
		case 8: \
			launch_backprop_exact_kernel_const_const(8, block_size_const, single_output_feature_map_group); \
			break; \
		case 9: \
			launch_backprop_exact_kernel_const_const(9, block_size_const, single_output_feature_map_group); \
			break; \
		case 10: \
			launch_backprop_exact_kernel_const_const(10, block_size_const, single_output_feature_map_group); \
			break; \
		};

#define launch_backprop_exact_kernel(window_width, block_size, single_output_feature_map_group) \
	switch (block_size) \
		{ \
		case 1: \
			launch_backprop_exact_kernel_const(window_width, 1, single_output_feature_map_group); \
			break; \
		case 2: \
			launch_backprop_exact_kernel_const(window_width, 2, single_output_feature_map_group); \
			break; \
		case 3: \
			launch_backprop_exact_kernel_const(window_width, 3, single_output_feature_map_group); \
			break; \
		case 4: \
			launch_backprop_exact_kernel_const(window_width, 4, single_output_feature_map_group); \
			break; \
		case 5: \
			launch_backprop_exact_kernel_const(window_width, 5, single_output_feature_map_group); \
			break; \
		};

#define launch_backprop_kernel_const(block_size_const, single_output_feature_map_group) \
	convolution_3d_deriviative_tex_upd_kernel_kepler<block_size_const,single_output_feature_map_group><<<kernel_dims.first, kernel_dims.second, 0, stream_id>>>(*input_errors_buffer, output_tex, weights_tex, packed_config_list, output_configuration_specific.dimension_sizes[0], output_configuration_specific.dimension_sizes[1], output_configuration_specific.dimension_sizes[2], input_configuration_specific.dimension_sizes[0], input_configuration_specific.dimension_sizes[1], input_configuration_specific.dimension_sizes[2], window_sizes[0], window_sizes[1], window_sizes[2], input_configuration_specific.feature_map_count, output_configuration_specific.feature_map_count, backward_output_feature_map_group_size, entry_count, packed_config_count);

#define launch_backprop_kernel(block_size, single_output_feature_map_group) \
	switch (block_size) \
		{ \
		case 1: \
			launch_backprop_kernel_const(1, single_output_feature_map_group); \
			break; \
		case 2: \
			launch_backprop_kernel_const(2, single_output_feature_map_group); \
			break; \
		case 3: \
			launch_backprop_kernel_const(3, single_output_feature_map_group); \
			break; \
		case 4: \
			launch_backprop_kernel_const(4, single_output_feature_map_group); \
			break; \
		case 5: \
			launch_backprop_kernel_const(5, single_output_feature_map_group); \
			break; \
		};

#define launch_update_weights_exact_kernel_const(window_width_const, single_output_z_group_const) \
	convolution_3d_update_weights_exact_upd_kernel_kepler<window_width_const, single_output_z_group_const><<<kernel_dims.first, kernel_dims.second, 0, stream_id>>>(*data[0], input_tex, output_tex, *learning_rate[0], packed_config_list, output_configuration_specific.dimension_sizes[0], output_configuration_specific.dimension_sizes[1], output_configuration_specific.dimension_sizes[2], input_configuration_specific.dimension_sizes[0], input_configuration_specific.dimension_sizes[1], input_configuration_specific.dimension_sizes[2], window_sizes[1], window_sizes[2], input_configuration_specific.feature_map_count, output_configuration_specific.feature_map_count, updater_output_z_group_count, texture_offset, entry_count, different_input, packed_config_count);

#define launch_update_weights_exact_kernel(window_width, single_output_z_group_const) \
	switch (window_width) \
		{ \
		case 1: \
			launch_update_weights_exact_kernel_const(1, single_output_z_group_const); \
			break; \
		case 2: \
			launch_update_weights_exact_kernel_const(2, single_output_z_group_const); \
			break; \
		case 3: \
			launch_update_weights_exact_kernel_const(3, single_output_z_group_const); \
			break; \
		case 4: \
			launch_update_weights_exact_kernel_const(4, single_output_z_group_const); \
			break; \
		case 5: \
			launch_update_weights_exact_kernel_const(5, single_output_z_group_const); \
			break; \
		case 6: \
			launch_update_weights_exact_kernel_const(6, single_output_z_group_const); \
			break; \
		case 7: \
			launch_update_weights_exact_kernel_const(7, single_output_z_group_const); \
			break; \
		case 8: \
			launch_update_weights_exact_kernel_const(8, single_output_z_group_const); \
			break; \
		case 9: \
			launch_update_weights_exact_kernel_const(9, single_output_z_group_const); \
			break; \
		case 10: \
			launch_update_weights_exact_kernel_const(10, single_output_z_group_const); \
			break; \
		};

#define launch_update_weights_kernel_const(single_output_z_group_const) \
	convolution_3d_update_weights_upd_kernel_kepler<single_output_z_group_const><<<kernel_dims.first, kernel_dims.second, 0, stream_id>>>(*data[0], input_tex, output_tex, *learning_rate[0], packed_config_list, output_configuration_specific.dimension_sizes[0], output_configuration_specific.dimension_sizes[1], output_configuration_specific.dimension_sizes[2], input_configuration_specific.dimension_sizes[0], input_configuration_specific.dimension_sizes[1], input_configuration_specific.dimension_sizes[2], window_sizes[0], window_sizes[1], window_sizes[2], input_configuration_specific.feature_map_count, output_configuration_specific.feature_map_count, updater_output_z_group_count, texture_offset, entry_count, different_input, packed_config_count);

		void convolution_3d_layer_updater_cuda_kepler::enqueue_test(
			unsigned int offset_input_entry_id,
			hipStream_t stream_id,
			const std::vector<const_cuda_linear_buffer_device_smart_ptr>& schema_data,
			const std::vector<cuda_linear_buffer_device_smart_ptr>& data,
			const_cuda_linear_buffer_device_smart_ptr input_neurons_buffer,
			cuda_linear_buffer_device_smart_ptr output_neurons_buffer,
			const std::vector<cuda_linear_buffer_device_smart_ptr>& additional_buffers,
			std::vector<cuda_memobject_smart_ptr>& dynamic_memobjects,
			unsigned int entry_count)
		{
			if (dynamic_memobjects[0] == 0)
				dynamic_memobjects[0] = cuda_texture_smart_ptr(new cuda_texture(input_neurons_buffer));
			cuda_texture& input_tex = *(dynamic_cast<cuda_texture *>(dynamic_memobjects[0].get()));
			int texture_offset = offset_input_entry_id * input_elem_count_per_entry;

			if (dynamic_memobjects[1] == 0)
				dynamic_memobjects[1] = cuda_texture_smart_ptr(new cuda_texture(data[0]));
			cuda_texture& weights_tex = *(dynamic_cast<cuda_texture *>(dynamic_memobjects[1].get()));

			if (forward_input_feature_map_group_count > 1)
				cuda_util::set_with_value(
					*cuda_config,
					*output_neurons_buffer,
					0.0F,
					output_elem_count_per_entry * entry_count,
					stream_id);

			int packed_config_count = forward_x_block_count * output_configuration_specific.dimension_sizes[1] * output_configuration_specific.dimension_sizes[2] * forward_output_feature_map_block_count * forward_input_feature_map_group_count;
			const packed_config<5> * packed_config_list = static_cast<const packed_config<5> *>((const void *)*additional_buffers[0]);

			std::pair<dim3, dim3> kernel_dims = cuda_util::get_grid_and_threadblock_sizes_sequential_access(
				*cuda_config,
				packed_config_count,
				entry_count,
				1);

			if (window_sizes[0] <= MAX_WINDOW_WIDTH)
			{
				if (forward_input_feature_map_group_count == 1)
				{
					launch_exact_kernel(window_sizes[0], forward_x_block_size, true);
				}
				else
				{
					launch_exact_kernel(window_sizes[0], forward_x_block_size, false);
				}
			}
			else
			{
				if (forward_input_feature_map_group_count == 1)
				{
					launch_kernel(forward_x_block_size, true);
				}
				else
				{
					launch_kernel(forward_x_block_size, false);
				}
			}
		}

		void convolution_3d_layer_updater_cuda_kepler::enqueue_backprop(
			hipStream_t stream_id,
			const std::vector<const_cuda_linear_buffer_device_smart_ptr>& schema_data,
			const std::vector<cuda_linear_buffer_device_smart_ptr>& data,
			const_cuda_linear_buffer_device_smart_ptr output_neurons_buffer,
			const_cuda_linear_buffer_device_smart_ptr input_neurons_buffer,
			cuda_linear_buffer_device_smart_ptr output_errors_buffer,
			cuda_linear_buffer_device_smart_ptr input_errors_buffer,
			const std::vector<cuda_linear_buffer_device_smart_ptr>& additional_buffers,
			std::vector<cuda_memobject_smart_ptr>& dynamic_memobjects,
			unsigned int entry_count)
		{
			if (!different_input)
				throw neural_network_exception("convolution_2d_layer_updater_cuda_kepler is not able to backprop to the same input");

			if (!backprop_required)
				throw neural_network_exception("convolution_2d_layer_updater_cuda_kepler is not configured to do backprop but requested to");

			if (dynamic_memobjects[2] == 0)
				dynamic_memobjects[2] = cuda_texture_smart_ptr(new cuda_texture(output_errors_buffer));
			cuda_texture& output_tex = *(dynamic_cast<cuda_texture *>(dynamic_memobjects[2].get()));

			if (dynamic_memobjects[1] == 0)
				dynamic_memobjects[1] = cuda_texture_smart_ptr(new cuda_texture(data[0]));
			cuda_texture& weights_tex = *(dynamic_cast<cuda_texture *>(dynamic_memobjects[1].get()));

			if (backward_output_feature_map_group_count > 1)
				cuda_util::set_with_value(
					*cuda_config,
					*input_errors_buffer,
					0.0F,
					input_elem_count_per_entry * entry_count,
					stream_id);

			int packed_config_count = backward_x_block_count * input_configuration_specific.dimension_sizes[1] * input_configuration_specific.dimension_sizes[2] * backward_input_feature_map_block_count * backward_output_feature_map_group_count;
			const packed_config<5> * packed_config_list = static_cast<const packed_config<5> *>((const void *)*additional_buffers[2]);

			std::pair<dim3, dim3> kernel_dims = cuda_util::get_grid_and_threadblock_sizes_sequential_access(
				*cuda_config,
				packed_config_count,
				entry_count,
				1);

			if (window_sizes[0] <= MAX_WINDOW_WIDTH)
			{
				if (backward_output_feature_map_group_count == 1)
				{
					launch_backprop_exact_kernel(window_sizes[0], backward_x_block_size, true);
				}
				else
				{
					launch_backprop_exact_kernel(window_sizes[0], backward_x_block_size, false);
				}
			}
			else
			{
				if (backward_output_feature_map_group_count == 1)
				{
					launch_backprop_kernel(backward_x_block_size, true);
				}
				else
				{
					launch_backprop_kernel(backward_x_block_size, false);
				}
			}
		}

		void convolution_3d_layer_updater_cuda_kepler::enqueue_update_weights(
			unsigned int offset_input_entry_id,
			hipStream_t stream_id,
			const std::vector<cuda_linear_buffer_device_smart_ptr>& data,
			const std::vector<const_cuda_linear_buffer_device_smart_ptr>& schema_data,
			const std::vector<const_cuda_linear_buffer_device_smart_ptr>& learning_rate,
			cuda_linear_buffer_device_smart_ptr output_errors_buffer,
			const_cuda_linear_buffer_device_smart_ptr input_neurons_buffer,
			const std::vector<cuda_linear_buffer_device_smart_ptr>& additional_buffers,
			std::vector<cuda_memobject_smart_ptr>& dynamic_memobjects,
			unsigned int entry_count)
		{
			// Update biases
			{
				int threadblock_size = get_threadblock_size_biases(output_elem_count_per_feature_map);
				dim3 grid_size(1, output_configuration_specific.feature_map_count, entry_count);
				dim3 block_size(threadblock_size, 1, 1);
				int min_iteration_count = output_elem_count_per_feature_map / threadblock_size;

				convolution_3d_update_biases_upd_kernel_kepler<<<grid_size, block_size, 0, stream_id>>>(
					*data[1],
					*output_errors_buffer,
					*learning_rate[1],
					output_configuration_specific.feature_map_count,
					output_elem_count_per_feature_map,
					min_iteration_count);
			}

			if (dynamic_memobjects[2] == 0)
				dynamic_memobjects[2] = cuda_texture_smart_ptr(new cuda_texture(output_errors_buffer));
			cuda_texture& output_tex = *(dynamic_cast<cuda_texture *>(dynamic_memobjects[2].get()));

			if (dynamic_memobjects[0] == 0)
				dynamic_memobjects[0] = cuda_texture_smart_ptr(new cuda_texture(input_neurons_buffer));
			cuda_texture& input_tex = *(dynamic_cast<cuda_texture *>(dynamic_memobjects[0].get()));
			int texture_offset = offset_input_entry_id * input_elem_count_per_entry;

			int packed_config_count = window_sizes[1] * window_sizes[2] * updater_output_z_group_count * updater_output_feature_map_block_count * input_configuration_specific.feature_map_count;
			const packed_config<5> * packed_config_list = static_cast<const packed_config<5> *>((const void *)*additional_buffers[1]);

			// Update weights
			{
				if (updater_window_x_block_count == 1)
				{
					std::pair<dim3, dim3> kernel_dims = cuda_util::get_grid_and_threadblock_sizes_sequential_access(
						*cuda_config,
						packed_config_count,
						entry_count,
						1);

					if (updater_output_z_group_count == 1)
					{
						launch_update_weights_exact_kernel(window_sizes[0], true);
					}
					else
					{
						launch_update_weights_exact_kernel(window_sizes[0], false);
					}
				}
				else
				{
					std::pair<dim3, dim3> kernel_dims = cuda_util::get_grid_and_threadblock_sizes_sequential_access(
						*cuda_config,
						updater_window_x_block_count,
						packed_config_count,
						entry_count);

					if (updater_output_z_group_count == 1)
					{
						launch_update_weights_kernel_const(true);
					}
					else
					{
						launch_update_weights_kernel_const(false);
					}
				}
			}
		}

		int convolution_3d_layer_updater_cuda_kepler::get_block_size(int width)
		{
			int block_count = (width + MAX_BLOCK_SIZE - 1) / MAX_BLOCK_SIZE;
			int block_size = (width + block_count - 1) / block_count;
			return block_size;
		}

		void convolution_3d_layer_updater_cuda_kepler::updater_configured()
		{
			std::tr1::shared_ptr<const convolution_layer> layer_derived = std::tr1::dynamic_pointer_cast<const convolution_layer>(layer_schema);

			for(std::vector<unsigned int>::const_iterator it = layer_derived->window_sizes.begin(); it != layer_derived->window_sizes.end(); ++it)
				window_sizes.push_back(static_cast<int>(*it));

			forward_x_block_size = get_block_size(output_configuration_specific.dimension_sizes[0]);
			forward_x_block_count = (output_configuration_specific.dimension_sizes[0] + forward_x_block_size - 1) / forward_x_block_size;
			forward_output_feature_map_block_count = (output_configuration_specific.feature_map_count + FEATURE_MAP_BLOCK_SIZE - 1) / FEATURE_MAP_BLOCK_SIZE;

			updater_output_feature_map_block_count = (output_configuration_specific.feature_map_count + FEATURE_MAP_BLOCK_SIZE - 1) / FEATURE_MAP_BLOCK_SIZE;
			updater_window_x_block_count = (window_sizes[0] <= MAX_WINDOW_WIDTH) ? 1 : (window_sizes[0] + WINDOW_WIDTH_LOCAL - 1) / WINDOW_WIDTH_LOCAL;
			{
				std::tr1::array<int, 3> size_list;
				size_list[0] = window_sizes[1];
				size_list[1] = window_sizes[2];
				size_list[2] = input_configuration_specific.feature_map_count;
				space_filling_curve<3>::fill_pattern(size_list, updater_config_ordered_list1);
			}

			if (backprop_required)
			{
				backward_x_block_size = get_block_size(input_configuration_specific.dimension_sizes[0]);
				backward_x_block_count = (input_configuration_specific.dimension_sizes[0] + backward_x_block_size - 1) / backward_x_block_size;
				backward_input_feature_map_block_count = (input_configuration_specific.feature_map_count + FEATURE_MAP_BLOCK_SIZE - 1) / FEATURE_MAP_BLOCK_SIZE;
			}
		}

		bool convolution_3d_layer_updater_cuda_kepler::is_in_place_backprop() const
		{
			return false;
		}

		std::vector<unsigned int> convolution_3d_layer_updater_cuda_kepler::get_linear_addressing_through_texture_per_entry() const
		{
			std::vector<unsigned int> res;

			res.push_back(input_elem_count_per_entry);
			res.push_back(output_elem_count_per_entry);

			return res;
		}

		int convolution_3d_layer_updater_cuda_kepler::get_threadblock_size_biases(int output_neuron_count)
		{
			int threadblock_size;

			if (output_neuron_count < 128)
			{
				threadblock_size = (output_neuron_count + 32 - 1) / 32 * 32;
			}
			else
			{
				int threadblock_count = (output_neuron_count + 128 - 1) / 128;
				threadblock_size = (output_neuron_count + threadblock_count - 1) / threadblock_count;
				threadblock_size = (threadblock_size + 32 - 1) / 32 * 32;
			}

			return threadblock_size;
		}

		std::vector<size_t> convolution_3d_layer_updater_cuda_kepler::get_sizes_of_additional_buffers_fixed() const
		{
			std::vector<size_t> res;

			res.push_back(sizeof(packed_config<5>) * forward_x_block_count * output_configuration_specific.dimension_sizes[1] * output_configuration_specific.dimension_sizes[2] * input_configuration_specific.feature_map_count * forward_output_feature_map_block_count);

			res.push_back(sizeof(packed_config<5>) * window_sizes[1] * window_sizes[2] * output_configuration_specific.dimension_sizes[1] * input_configuration_specific.feature_map_count * updater_output_feature_map_block_count);

			if (backprop_required)
				res.push_back(sizeof(packed_config<5>) * backward_x_block_count * input_configuration_specific.dimension_sizes[1] * input_configuration_specific.dimension_sizes[2] * output_configuration_specific.feature_map_count * backward_input_feature_map_block_count);

			return res;
		}

		void convolution_3d_layer_updater_cuda_kepler::fill_additional_buffers(const std::vector<cuda_linear_buffer_device_smart_ptr>& additional_buffers) const
		{
			{
				std::vector<packed_config<5> > task_list;
				packed_config<5> new_elem;

				for(int input_feature_map_group_id = 0; input_feature_map_group_id < forward_input_feature_map_group_count; ++input_feature_map_group_id)
				{
					new_elem.set_val(4, input_feature_map_group_id * forward_input_feature_map_group_size);
					for(int output_feature_map_block_id = 0; output_feature_map_block_id < forward_output_feature_map_block_count; ++output_feature_map_block_id)
					{
						new_elem.set_val(3, output_feature_map_block_id * FEATURE_MAP_BLOCK_SIZE);
						for(int z = 0; z < output_configuration_specific.dimension_sizes[2]; ++z)
						{
							new_elem.set_val(2, z);
							for(int y = 0; y < output_configuration_specific.dimension_sizes[1]; ++y)
							{
								new_elem.set_val(1, y);
								for(int x = 0; x < forward_x_block_count; ++x)
								{
									new_elem.set_val(0, x * forward_x_block_size);
									task_list.push_back(new_elem);
								}
							}
						}
					}
				}

				cuda_safe_call(hipMemcpy(*additional_buffers[0], &(*task_list.begin()), sizeof(packed_config<5>) * task_list.size(), hipMemcpyHostToDevice));
			}

			{
				std::vector<packed_config<5> > task_list;
				packed_config<5> new_elem;

				for(std::vector<std::tr1::array<int, 2> >::const_iterator it2 = updater_config_ordered_list2.begin(); it2 != updater_config_ordered_list2.end(); ++it2)
				{
					new_elem.set_val(3, it2->at(0) * FEATURE_MAP_BLOCK_SIZE); 
					new_elem.set_val(4, it2->at(1));
					for(std::vector<std::tr1::array<int, 3> >::const_iterator it1 = updater_config_ordered_list1.begin(); it1 != updater_config_ordered_list1.end(); ++it1)
					{
						new_elem.set_val(0, it1->at(0));
						new_elem.set_val(1, it1->at(1));
						new_elem.set_val(2, it1->at(2));
						task_list.push_back(new_elem);
					}
				}

				cuda_safe_call(hipMemcpy(*additional_buffers[1], &(*task_list.begin()), sizeof(packed_config<5>) * task_list.size(), hipMemcpyHostToDevice));
			}

			if (backprop_required)
			{
				std::vector<packed_config<5> > task_list;
				packed_config<5> new_elem;

				for(int output_feature_map_group_id = 0; output_feature_map_group_id < backward_output_feature_map_group_count; ++output_feature_map_group_id)
				{
					new_elem.set_val(4, output_feature_map_group_id * backward_output_feature_map_group_size);
					for(int input_feature_map_block_id = 0; input_feature_map_block_id < backward_input_feature_map_block_count; ++input_feature_map_block_id)
					{
						new_elem.set_val(3, input_feature_map_block_id * FEATURE_MAP_BLOCK_SIZE);
						for(int z = 0; z < input_configuration_specific.dimension_sizes[2]; ++z)
						{
							new_elem.set_val(2, z);
							for(int y = 0; y < input_configuration_specific.dimension_sizes[1]; ++y)
							{
								new_elem.set_val(1, y);
								for(int x = 0; x < backward_x_block_count; ++x)
								{
									new_elem.set_val(0, x * backward_x_block_size + (backward_x_block_size - 1));
									task_list.push_back(new_elem);
								}
							}
						}
					}
				}

				cuda_safe_call(hipMemcpy(*additional_buffers[2], &(*task_list.begin()), sizeof(packed_config<5>) * task_list.size(), hipMemcpyHostToDevice));
			}
		}

		void convolution_3d_layer_updater_cuda_kepler::set_max_entry_count(unsigned int max_entry_count)
		{
			forward_input_feature_map_group_count = cuda_util::get_group_count(
				*cuda_config,
				forward_x_block_count * output_configuration_specific.dimension_sizes[1] * output_configuration_specific.dimension_sizes[2] * forward_output_feature_map_block_count * max_entry_count,
				input_configuration_specific.feature_map_count);
			forward_input_feature_map_group_size = (input_configuration_specific.feature_map_count + forward_input_feature_map_group_count - 1) / forward_input_feature_map_group_count;

			updater_output_z_group_count = cuda_util::get_group_count(
				*cuda_config,
				updater_output_feature_map_block_count * input_configuration_specific.feature_map_count * max_entry_count * updater_window_x_block_count * window_sizes[1] * window_sizes[2],
				output_configuration_specific.dimension_sizes[2]);
			updater_output_z_group_size = (output_configuration_specific.dimension_sizes[2] + updater_output_z_group_count - 1) / updater_output_z_group_count;
			{
				std::tr1::array<int, 2> size_list;
				size_list[0] = updater_output_feature_map_block_count;
				size_list[1] = updater_output_z_group_count;
				space_filling_curve<2>::fill_pattern(size_list, updater_config_ordered_list2);
			}

			if (backprop_required)
			{
				backward_output_feature_map_group_count = cuda_util::get_group_count(
					*cuda_config,
					backward_x_block_count * input_configuration_specific.dimension_sizes[1] * input_configuration_specific.dimension_sizes[2] * backward_input_feature_map_block_count * max_entry_count,
					output_configuration_specific.feature_map_count);
				backward_output_feature_map_group_size = (output_configuration_specific.feature_map_count + backward_output_feature_map_group_count - 1) / backward_output_feature_map_group_count;
			}
		}

		int convolution_3d_layer_updater_cuda_kepler::get_dynamic_memobject_count() const
		{
			return 3;
		}
	}
}
